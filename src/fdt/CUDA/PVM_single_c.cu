#include "hip/hip_runtime.h"
/*  PVM_single_c.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include "diffmodels_utils.h"
#include "levenberg_marquardt.cu"
#include "options.h"

#include <fstream>

/////////////////////////////////////
/////////////////////////////////////
/// 	    PVM_single_c	  /// 
/////////////////////////////////////
/////////////////////////////////////

__device__ 
inline float isoterm_PVM_single_c(const int pt,const float* _d,const float *bvals){
  	return exp(-bvals[pt]**_d);
}

__device__ 
inline float isoterm_lambda_PVM_single_c(const int pt,const float lambda,const float *bvals){
  	return(-2*bvals[pt]*lambda*exp(-bvals[pt]*lambda*lambda));
}

__device__ 
inline float anisoterm_PVM_single_c(const int pt,const float* _d,const float3 x, const float *bvecs, const float *bvals, const int ndirections){
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	return exp(-bvals[pt]**_d*dp*dp);
}

__device__ 
inline float anisoterm_lambda_PVM_single_c(const int pt,const float lambda,const float3 x, const float *bvecs, const float *bvals, const int ndirections){
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	return(-2*bvals[pt]*lambda*dp*dp*exp(-bvals[pt]*lambda*lambda*dp*dp));
}

__device__ 
inline float anisoterm_th_PVM_single_c(const int pt,const float* _d,const float3 x, const float _th,const float _ph,const float *bvecs, const float *bvals, const int ndirections){
	float sinth,costh,sinph,cosph;
	sincos(_th,&sinth,&costh);
	sincos(_ph,&sinph,&cosph);
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	float dp1 = costh*(bvecs[pt]*cosph+bvecs[ndirections+pt]*sinph)-bvecs[(2*ndirections)+pt]*sinth;
  	return(-2*bvals[pt]**_d*dp*dp1*exp(-bvals[pt]**_d*dp*dp));
}

__device__ 
inline float anisoterm_ph_PVM_single_c(const int pt,const float* _d,const float3 x, const float _th,const float _ph,const float *bvecs, const float *bvals, const int ndirections){
	float sinth,sinph,cosph;
	sinth=sin(_th);
	sincos(_ph,&sinph,&cosph);
  	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	float dp1 = sinth*(-bvecs[pt]*sinph+bvecs[ndirections+pt]*cosph);
  	return(-2*bvals[pt]**_d*dp*dp1*exp(-bvals[pt]**_d*dp*dp));
}

//If the sum of the fractions is >1, then zero as many fractions
//as necessary, so that the sum becomes smaller than 1.
//in diffmodel.cc
__device__ void fix_fsum_PVM_single_c(		//INPUT 
						int nfib,
						//INPUT - OUTPUT){
						float *fs)
{
  	float sumf=0.0;
  	for(int i=0;i<nfib;i++){
    		sumf+=fs[i];
    		if(sumf>=1){
      			for(int j=i;j<nfib;j++) 
				fs[j]=FSMALL_gpu;  //make the fraction almost zero
      			break;
    		}
  	}
}

//in diffmodel.cc
__device__ void sort_PVM_single_c(int nfib,float* params)
{
	float temp_f, temp_th, temp_ph;
	// Order vector descending using f parameters as index
  	for(int i=1; i<(nfib); i++){ 
    		for(int j=0; j<(nfib-i); j++){ 
      			if (params[2+j*3] < params[2+(j+1)*3]){ 
        			temp_f = params[2+j*3];
				temp_th = params[2+j*3+1];
				temp_ph = params[2+j*3+2];
        			params[2+j*3] = params[2+(j+1)*3]; 
				params[2+j*3+1] = params[2+(j+1)*3+1]; 
				params[2+j*3+2] = params[2+(j+1)*3+2]; 
        			params[2+(j+1)*3] = temp_f; 
				params[2+(j+1)*3+1] = temp_th; 
				params[2+(j+1)*3+2] = temp_ph; 
      			} 
    		} 
  	} 
}

__device__  void fractions_deriv_PVM_single_c(	//INPUT
						const float*	params,
						const float* 	fs, 
						const int	nfib,
						const int	idSubVOX,
						//OUTPUT
						float* 		Deriv) 
{
	int nparams_per_fibre=3;
  	float fsum;
	int k=idSubVOX%nfib;
	for (int j=0; j<nfib; j++){
		Deriv[j*nfib+k]=0;
    	}

  	int kk = 2+(k*nparams_per_fibre);
	float sinparamkk = sin(2*params[kk]);

	for (int j=0; j<nfib; j++){
		int jj = 2+(j*nparams_per_fibre);
      		if (j==k){
			fsum=1; 
			for (int n=0; n<=(j-1); n++){
	  			fsum-=fs[n];
			}
			Deriv[j*nfib+k]=sinparamkk*fsum;
      		}else if (j>k){
			float sinparam = sin(params[jj]);
			fsum=0;
			for (int n=0; n<=(j-1); n++){
	  			fsum+=Deriv[n*nfib+k];
			}
			Deriv[j*nfib+k]=  -(sinparam*sinparam)*fsum;
      		}
    	}
}

//cost function PVM_single_c
__device__ void cf_PVM_single_c(	//INPUT
					const float*		params,
					const float*		mdata,
					const float*		bvecs, 
					const float*		bvals,
					const int 		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,
					float*			reduction,	//shared memory
					float* 			fs,		//shared memory
					float*			x,		//shared memory	
					float* 			_d,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					double*			cfv)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}
	if(idSubVOX==0){
		*_d = lambda2d_gpu(params[1]);
		*cfv = 0.0;
		*sumf=0;
		float partial_fsum;
		for(int k=0;k<nfib;k++){
			int kk = 2+3*(k);
    			//partial_fsum ///////////
			partial_fsum=1.0;
			for(int j=0;j<k;j++)
				partial_fsum-=fs[j];
    			//////////////////////////
			fs[k] = beta2f_gpu(params[kk])*partial_fsum;
    			*sumf += fs[k];
		}
	}

	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	
	float err;
	float3 x2;
	int dir_iter=idSubVOX;

	__syncthreads();
	
	reduction[idSubVOX]=0;
	for(int dir=0;dir<ndir;dir++){
		err = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	
			err += fs[k]*anisoterm_PVM_single_c(dir_iter,_d,x2,bvecs,bvals,ndirections); 
    		}
		if(m_include_f0){
			//partial_fsum ///////////
			float partial_fsum=1.0;
			for(int j=0;j<nfib;j++)
				partial_fsum-=fs[j];
	     		//////////////////////////
			float temp_f0=beta2f_gpu(params[nparams-1])*partial_fsum;
			err= (params[0]*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single_c(dir_iter,_d,bvals))+err))-mdata[dir_iter];
		}else{
			err = params[0]*((1-*sumf)*isoterm_PVM_single_c(dir_iter,_d,bvals)+err)-mdata[dir_iter];
		}
		reduction[idSubVOX]+= err*err;  
		dir_iter+=THREADS_BLOCK_FIT;
  	}  

	__syncthreads();

	if(idSubVOX==0){
		for(int i=0;i<THREADS_BLOCK_FIT;i++){
			*cfv+=reduction[i];
		}	
	}	
}


//gradient function PVM_single_c
__device__ void grad_PVM_single_c(	//INPUT
					const float*		params,
					const float*		mdata,
					const float*		bvecs, 
					const float*		bvals,
					const int		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,	
					float*			J,		//shared memory	
					float*			reduction,	//shared memory
					float* 			fs,		//shared memory
					float*			f_deriv,	//shared memory
					float*			x,		//shared memory
					float* 			_d,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					float*			grad)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}
	if(idSubVOX==0){
		*_d = lambda2d_gpu(params[1]);
		*sumf=0;
		float partial_fsum;
		for(int k=0;k<nfib;k++){
			int kk = 2+3*(k);
    			//partial_fsum ///////////
			partial_fsum=1.0;
			for(int j=0;j<k;j++)
				partial_fsum-=fs[j];
    			//////////////////////////
			fs[k] = beta2f_gpu(params[kk])*partial_fsum;
    			*sumf += fs[k];
		}
		for (int p=0;p<nparams;p++) grad[p]=0;
	}

	__syncthreads();

  	if(idSubVOX<nfib){ 
		fractions_deriv_PVM_single_c(params,fs,nfib,idSubVOX,f_deriv); 
	} 

	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	float* myJ = &J[idSubVOX*nparams];
	float diff;
  	float sig;
	float Iso_term;
	float3 xx;
	int dir_iter=idSubVOX;
  	//float Aniso_terms[MAXNFIBRES];  //reuse Shared J --- myJ[kk+1]

	__syncthreads();

  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) myJ[p]=0;
		if(dir<ndir){
    			for(int k=0;k<nfib;k++){
				int kk = 2+3*(k) +1;
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
     				xx.z=x[k*3+2];	
      				//Aniso_terms[k]=anisoterm_PVM_single_c(dir_iter,_d,xx,bvecs,bvals,ndirections);
				myJ[kk] = anisoterm_PVM_single_c(dir_iter,_d,xx,bvecs,bvals,ndirections);
    			}
			Iso_term=isoterm_PVM_single_c(dir_iter,_d,bvals);  //Precompute some terms for this datapoint
    			sig = 0;
    			for(int k=0;k<nfib;k++){
     				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];		
      				sig += fs[k]*myJ[kk+1];//Aniso_terms[k];
     				myJ[1] += params[0]*fs[k]*anisoterm_lambda_PVM_single_c(dir_iter,params[1],xx,bvecs,bvals,ndirections);
     				myJ[kk] = 0;
      				for (int j=0;j<nfib;j++){
					if(f_deriv[j*nfib+k]!=0){
	  					//myJ[kk] += params[0]*(Aniso_terms[j]-Iso_term)*f_deriv[j*nfib+k]; 
						myJ[kk] += params[0]*(myJ[2+j*3+1]-Iso_term)*f_deriv[j*nfib+k]; 
					}
      				}
			}
			for(int k=0;k<nfib;k++){
     				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];		
      				myJ[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single_c(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
      				myJ[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single_c(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
    			}
    			if(m_include_f0){
				//partial_fsum ///////////
    				float partial_fsum=1.0;
    				for(int j=0;j<(nfib);j++)
					partial_fsum-=fs[j];
				//////////////////////////
				float temp_f0=beta2f_gpu(params[nparams-1])*partial_fsum;

    				//derivative with respect to f0
    				myJ[nparams-1]= params[0]*(1-Iso_term)*sin(float(2*params[nparams-1]))*partial_fsum; 
				sig=params[0]*((temp_f0+(1-*sumf-temp_f0)*Iso_term)+sig);
    				myJ[1] += params[0]*(1-*sumf-temp_f0)*isoterm_lambda_PVM_single_c(dir_iter,params[1],bvals);
    			}else{
				sig = params[0]*((1-*sumf)*Iso_term+sig);
	    			myJ[1] += params[0]*(1-*sumf)*isoterm_lambda_PVM_single_c(dir_iter,params[1],bvals);
    			}
    			diff = sig - mdata[dir_iter];
    			myJ[0] = sig/params[0]; 
		}

		for (int p=0;p<nparams;p++){ 
			reduction[idSubVOX]=2*myJ[p]*diff;

			__syncthreads();
			if(idSubVOX==0){
				for(int i=0;i<THREADS_BLOCK_FIT;i++){
					grad[p] += reduction[i];
				}
			}
			__syncthreads(); 
		} 
		dir_iter+=THREADS_BLOCK_FIT;
  	}
}


//hessian function PVM_single_c
__device__ void hess_PVM_single_c(	//INPUT
					const float*		params,
					const float*		bvecs, 
					const float*		bvals,
					const int 		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,		
					float*			J,		//shared memory
					float*			reduction,	//shared memory
					float* 			fs,		//shared memory
					float*			f_deriv,	//shared memory
					float*			x,		//shared memory
					float* 			_d,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					float*			hess)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}
	if(idSubVOX==0){
		*_d = lambda2d_gpu(params[1]);
		*sumf=0;
		float partial_fsum;
		for(int k=0;k<nfib;k++){
			int kk = 2+3*(k);
    			//partial_fsum ///////////
			partial_fsum=1.0;
			for(int j=0;j<k;j++)
				partial_fsum-=fs[j];
    			//////////////////////////
			fs[k] = beta2f_gpu(params[kk])*partial_fsum;
    			*sumf += fs[k];
		}
		for (int p=0;p<nparams;p++){
			for (int p2=0;p2<nparams;p2++){ 
				hess[p*nparams+p2] = 0;
			}
		}
	}

	__syncthreads();

  	if(idSubVOX<nfib){ 
		fractions_deriv_PVM_single_c(params,fs,nfib,idSubVOX,f_deriv); 
	} 

  	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	float* myJ = &J[idSubVOX*nparams];
  	float sig;
	float Iso_term;
	float3 xx;
	int dir_iter=idSubVOX;
  	//float Aniso_terms[MAXNFIBRES]; //reuse Shared J --- myJ[kk+1]

	__syncthreads();

  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) myJ[p]=0;
		if(dir<ndir){	
    			for(int k=0;k<nfib;k++){
				int kk = 2+3*(k) +1;
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];	
      				//Aniso_terms[k]=anisoterm_PVM_single_c(dir_iter,_d,xx,bvecs,bvals,ndirections);
				myJ[kk] = anisoterm_PVM_single_c(dir_iter,_d,xx,bvecs,bvals,ndirections);
    			}
			Iso_term=isoterm_PVM_single_c(dir_iter,_d,bvals);  //Precompute some terms for this datapoint
    			sig = 0;
    			for(int k=0;k<nfib;k++){
      				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];		 
      				sig += fs[k]*myJ[kk+1];//Aniso_terms[k];
      				myJ[1] += params[0]*fs[k]*anisoterm_lambda_PVM_single_c(dir_iter,params[1],xx,bvecs,bvals,ndirections);	 
      				for (int j=0; j<nfib; j++){
					if (f_deriv[j*nfib+k]!=0)
	  				//myJ[kk] += params[0]*(Aniso_terms[j]-Iso_term)*f_deriv[j*nfib+k]; 
					myJ[kk] += params[0]*(myJ[2+3*j+1]-Iso_term)*f_deriv[j*nfib+k]; 
      				}
			}
			for(int k=0;k<nfib;k++){
				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];
      				myJ[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single_c(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
      				myJ[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single_c(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
    			}
    			if(m_include_f0){
				//partial_fsum ///////////
	    			float partial_fsum=1.0;
	    			for(int j=0;j<(nfib);j++)
					partial_fsum-=fs[j];
	    			//////////////////////////
    				float temp_f0=beta2f_gpu(params[nparams-1])*partial_fsum;
    				//derivative with respect to f0
    				myJ[nparams-1]= params[0]*(1-Iso_term)*sin(float(2*params[nparams-1]))*partial_fsum; 
				sig= params[0]*((temp_f0+(1-*sumf-temp_f0)*Iso_term)+sig);
    				myJ[1] += params[0]*(1-*sumf-temp_f0)*isoterm_lambda_PVM_single_c(dir_iter,params[1],bvals);
    			}else{
	    			sig = params[0]*((1-*sumf)*Iso_term+sig);
	    			myJ[1] += params[0]*(1-*sumf)*isoterm_lambda_PVM_single_c(dir_iter,params[1],bvals);
    			}
    			myJ[0] = sig/params[0]; 
		}

		for (int p=0;p<nparams;p++){
			for (int p2=p;p2<nparams;p2++){ 

				reduction[idSubVOX]=2*(myJ[p]*myJ[p2]);
				__syncthreads();
				if(idSubVOX==0){
					for(int i=0;i<THREADS_BLOCK_FIT;i++){
						hess[p*nparams+p2] += reduction[i];
					}
				}
				__syncthreads(); 
			}
		}
		dir_iter+=THREADS_BLOCK_FIT;
  	}

	if(idSubVOX==0){
  		for (int j=0; j<nparams; j++) {
    			for (int i=j+1; i<nparams; i++) {
     				hess[i*nparams+j]=hess[j*nparams+i];	
    			}
  		}
	}
}
//in diffmodel.cc
extern "C" __global__ void fit_PVM_single_c_kernel(	//INPUT
							const float* 		data, 
							const float* 		bvecs, 
							const float* 		bvals, 
							const int 		nvox,
							const int		ndirections, 
							const int 		nfib,
							const int		nparams,
							const bool		m_eval_BIC,
							const bool 		m_include_f0,
							const bool	 	m_return_fanning,
							const bool		gradnonlin,
							//INPUT - OUTPUT
							float* 			params)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;
	int threadsBlock = blockDim.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	double* pcf = (double*) shared;					//1   
	double* ncf = (double*) &pcf[1];				//1   
	double* lambda = (double*) &ncf[1];				//1  
	double* cftol = (double*) &lambda[1];				//1  
	double* ltol = (double*) &cftol[1];				//1  
	double* olambda = (double*) &ltol[1];				//1  

	float* J = (float*)&olambda[1];					//threadsBlock*nparams
	float* reduction = (float*)&J[threadsBlock*nparams];		//threadsBlock
	float* myparams = (float*) &reduction[threadsBlock];		//nparams
	float* grad = (float*) &myparams[nparams];			//nparams      
   	float* hess = (float*) &grad[nparams];				//nparams*nparams   
	float* step = (float*) &hess[nparams*nparams];			//nparams      
	float* inverse = (float*) &step[nparams];			//nparams   

	float* fs = (float*) &inverse[nparams];				//nfib
	float* f_deriv = (float*) &fs[nfib];				//nfib*nfib
  	float* x = (float*) &f_deriv[nfib*nfib];			//nfib*3
	float* _d = (float*) &x[nfib*3];				//1
  	float* sumf = (float*) &_d[1];					//1

	float* C = (float*)&sumf[1];					//nparams*nparams;
	float* el =  (float*)&C[nparams*nparams];			//nparams

	int* indx = (int*)&el[nparams];					//nparams
	int* success = (int*) &indx[nparams];				//1
	int* end = (int*) &success[1];					//1   
	////////// DYNAMIC SHARED MEMORY ///////////

	if(idSubVOX<nparams){
		myparams[idSubVOX]=params[(idVOX*nparams)+idSubVOX];
	}

	__syncthreads();

	int pos_bvals, pos_bvecs;
	if(gradnonlin){ 
		pos_bvals=idVOX*ndirections;
		pos_bvecs=idVOX*3*ndirections;
	}else{ 
		pos_bvals=0;
		pos_bvecs=0;
	}
	//do the fit
	levenberg_marquardt_PVM_single_c_gpu(&data[idVOX*ndirections],&bvecs[pos_bvecs],&bvals[pos_bvals],ndirections,nfib,nparams,m_include_f0,idSubVOX,step,grad,hess,inverse, pcf,ncf,lambda,cftol,ltol,olambda,success,end,J,reduction,fs,f_deriv,x,_d,sumf,C,el,indx,myparams);

	__syncthreads();

	// finalise parameters
	// m_s0-myparams[0] 	m_d-myparams[1] 	m_f-m_th-m_ph-myparams[2,3,4,5, etc..]   	m_f0-myparams[nparams-1]
	
	if(idSubVOX==0){
  		myparams[1] = lambda2d_gpu(myparams[1]); 
  		for(int k=0;k<nfib;k++){
    			int kk = 2 + 3*(k);
    			//partial_fsum ///////////
			float partial_fsum=1.0;
			for(int j=0;j<k;j++)
				partial_fsum-=myparams[2 + 3*j];
    			//////////////////////////
    			myparams[kk]  = beta2f_gpu(myparams[kk])*partial_fsum;
  		}
  
  		if (m_include_f0){
			//partial_fsum ///////////
	    		float partial_fsum=1.0;
	    		for(int j=0;j<(nfib);j++){
				partial_fsum-=myparams[2 + 3*j];
			}
	    		//////////////////////////
    			myparams[nparams-1]= beta2f_gpu(myparams[nparams-1])*partial_fsum;
		}
		sort_PVM_single_c(nfib,myparams);
	}
	__syncthreads();

	if(idSubVOX<nparams){
		params[(idVOX*nparams)+idSubVOX] = myparams[idSubVOX];
	}
}

//in diffmodel.cc
extern "C" __global__ void get_residuals_PVM_single_c_kernel(	//INPUT
								const float* 		data, 
								const float* 		params,
								const float* 		bvecs, 
								const float* 		bvals, 
								const int 		nvox, 
								const int		ndirections,
								const int 		nfib, 
								const int		nparams,
								const bool 		m_include_f0,
								const bool		gradnonlin,
								const bool* 		includes_f0,								
								//OUTPUT
								float*			residuals)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;
	int threadsBlock = blockDim.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	float* myparams = (float*) shared;			//nparams
	float* fs = (float*) &myparams[nparams];		//nfib
  	float* x = (float*) &fs[nfib];				//nfib*3
	float* _d = (float*) &x[nfib*3];			//1
  	float* sumf = (float*) &_d[1];				//1
	int* my_include_f0 = (int*) &sumf[1];			//1		
	////////// DYNAMIC SHARED MEMORY ///////////
	
	float val; 
	float predicted_signal;
	float mydata;
	

	if(idSubVOX==0){
		*my_include_f0 = includes_f0[idVOX];

		//m_s0-myparams[0]  m_d-myparams[1] m_f-m_th-m_ph-myparams[2,3,4,5 etc..]  m_f0-myparams[nparams-1]
  		
  		myparams[0]=params[(idVOX*nparams)+0];
		if(myparams[1]<0)  myparams[1] = 0;	//This can be due to numerical errors..sqrt
  		else myparams[1] = d2lambda_gpu(params[(idVOX*nparams)+1]);

		float partial_fsum;	
  		for(int k=0;k<nfib;k++){
    			int kk = 2+3*k;
			//partial_fsum ///////////
			partial_fsum=1.0;
			for(int j=0;j<k;j++)
				partial_fsum-=fs[j];
	     		//////////////////////////
			fs[k] = params[(idVOX*nparams)+kk];
			float tmpr=fs[k]/partial_fsum;
    			if (tmpr>1.0) tmpr=1; //This can be due to numerical errors
			if (tmpr<0.0) tmpr=0; //This can be due to numerical errors..sqrt
    			myparams[kk]   = f2beta_gpu(tmpr);
    			myparams[kk+1] = params[(idVOX*nparams)+kk+1];
    			myparams[kk+2] = params[(idVOX*nparams)+kk+2];
  		}
  		if (*my_include_f0){
			//partial_fsum ///////////
			partial_fsum=1.0;
			for(int j=0;j<nfib;j++)
				partial_fsum-=fs[j];
	     		//////////////////////////	
			float tmpr=params[(idVOX*nparams)+nparams-1]/partial_fsum;
    			if (tmpr>1.0) tmpr=1; //This can be due to numerical errors..asin
			if (tmpr<0.0) tmpr=0; //This can be due to numerical errors..sqrt
    			myparams[nparams-1]= f2beta_gpu(tmpr);	
		}
	}

	__syncthreads();

	if(idSubVOX<nfib){
		int kk = 2+3*idSubVOX;
		float sinth,costh,sinph,cosph;
		sincos(myparams[kk+1],&sinth,&costh);
		sincos(myparams[kk+2],&sinph,&cosph);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	if(idSubVOX==0){
		float partial_fsum;	
		*sumf=0;
		for(int k=0;k<nfib;k++){
    			int kk = 2+3*k;
			////// partial_fsum //////
			partial_fsum=1.0;
			for(int j=0;j<k;j++)
				partial_fsum-=fs[j];
    			//////////////////////////
	    		fs[k] = beta2f_gpu(myparams[kk])*partial_fsum;
	    		*sumf += fs[k];
		}
		*_d = lambda2d_gpu(myparams[1]);
	}

	int ndir = ndirections/threadsBlock;
	if(idSubVOX<(ndirections%threadsBlock)) ndir++;
	
	float3 x2;
	int dir_iter=idSubVOX; 

	__syncthreads();

	int pos_bvals, pos_bvecs;
	if(gradnonlin){ 
		pos_bvals=idVOX*ndirections;
		pos_bvecs=idVOX*3*ndirections;
	}else{ 
		pos_bvals=0;
		pos_bvecs=0;
	}

	for(int dir=0;dir<ndir;dir++){
		mydata = data[(idVOX*ndirections)+dir_iter];
		predicted_signal=0;	//pred = 0;
		val = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
      			val += fs[k]*anisoterm_PVM_single_c(dir_iter,_d,x2,&bvecs[pos_bvecs],&bvals[pos_bvals],ndirections);
    		}	
    		if (*my_include_f0){
			//partial_fsum ///////////
			float partial_fsum=1.0;
			for(int j=0;j<nfib;j++)
				partial_fsum-=fs[j];
	     		//////////////////////////
      			float temp_f0= beta2f_gpu(myparams[nparams-1])*partial_fsum;
      			predicted_signal = myparams[0]*(temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single_c(dir_iter,_d,&bvals[pos_bvals])+val);
    		}else{
      			predicted_signal = myparams[0]*((1-*sumf)*isoterm_PVM_single_c(dir_iter,_d,&bvals[pos_bvals])+val); 
		}

		//residuals=m_data-predicted_signal;
		residuals[idVOX*ndirections+dir_iter]= mydata - predicted_signal;

		dir_iter+=threadsBlock;
	}
}
