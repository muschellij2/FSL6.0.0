#include "hip/hip_runtime.h"
/*  solver_mult_inverse.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include "options.h"

//X = A.i() * B . Used in Levenberg-Marquardt
//MATRIX INVERSE AS NEWMAT LU SOLVER
//implemented in NEWMAT:newmat7.cpp GeneralSolvI.
__device__ void solver(	//INPUT
			float *A, 
			float *P,
			int length,
			//TO USE
			float *C,
			float *el,
			int *indx,	
			//OUTPUT
			float *B)  
{  
	//double C[NPARAMS*NPARAMS];

	for(int i=0;i<length;i++){
		for(int j=0;j<length;j++){
			C[i*length+j]=A[i*length+j];
		}
	}
	
 	bool d=true; 
  	//int indx[NPARAMS];

   	float* akk = C;   
	float big = fabs(*akk); 
	int mu = 0; 
	float* ai = akk; 
	int k;

	for (k = 1; k<length; k++){
      		ai += length; 
		const float trybig = fabs(*ai);
      		if (big < trybig){ 
			big = trybig; 
			mu = k; 
		}
   	}

   	if(length) for (k = 0;;){

		indx[k] = mu;
		if (mu != k){
         		float* a1 = C + length*k; 
			float* a2 = C + length*mu; 
			d = !d;
         		int j = length;
         		while (j--){ 
				const float temp = *a1; 
				*a1++ = *a2; 
				*a2++ = temp; 
			}
      		}

      		float diag = *akk; 
		big = 0; 
		mu = k + 1;
      		if (diag != 0){
         		ai = akk; 
			int i = length - k - 1;
         		while (i--){
            			ai += length; 
				float* al = ai; 
				float mult = *al / diag; 
				*al = mult;
            			int l = length - k - 1; 
				float* aj = akk;
				if (l-- != 0){
				
					float aux=al[1]-(mult* *(++aj));
					*(++al) = aux;
					//*(++al) = __dadd_rn (*al,-mult* *(++aj)); //FAIL in cuda 4.2 compiler
					
               				const float trybig = fabs(*al);
               				if (big < trybig){ 
						big = trybig; 
						mu = length - i - 1; 
					}
               				while (l--){ 
						float aux= al[1]-(mult* *(++aj));
						*(++al) = aux;
						//*(++al) = __dadd_rn (*al,-mult* *(++aj)); //FAIL in cuda 4.2 compiler
					}
           			 }
         		}
      		}
      		if (++k == length) break;      
      		akk += length + 1;
   	}


//////////////////////////////

	//double el[NPARAMS];

	for(int e=0;e<length;e++){
		el[e]=P[e];		
    	}
		
   	int j;
	int ii = length; 
	int ip;    
	float temp;
	int i;
     
	for (i=0; i<length; i++){
 		ip = indx[i]; 
		temp = el[ip]; 
		el[ip] = el[i];
		el[i] = temp;
      		if (temp != 0.0) { ii = i; break; }
   	}
	
  	float* bi; 
	float* ai2;
   	i = ii + 1;

  	if (i < length){
      		bi = el + ii; 
		ai2 = C + ii + i * length;
      		for (;;){
         		int ip = indx[i]; 
			float sum = el[ip]; 
			el[ip] = el[i];
         		float* aij = ai2; 
			float* bj = bi; 
			j = i - ii;
         		while (j--){ 
				sum -=  *aij++* *bj++; 
			}
         		el[i] = sum;
         		if (++i == length) break;
         		ai2 += length;
      		}
   	}

   	ai2 = C + length*length;

   	for (i = length - 1; i >= 0; i--){
      		float* bj = el+i; 
		ai2 -= length; 
		float* ajx = ai2+i;
      		float sum = *bj; 
		float diag = *ajx;
      		j = length - i; 
		while(--j){ 
			sum -= *(++ajx)* *(++bj);  
		}
      		el[i] = sum / diag;
			
   	}
	for(int e=0;e<length;e++){
		B[e]=el[e];
    	}
}

