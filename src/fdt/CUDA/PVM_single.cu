#include "hip/hip_runtime.h"
/*  PVM_single.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include "diffmodels_utils.h"
#include "levenberg_marquardt.cu"
#include "options.h"

//#include <fstream>

/////////////////////////////////////
/////////////////////////////////////
/// 	    PVM_single		  /// 
/////////////////////////////////////
/////////////////////////////////////

__device__ 
inline float isoterm_PVM_single(const int pt,const float* _d,const float *bvals){
  	return exp(-bvals[pt]**_d);
}

__device__ 
inline float isoterm_d_PVM_single(const int pt,const float* _d,const float *bvals){
  	return (-bvals[pt]*exp(-bvals[pt]**_d));
}

__device__ 
inline float anisoterm_PVM_single(const int pt,const float* _d,const float3 x, const float *bvecs, const float *bvals, const int ndirections){
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	return exp(-bvals[pt]**_d*dp*dp);
}

__device__ 
inline float anisoterm_d_PVM_single(const int pt,const float* _d,const float3 x,const float *bvecs, const float *bvals, const int ndirections){
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
  	return(-bvals[pt]*dp*dp*exp(-bvals[pt]**_d*dp*dp));
}

__device__ 
inline float anisoterm_th_PVM_single(const int pt,const float* _d,const float3 x, const float _th,const float _ph,const float *bvecs, const float *bvals, const int ndirections){
	float sinth,costh,sinph,cosph;
	sincos(_th,&sinth,&costh);
	sincos(_ph,&sinph,&cosph);
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	float dp1 = (costh*(bvecs[pt]*cosph+bvecs[ndirections+pt]*sinph)-bvecs[(2*ndirections)+pt]*sinth);
  	return(-2*bvals[pt]**_d*dp*dp1*exp(-bvals[pt]**_d*dp*dp));
}

__device__ 
inline float anisoterm_ph_PVM_single(const int pt,const float* _d,const float3 x, const float _th,const float _ph,const float *bvecs, const float *bvals, const int ndirections){
	float sinth,sinph,cosph;
	sinth=sin(_th);
	sincos(_ph,&sinph,&cosph);
  	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	float dp1 = sinth*(-bvecs[pt]*sinph+bvecs[ndirections+pt]*cosph);
  	return(-2*bvals[pt]**_d*dp*dp1*exp(-bvals[pt]**_d*dp*dp));
}

//in diffmodel.cc
__device__ void fix_fsum_PVM_single(	//INPUT 
					bool m_include_f0, 
					int nfib,
					int nparams,
					//INPUT - OUTPUT){
					float *params)
{
  	float sum=0;
  	if (m_include_f0) 
    		sum=params[nparams-1];
  	for(int i=0;i<nfib;i++){
    		sum += params[2+(i*3)];
    		if(sum>=1){
			for(int j=i;j<nfib;j++)
				params[2+(j*3)]=FSMALL_gpu; 
			break;
		}
  	}
}

//in diffmodel.cc
__device__  void sort_PVM_single(int nfib,float* params)
{
	float temp_f, temp_th, temp_ph;
	// Order vector descending using f parameters as index
  	for(int i=1; i<(nfib); i++){ 
    		for(int j=0; j<(nfib-i); j++){ 
      			if (params[2+j*3] < params[2+(j+1)*3]){ 
        			temp_f = params[2+j*3];
				temp_th = params[2+j*3+1];
				temp_ph = params[2+j*3+2];
        			params[2+j*3] = params[2+(j+1)*3]; 
				params[2+j*3+1] = params[2+(j+1)*3+1]; 
				params[2+j*3+2] = params[2+(j+1)*3+2]; 
        			params[2+(j+1)*3] = temp_f; 
				params[2+(j+1)*3+1] = temp_th; 
				params[2+(j+1)*3+2] = temp_ph; 
      			} 
    		} 
  	} 
}

//cost function PVM_single
__device__ void cf_PVM_single(	//INPUT
				const float*		params,
				const float*		mdata,
				const float*		bvecs, 
				const float*		bvals,
				const int		ndirections,
				const int		nfib,
				const int 		nparams,
				const bool 		m_include_f0,
				const int		idSubVOX,
				float*			reduction,	//shared memory
				float* 			fs,		//shared memory
				float*			x,		//shared memory	
				float* 			_d,		//shared memory
				float* 			sumf,		//shared memory
				//OUTPUT
				double*			cfv)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		fs[idSubVOX] = x2f_gpu(params[kk]);
		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 	

	if(idSubVOX==0){
		*_d = abs(params[1]);
		*cfv = 0.0;
		*sumf=0;
		for(int k=0;k<nfib;k++) *sumf+= fs[k];
	}
	
	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	
	float err;
	float3 x2;
	int dir_iter=idSubVOX;

	__syncthreads();

	reduction[idSubVOX]=0;
	for(int dir=0;dir<ndir;dir++){
		err = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	
			err += fs[k]*anisoterm_PVM_single(dir_iter,_d,x2,bvecs,bvals,ndirections); 
    		}
		if(m_include_f0){
			float temp_f0=x2f_gpu(params[nparams-1]);
			err= (params[0]*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,bvals))+err))-mdata[dir_iter];
		}else{
			err =  (params[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,bvals)+err))-mdata[dir_iter];
		}
		reduction[idSubVOX]+= err*err;  
		dir_iter+=THREADS_BLOCK_FIT;
  	}  
	__syncthreads();

	if(idSubVOX==0){
		for(int i=0;i<THREADS_BLOCK_FIT;i++){
			*cfv+=reduction[i];
		}
	}	
}

//gradient function PVM_single
__device__ void grad_PVM_single(	//INPUT
					const float*		params,
					const float*		mdata,
					const float*		bvecs, 
					const float*		bvals,
					const int 		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,	
					float*			J,		//shared memory	
					float*			reduction,	//shared memory
					float* 			fs,		//shared memory
					float*			x,		//shared memory
					float* 			_d,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					float*			grad)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		fs[idSubVOX] = x2f_gpu(params[kk]);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 

	if(idSubVOX==0){
		*sumf=0;
		for(int i=0;i<nfib;i++) *sumf+=fs[i];
		*_d = abs(params[1]);
		for (int p=0;p<nparams;p++) grad[p]=0;
	}

	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	float* myJ = &J[idSubVOX*nparams];
	float diff;
  	float sig;
	float3 xx;
	int dir_iter=idSubVOX;

	__syncthreads();

  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) myJ[p]=0;
		if(dir<ndir){
    			sig = 0;
    			for(int k=0;k<nfib;k++){
      				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];			
				sig +=  fs[k]*anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
				myJ[1] +=  (params[1]>0?1.0:-1.0)*params[0]*fs[k]*anisoterm_d_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
      				myJ[kk] = params[0]*(anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections)-isoterm_PVM_single(dir_iter,_d,bvals)) * two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);
      				myJ[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
      				myJ[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
    			}

    			if(m_include_f0){
				float temp_f0=x2f_gpu(params[nparams-1]);
				myJ[nparams-1]= params[0]*(1-isoterm_PVM_single(dir_iter,_d,bvals))* two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
				sig= params[0]*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,bvals))+sig);
    				myJ[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf-temp_f0)*isoterm_d_PVM_single(dir_iter,_d,bvals);
    			}else{
				sig = params[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,bvals)+sig);
				myJ[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf)*isoterm_d_PVM_single(dir_iter,_d,bvals);
    			}
    			diff = sig - mdata[dir_iter];
    			myJ[0] = sig/params[0];
		}

		for (int p=0;p<nparams;p++){ 
			reduction[idSubVOX]=2*myJ[p]*diff;

			__syncthreads();
			if(idSubVOX==0){
				for(int i=0;i<THREADS_BLOCK_FIT;i++){
					grad[p] += reduction[i];
				}
			}
			__syncthreads(); 
		} 
		dir_iter+=THREADS_BLOCK_FIT;
  	}
}

//hessian function PVM_single
__device__ void hess_PVM_single(	//INPUT
					const float*		params,
					const float*		bvecs, 
					const float*		bvals,
					const int		ndirections,
					const int 		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,
					float*			J,		//shared memory
					float*			reduction,	//shared memory					
					float* 			fs,		//shared memory
					float*			x,		//shared memory
					float* 			_d,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					float*			hess)
{
	if(idSubVOX<nfib){
		int kk = 2+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
    		fs[idSubVOX] = x2f_gpu(params[kk]);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 

	if(idSubVOX==0){
		*sumf=0;
		for(int i=0;i<nfib;i++) *sumf+=fs[i];
		*_d = abs(params[1]);
		for (int p=0;p<nparams;p++){
			for (int p2=0;p2<nparams;p2++){ 
				hess[p*nparams+p2] = 0;
			}
		}
	}

	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	float* myJ = &J[idSubVOX*nparams];
  	float sig;
	float3 xx;
	int dir_iter=idSubVOX; 

	__syncthreads(); 
	
  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) myJ[p]=0;
		if(dir<ndir){
    			sig = 0;
    			for(int k=0;k<nfib;k++){
      				int kk = 2+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];		
				sig += fs[k]*anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
      				myJ[1] += (params[1]>0?1.0:-1.0)*params[0]*fs[k]*anisoterm_d_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections);
      				myJ[kk] = params[0]*(anisoterm_PVM_single(dir_iter,_d,xx,bvecs,bvals,ndirections)-isoterm_PVM_single(dir_iter,_d,bvals)) * two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);
		      		myJ[kk+1] = params[0]*fs[k]*anisoterm_th_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
		      		myJ[kk+2] = params[0]*fs[k]*anisoterm_ph_PVM_single(dir_iter,_d,xx,params[kk+1],params[kk+2],bvecs,bvals,ndirections);
    			}	

    			if(m_include_f0){
				float temp_f0=x2f_gpu(params[nparams-1]);
				myJ[nparams-1]= params[0]*(1-isoterm_PVM_single(dir_iter,_d,bvals))* two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
				sig=params[0]*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,bvals))+sig);
    				myJ[1] += (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf-temp_f0)*isoterm_d_PVM_single(dir_iter,_d,bvals);	
    			}else{
				sig = params[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,bvals)+sig);
	    			myJ[1] +=  (params[1]>0?1.0:-1.0)*params[0]*(1-*sumf)*isoterm_d_PVM_single(dir_iter,_d,bvals);
    			}   
    			myJ[0] = sig/params[0];
		}

		for (int p=0;p<nparams;p++){
			for (int p2=p;p2<nparams;p2++){ 

				reduction[idSubVOX]=2*(myJ[p]*myJ[p2]);
				__syncthreads();
				if(idSubVOX==0){
					for(int i=0;i<THREADS_BLOCK_FIT;i++){
						hess[p*nparams+p2] += reduction[i];
					}
				}
				__syncthreads(); 
			}
		}
		dir_iter+=THREADS_BLOCK_FIT;
  	}

	if(idSubVOX==0){
	  	for (int j=0; j<nparams; j++) {
	    		for (int i=j+1; i<nparams; i++) {
	     			hess[i*nparams+j]=hess[j*nparams+i];	
	    		}
	  	}
	}
}

//in diffmodel.cc
extern "C" __global__ void fit_PVM_single_kernel(	//INPUT
							const float* 		data, 
							const float* 		bvecs,
							const float* 		bvals, 
							const int 		nvox, 
							const int		ndirections,
							const int 		nfib, 
							const int		nparams,
							const bool 		m_include_f0,
							const bool		gradnonlin,
							//INPUT-OUTPUT
							float* 		params)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;
	int threadsBlock = blockDim.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	double* pcf = (double*) shared;					//1   
	double* ncf = (double*) &pcf[1];				//1   
	double* lambda = (double*) &ncf[1];				//1  
	double* cftol = (double*) &lambda[1];				//1  
	double* ltol = (double*) &cftol[1];				//1  
	double* olambda = (double*) &ltol[1];				//1  

	float* J = (float*)&olambda[1];					//threadsBlock*nparams
	float* reduction = (float*)&J[threadsBlock*nparams];		//threadsBlock
	float* myparams = (float*) &reduction[threadsBlock];		//nparams
	float* grad = (float*) &myparams[nparams];			//nparams      
   	float* hess = (float*) &grad[nparams];				//nparams*nparams   
	float* step = (float*) &hess[nparams*nparams];			//nparams      
 	float* inverse = (float*) &step[nparams];			//nparams   

	float* fs = (float*) &inverse[nparams];				//nfib
  	float* x = (float*) &fs[nfib];					//nfib*3
	float* _d = (float*) &x[nfib*3];				//1
  	float* sumf = (float*) &_d[1];					//1

	float* C = (float*)&sumf[1];					//nparams*nparams;
	float* el =  (float*)&C[nparams*nparams];			//nparams

	int* indx = (int*)&el[nparams];					//nparams
	int* success = (int*) &indx[nparams];				//1
	int* end = (int*) &success[1];					//1    
	////////// DYNAMIC SHARED MEMORY ///////////

	if(idSubVOX<nparams){
		myparams[idSubVOX]=params[(idVOX*nparams)+idSubVOX];
	}

	__syncthreads();

	int pos_bvals, pos_bvecs;
	if(gradnonlin){ 
		pos_bvals=idVOX*ndirections;
		pos_bvecs=idVOX*3*ndirections;
	}else{ 
		pos_bvals=0;
		pos_bvecs=0;
	}
	// do the fit
	levenberg_marquardt_PVM_single_gpu(&data[idVOX*ndirections],&bvecs[pos_bvecs],&bvals[pos_bvals],ndirections,nfib,nparams,m_include_f0,idSubVOX,step,grad,hess,inverse, pcf,ncf,lambda,cftol,ltol,olambda,success,end,J,reduction,fs,x,_d,sumf,C,el,indx,myparams);

	__syncthreads();
	
  	// finalise parameters
	//m_s0 in myparams[0] 	m_d in myparams[1] 	m_f-m_th-m_ph in myparams[2,3,4,5, etc..]   	m_f0 in myparams[nparams-1]

	if(idSubVOX==0){  	
  		myparams[1] = abs(myparams[1]); 
  		for(int k=1;k<=nfib;k++){
    			int kk = 2 + 3*(k-1);
    			myparams[kk] = x2f_gpu(myparams[kk]);
  		}
  		if(m_include_f0)
    			myparams[nparams-1]=x2f_gpu(myparams[nparams-1]);

  		sort_PVM_single(nfib,myparams);
  		fix_fsum_PVM_single(m_include_f0,nfib,nparams,myparams);
	}
	__syncthreads();

	if(idSubVOX<nparams){
		params[idVOX*nparams+idSubVOX]=myparams[idSubVOX];	
	}
}

//in diffmodel.cc
extern "C" __global__ void get_residuals_PVM_single_kernel(	//INPUT
								const float* 		data, 
								const float* 		params,
								const float* 		bvecs, 
								const float* 		bvals, 
								const int 		nvox, 
								const int		ndirections,
								const int 		nfib, 
								const int		nparams,
								const bool 		m_include_f0,
								const bool		gradnonlin,
								const bool* 		includes_f0,
								//OUTPUT
								float*			residuals)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;
	int threadsBlock = blockDim.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	float* myparams = (float*) shared;			//nparams
	float* fs = (float*) &myparams[nparams];		//nfib
  	float* x = (float*) &fs[nfib];				//nfib*3
	float* _d = (float*) &x[nfib*3];			//1
  	float* sumf = (float*) &_d[1];				//1
	int* my_include_f0 = (int*) &sumf[1];			//1	
	////////// DYNAMIC SHARED MEMORY ///////////

	float val;
	float predicted_signal;
	float mydata;

	if(idSubVOX==0){
		*my_include_f0 = includes_f0[idVOX];

		//m_s0-myparams[0]  m_d-myparams[1]  m_f-m_th-m_ph-myparams[2,3,4,5 etc..]  m_f0-myparams[nparams-1]

		myparams[0]=params[(idVOX*nparams)+0];
		myparams[1]=params[(idVOX*nparams)+1];

  		if (*my_include_f0)
    			myparams[nparams-1]=f2x_gpu(params[(idVOX*nparams)+nparams-1]);
	}

	if(idSubVOX<nfib){
		int kk = 2+3*idSubVOX;
		float sinth,costh,sinph,cosph;

		myparams[kk]   = f2x_gpu(params[(idVOX*nparams)+kk]);
    		myparams[kk+1] = params[(idVOX*nparams)+kk+1];
    		myparams[kk+2] = params[(idVOX*nparams)+kk+2];

		sincos(myparams[kk+1],&sinth,&costh);
		sincos(myparams[kk+2],&sinph,&cosph);

    		fs[idSubVOX] = x2f_gpu(myparams[kk]);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 

	if(idSubVOX==0){
		*sumf=0;
		for(int i=0;i<nfib;i++) *sumf+=fs[i];
		*_d = abs(myparams[1]);
	}
  	
  	int ndir = ndirections/threadsBlock;
	if(idSubVOX<(ndirections%threadsBlock)) ndir++;
	
	float3 x2;
	int dir_iter=idSubVOX; 

	__syncthreads();

	int pos_bvals, pos_bvecs;
	if(gradnonlin){ 
		pos_bvals=idVOX*ndirections;
		pos_bvecs=idVOX*3*ndirections;
	}else{ 
		pos_bvals=0;
		pos_bvecs=0;
	}

	for(int dir=0;dir<ndir;dir++){
		mydata = data[(idVOX*ndirections)+dir_iter];
  		predicted_signal=0;	//pred = 0;
    		val = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
      			val += fs[k]*anisoterm_PVM_single(dir_iter,_d,x2,&bvecs[pos_bvecs],&bvals[pos_bvals],ndirections);
    		}	
    		if (*my_include_f0){
      			float temp_f0=x2f_gpu(myparams[nparams-1]);
      			predicted_signal = myparams[0]*(temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_single(dir_iter,_d,&bvals[pos_bvals])+val);
    		}else{
      			predicted_signal = myparams[0]*((1-*sumf)*isoterm_PVM_single(dir_iter,_d,&bvals[pos_bvals])+val); 
		}
	
		//residuals=m_data-predicted_signal;
		residuals[idVOX*ndirections+dir_iter]= mydata - predicted_signal;

		dir_iter+=threadsBlock;
  	}
}

