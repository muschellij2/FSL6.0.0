#include "hip/hip_runtime.h"
/*  PVM_multi.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include "diffmodels_utils.h"
#include "levenberg_marquardt.cu"
#include "options.h"

/////////////////////////////////////
/////////////////////////////////////
/// 	    PVM_multi	 	  /// 
/////////////////////////////////////
/////////////////////////////////////

__device__ inline float isoterm_PVM_multi(const int pt,const float* _a,const float* _b, const float *bvals){
	return exp(-*_a*log(1+bvals[pt]**_b));
}

__device__ inline float isoterm_a_PVM_multi(const int pt,const float* _a,const float* _b, const float *bvals){
    	return  -log(1+bvals[pt]**_b)*exp(-*_a*log(1+bvals[pt]**_b));
}

__device__ inline float isoterm_b_PVM_multi(const int pt,const float* _a,const float* _b, const float *bvals){
      	return -*_a*bvals[pt]/(1+bvals[pt]**_b)*exp(-*_a*log(1+bvals[pt]**_b));
}

__device__ inline float anisoterm_PVM_multi(const int pt,const float* _a,const float* _b,const float3 x,const float *bvecs, const float *bvals, const float R, const float invR, const int ndirections,const int Gamma_for_ball_only){
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	if(Gamma_for_ball_only==1){
		return exp(-bvals[pt]**_a**_b*dp*dp);
	}else if(Gamma_for_ball_only==2){
		return exp(-bvals[pt]*3**_a**_b*invR*((1-R)*dp*dp+R));		
	}else{
  		return exp(-*_a*log(1+bvals[pt]**_b*(dp*dp)));
	}
}
 
__device__ inline float anisoterm_a_PVM_multi(const int pt,const float* _a,const float* _b,const float3 x,const float *bvecs, const float *bvals, const float R, const float invR, const int ndirections,const int Gamma_for_ball_only){
	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	if(Gamma_for_ball_only==1){
		return (-bvals[pt]**_b*dp*dp* exp(-bvals[pt]**_a**_b*dp*dp));
  	}else if(Gamma_for_ball_only==2){
		float dp2=bvals[pt]*3**_b*invR*((1-R)*dp*dp+R);
		return(-dp2*exp(-dp2**_a));
	}else{
		return -log(1+bvals[pt]*(dp*dp)**_b)* exp(-*_a*log(1+bvals[pt]*(dp*dp)**_b));
  	}
  	
}

__device__ inline float anisoterm_b_PVM_multi(const int pt,const float* _a,const float* _b,const float3 x,const float *bvecs, const float *bvals, const float R, const float invR, const int ndirections,const int Gamma_for_ball_only){
  	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
	if(Gamma_for_ball_only==1){
		return(-bvals[pt]**_a*dp*dp*exp(-bvals[pt]**_a**_b*dp*dp));
  	}else if(Gamma_for_ball_only==2){
		float dp2=bvals[pt]*3**_a*invR*((1-R)*dp*dp+R);
		return(-dp2*exp(-dp2**_b));
  	}else{
		return (-*_a*bvals[pt]*(dp*dp)/ (1+bvals[pt]*(dp*dp)**_b)*exp(-*_a*log(1+bvals[pt]*(dp*dp)**_b)));
  	}
}

__device__ inline float anisoterm_th_PVM_multi(const int pt,const float* _a,const float* _b,const float3 x,const float _th,const float _ph,const float *bvecs, const float *bvals, const float R, const float invR, const int ndirections,const int Gamma_for_ball_only){
	float sinth,costh,sinph,cosph;
	sincos(_th,&sinth,&costh);
	sincos(_ph,&sinph,&cosph);
  	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
  	float dp1 = costh* (bvecs[pt]*cosph + bvecs[ndirections+pt]*sinph) - bvecs[(2*ndirections)+pt]*sinth;
	if(Gamma_for_ball_only==1){
  		return(-2*bvals[pt]**_a**_b*dp*dp1*exp(-bvals[pt]**_a**_b*dp*dp));
  	}else if(Gamma_for_ball_only==2){
		float dp2=2*bvals[pt]*3**_a**_b*invR*(1-R)*dp1;
		return(-dp2*exp(-bvals[pt]*3**_a**_b*invR*((1-R)*dp*dp+R)));
  	}else{
		return  (-*_a**_b*bvals[pt]/(1+bvals[pt]*(dp*dp)**_b)*exp(-*_a*log(1+bvals[pt]*(dp*dp)**_b))*2*dp*dp1);	
	}
}

__device__ inline float anisoterm_ph_PVM_multi(const int pt,const float* _a,const float* _b,const float3 x,const float _th,const float _ph,const float *bvecs, const float *bvals, const float R, const float invR, const int ndirections,const int Gamma_for_ball_only){
	float sinth,sinph,cosph;
	sinth=sin(_th);
	sincos(_ph,&sinph,&cosph);
  	float dp = bvecs[pt]*x.x+bvecs[ndirections+pt]*x.y+bvecs[(2*ndirections)+pt]*x.z;
  	float dp1 = sinth* (-bvecs[pt]*sinph + bvecs[ndirections+pt]*cosph);
	if(Gamma_for_ball_only==1){
  		return(-2*bvals[pt]**_a**_b*dp*dp1*exp(-bvals[pt]**_a**_b*dp*dp));
  	}else if(Gamma_for_ball_only==2){
		float dp2=2*bvals[pt]*3**_a**_b*invR*(1-R)*dp1;
		return(-dp2*exp(-bvals[pt]*3**_a**_b*invR*((1-R)*dp*dp+R)));
 	}else{
		return  (-*_a**_b*bvals[pt]/(1+bvals[pt]*(dp*dp)**_b)*exp(-*_a*log(1+bvals[pt]*(dp*dp)**_b))*2*dp*dp1);
  	}
}

//in diffmodel.cc
__device__ void fix_fsum_PVM_multi(	//INPUT 
					bool m_include_f0, 
					int nfib,
					int nparams,
					//INPUT - OUTPUT){
					float *params)
{
  	float sumf=0;
  	if (m_include_f0) 
    		sumf=params[nparams-1];
  	for(int i=0;i<nfib;i++){
    		if (params[3+(i*3)]==0) 
			params[3+(i*3)]=FSMALL_gpu;
    		sumf+=params[3+(i*3)];
    		if(sumf>=1){
			for(int j=i;j<nfib;j++)
				params[3+(j*3)]=FSMALL_gpu;
			break;
		}
	}
}

//in diffmodel.cc
__device__ void sort_PVM_multi(int nfib,float* params)
{
	float temp_f, temp_th, temp_ph;
	// Order vector descending using f parameters as index
	for(int i=1; i<(nfib); i++){ 
    		for(int j=0; j<(nfib-i); j++){ 
      			if (params[3+j*3] < params[3+(j+1)*3]){ 
        			temp_f = params[3+j*3];
				temp_th = params[3+j*3+1];
				temp_ph = params[3+j*3+2];
        			params[3+j*3] = params[3+(j+1)*3]; 
				params[3+j*3+1] = params[3+(j+1)*3+1]; 
				params[3+j*3+2] = params[3+(j+1)*3+2]; 
        			params[3+(j+1)*3] = temp_f; 
				params[3+(j+1)*3+1] = temp_th; 
				params[3+(j+1)*3+2] = temp_ph; 
      			} 
    		} 
  	} 
}

//cost function PVM_multi
__device__ void cf_PVM_multi(		//INPUT
					const float*		params,
					const float*		mdata,
					const float*		bvecs, 
					const float*		bvals,
					const float		R,
					const float		invR,
					const int		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,
					const int		Gamma_for_ball_only,
					float*			reduction,	//shared memory
					float* 			fs,		//shared memory
					float*			x,		//shared memory	
					float* 			_a,		//shared memory
					float* 			_b,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					double*			cfv)
{
	if(idSubVOX<nfib){
		int kk = 3+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}
	if(idSubVOX==0){
		*_a= abs(params[1]);
		*_b= abs(params[2]); 
		*cfv = 0.0;
		*sumf=0;
		for(int k=0;k<nfib;k++){
			fs[k] = x2f_gpu(params[3+3*k]);
		 	*sumf+= fs[k];
		}
	}

	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	
	float err;
	float3 x2;
	int dir_iter=idSubVOX;

	__syncthreads();
	
	reduction[idSubVOX]=0;
	for(int dir=0;dir<ndir;dir++){
    		err = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
			err += fs[k]*anisoterm_PVM_multi(dir_iter,_a,_b,x2,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only); 
    		}
		if(m_include_f0){
			float temp_f0=x2f_gpu(params[nparams-1]);
			err = (abs(params[0])*(temp_f0+((1-*sumf-temp_f0)*isoterm_PVM_multi(dir_iter,_a,_b,bvals)+err)))-mdata[dir_iter];
		}else{
			err = abs(params[0])*((1-*sumf)*isoterm_PVM_multi(dir_iter,_a,_b,bvals)+err)-mdata[dir_iter];
		}
		reduction[idSubVOX]+= err*err;  
		dir_iter+=THREADS_BLOCK_FIT;
  	}  

	__syncthreads();

	if(idSubVOX==0){
		for(int i=0;i<THREADS_BLOCK_FIT;i++){
			*cfv+=reduction[i];
		}
	}	
}

//gradient function PVM_multi
__device__ void grad_PVM_multi(		//INPUT
					const float*		params,
					const float*		mdata,
					const float*		bvecs, 
					const float*		bvals,
					const float		R,
					const float		invR,
					const int		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,
					const int		Gamma_for_ball_only,
					float*			J,		//shared memory
					float*			reduction,	//shared memory
					float* 			fs,		//shared memory
					float*			x,		//shared memory	
					float* 			_a,		//shared memory
					float* 			_b,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					float*			grad)
{	
	if(idSubVOX<nfib){
		int kk = 3+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}
	if(idSubVOX==0){
		*_a= abs(params[1]);
		*_b= abs(params[2]); 
		*sumf=0;
		for(int k=0;k<nfib;k++){
			fs[k] = x2f_gpu(params[3+3*k]);
	 		*sumf+= fs[k];
		}
		for (int p=0;p<nparams;p++) grad[p]=0;
	}

  	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	float* myJ = &J[idSubVOX*nparams];
	float diff;
  	float sig;
	float3 xx;
	int dir_iter=idSubVOX;

	__syncthreads();

  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) myJ[p]=0;
		if(dir<ndir){
    			sig = 0;
    			for(int k=0;k<nfib;k++){
      				int kk = 3+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];		
      				sig += fs[k]*anisoterm_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);

      				myJ[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*fs[k]*
				anisoterm_a_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only); 

				myJ[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*fs[k]*
				anisoterm_b_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);

				myJ[kk] = abs(params[0])*(anisoterm_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only)
				-isoterm_PVM_multi(dir_iter,_a,_b,bvals))*two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]); 

      				myJ[kk+1] = abs(params[0])*fs[k]*
				anisoterm_th_PVM_multi(dir_iter,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);  

      				myJ[kk+2] = abs(params[0])*fs[k]*
				anisoterm_ph_PVM_multi(dir_iter,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);
    			}
    			if(m_include_f0){
				float temp_f0=x2f_gpu(params[nparams-1]);
				myJ[nparams-1]= abs(params[0])*(1-isoterm_PVM_multi(dir_iter,_a,_b,bvals))*
				two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);

				sig=abs(params[0])*((temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_multi(dir_iter,_a,_b,bvals))+sig);
    				myJ[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-*sumf-temp_f0)*isoterm_a_PVM_multi(dir_iter,_a,_b,bvals);
				myJ[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-*sumf-temp_f0)*isoterm_b_PVM_multi(dir_iter,_a,_b,bvals);
    			}else{
	    			sig = abs(params[0]) * ((1-*sumf)*isoterm_PVM_multi(dir_iter,_a,_b,bvals)+sig);
	    			myJ[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-*sumf)*isoterm_a_PVM_multi(dir_iter,_a,_b,bvals);
	    			myJ[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-*sumf)*isoterm_b_PVM_multi(dir_iter,_a,_b,bvals);	
    			}
    
    			diff = sig - mdata[dir_iter];
    			myJ[0] = (params[0]>0?1.0:-1.0)*sig/params[0]; 
		}

		for (int p=0;p<nparams;p++){ 
			reduction[idSubVOX]=2*myJ[p]*diff;

			__syncthreads();
			if(idSubVOX==0){
				for(int i=0;i<THREADS_BLOCK_FIT;i++){
					grad[p] += reduction[i];
				}
			}
			__syncthreads(); 
		} 
		dir_iter+=THREADS_BLOCK_FIT;
  	}
}

//hessian function PVM_multi 
__device__ void hess_PVM_multi(		//INPUT
					const float*		params,
					const float*		bvecs, 
					const float*		bvals,
					const float		R,
					const float		invR,
					const int 		ndirections,
					const int		nfib,
					const int 		nparams,
					const bool 		m_include_f0,
					const int		idSubVOX,
					const int		Gamma_for_ball_only,
					float*			J,		//shared memory
					float*			reduction,	//shared memory
					float* 			fs,		//shared memory
					float*			x,		//shared memory	
					float* 			_a,		//shared memory
					float* 			_b,		//shared memory
					float* 			sumf,		//shared memory
					//OUTPUT
					float*			hess)
{
	if(idSubVOX<nfib){
		int kk = 3+3*(idSubVOX);
		float sinth,costh,sinph,cosph;
		sincos(params[kk+1],&sinth,&costh);
		sincos(params[kk+2],&sinph,&cosph);
		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}
	if(idSubVOX==0){
		*_a= abs(params[1]);
		*_b= abs(params[2]); 
		*sumf=0;
		for(int k=0;k<nfib;k++){
			fs[k] = x2f_gpu(params[3+3*k]);
			*sumf+= fs[k];
		}
		for (int p=0;p<nparams;p++){
			for (int p2=0;p2<nparams;p2++){ 
				hess[p*nparams+p2] = 0;
			}
		}
	}

  	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	int max_dir = ndirections/THREADS_BLOCK_FIT;
	if(ndirections%THREADS_BLOCK_FIT) max_dir++;

	float* myJ = &J[idSubVOX*nparams];
  	float sig;
	float3 xx;
	int dir_iter=idSubVOX; 

	__syncthreads(); 
	
  	for(int dir=0;dir<max_dir;dir++){
		for (int p=0; p<nparams; p++) myJ[p]=0;
		if(dir<ndir){
    			sig = 0;
    			for(int k=0;k<nfib;k++){
      				int kk = 3+3*(k);
      				xx.x=x[k*3];
      				xx.y=x[k*3+1];
      				xx.z=x[k*3+2];		
      				sig += fs[k]*anisoterm_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);

      				float cov = two_pi_gpu*sign_gpu(params[kk])*1/(1+params[kk]*params[kk]);	
      				myJ[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*fs[k]*
				anisoterm_a_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);

				myJ[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*fs[k]*
				anisoterm_b_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);

				myJ[kk] = abs(params[0])*
				(anisoterm_PVM_multi(dir_iter,_a,_b,xx,bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only)-
				isoterm_PVM_multi(dir_iter,_a,_b,bvals))*cov;

      				myJ[kk+1] = abs(params[0])*fs[k]*
				anisoterm_th_PVM_multi(dir_iter,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);

      				myJ[kk+2] = abs(params[0])*fs[k]*
				anisoterm_ph_PVM_multi(dir_iter,_a,_b,xx,params[kk+1],params[kk+2],bvecs,bvals,R,invR,ndirections,Gamma_for_ball_only);
    			}
    			if(m_include_f0){
				float temp_f0=x2f_gpu(params[nparams-1]);
				myJ[nparams-1]= abs(params[0])*(1-isoterm_PVM_multi(dir_iter,_a,_b,bvals))*two_pi_gpu*sign_gpu(params[nparams-1])*1/(1+params[nparams-1]*params[nparams-1]);
	    			sig = abs(params[0])* (temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_multi(dir_iter,_a,_b,bvals)+sig);
    				myJ[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-*sumf-temp_f0)*isoterm_a_PVM_multi(dir_iter,_a,_b,bvals);
				myJ[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-*sumf-temp_f0)*isoterm_b_PVM_multi(dir_iter,_a,_b,bvals);
    			}else{
				sig = abs(params[0])*((1-*sumf)*isoterm_PVM_multi(dir_iter,_a,_b,bvals)+sig);
	    			myJ[1] += (params[1]>0?1.0:-1.0)*abs(params[0])*(1-*sumf)*isoterm_a_PVM_multi(dir_iter,_a,_b,bvals);
	    			myJ[2] += (params[2]>0?1.0:-1.0)*abs(params[0])*(1-*sumf)*isoterm_b_PVM_multi(dir_iter,_a,_b,bvals);	
    			}
	
    			myJ[0] = sig/params[0]; 
		}

		for (int p=0;p<nparams;p++){
			for (int p2=p;p2<nparams;p2++){ 

				reduction[idSubVOX]=2*(myJ[p]*myJ[p2]);
				__syncthreads();
				if(idSubVOX==0){
					for(int i=0;i<THREADS_BLOCK_FIT;i++){
						hess[p*nparams+p2] += reduction[i];
					}
				}
				__syncthreads(); 
			}
		}
		dir_iter+=THREADS_BLOCK_FIT;
  	}

	if(idSubVOX==0){
  		for (int j=0; j<nparams; j++) {
    			for (int i=j+1; i<nparams; i++) {
     				hess[i*nparams+j]=hess[j*nparams+i];	
    			}
  		}
	}
}

//in diffmodel.cc
extern "C" __global__ void fit_PVM_multi_kernel(	//INPUT
							const float* 		data, 
							const float* 		params_PVM_single_c,
							const float* 		bvecs, 
							const float* 		bvals, 
							const float		R,
							const float		invR,
							const int 		nvox, 
							const int		ndirections,
							const int 		nfib, 	
							const int		nparams,
							const int		Gamma_for_ball_only,			
							const bool 		m_include_f0,
							const bool		gradnonlin,
							//OUTPUT
							float* 			params)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;
	int threadsBlock = blockDim.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	double* pcf = (double*) shared;					//1   
	double* ncf = (double*) &pcf[1];				//1   
	double* lambda = (double*) &ncf[1];				//1  
	double* cftol = (double*) &lambda[1];				//1  
	double* ltol = (double*) &cftol[1];				//1  
	double* olambda = (double*) &ltol[1];				//1  

	float* J = (float*)&olambda[1];					//threadsBlock*nparams
	float* reduction = (float*)&J[threadsBlock*nparams];		//threadsBlock
	float* myparams = (float*) &reduction[threadsBlock];		//nparams
	float* grad = (float*) &myparams[nparams];			//nparams      
   	float* hess = (float*) &grad[nparams];				//nparams*nparams   
	float* step = (float*) &hess[nparams*nparams];			//nparams      
 	float* inverse = (float*) &step[nparams];			//nparams   

	float* fs = (float*) &inverse[nparams];				//nfib
  	float* x = (float*) &fs[nfib];					//nfib*3
	float* _a = (float*) &x[nfib*3];				//1
	float* _b = (float*) &_a[1];					//1
  	float* sumf = (float*) &_b[1];					//1

	float* C = (float*)&sumf[1];					//nparams*nparams;
	float* el =  (float*)&C[nparams*nparams];			//nparams

	int* indx = (int*)&el[nparams];					//nparams
	int* success = (int*) &indx[nparams];				//1
	int* end = (int*) &success[1];					//1   
	////////// DYNAMIC SHARED MEMORY ///////////

	if(idSubVOX==0){
		
		int nparams_single_c = nparams-1;

		myparams[0] = params_PVM_single_c[(idVOX*nparams_single_c)+0];			//pvm1.get_s0();
  		myparams[1] = 1.0;								//start with d=d_std
  		for(int i=0,ii=3;i<nfib;i++,ii+=3){
    			myparams[ii] = f2x_gpu(params_PVM_single_c[(idVOX*nparams_single_c)+ii-1]);
    			myparams[ii+1] = params_PVM_single_c[(idVOX*nparams_single_c)+ii];
    			myparams[ii+2] = params_PVM_single_c[(idVOX*nparams_single_c)+ii+1];
  		}
		myparams[2] = params_PVM_single_c[(idVOX*nparams_single_c)+1] ; 		//pvm1.get_d();
  		if (m_include_f0)
			myparams[nparams-1]=f2x_gpu(params_PVM_single_c[(idVOX*nparams_single_c)+nparams_single_c-1]);
	}

	__syncthreads();

	int pos_bvals, pos_bvecs;
	if(gradnonlin){ 
		pos_bvals=idVOX*ndirections;
		pos_bvecs=idVOX*3*ndirections;
	}else{ 
		pos_bvals=0;
		pos_bvecs=0;
	}
  	//do the fit
	levenberg_marquardt_PVM_multi_gpu(&data[idVOX*ndirections],&bvecs[pos_bvecs],&bvals[pos_bvals],R,invR, 
	ndirections,nfib,nparams,m_include_f0,idSubVOX,Gamma_for_ball_only,
	step,grad,hess,inverse, pcf,ncf,lambda,cftol,ltol,olambda,success,end,J,
	reduction,fs,x,_a,_b,sumf,C,el,indx,myparams);

	__syncthreads();

  	// finalise parameters
	//m_s0-myparams[0] 	m_d-myparams[1] 	m_d_std-myparams[2]		m_f-m_th-m_ph-myparams[3,4,5,6 etc..]   	m_f0-myparams[nparams-1]

	if(idSubVOX==0){  	
		float aux = myparams[1];

  		myparams[1] = abs(aux*myparams[2]);
		myparams[2] = sqrt(float(abs(aux*myparams[2]*myparams[2])));
  		for(int i=3,k=0;k<nfib;i+=3,k++){
    			myparams[i]  = x2f_gpu(myparams[i]);
  		}
  		if (m_include_f0)
    			myparams[nparams-1]=x2f_gpu(myparams[nparams-1]);

		sort_PVM_multi(nfib,myparams);
  		fix_fsum_PVM_multi(m_include_f0,nfib,nparams,myparams);
	}
	__syncthreads();

	if(idSubVOX<nparams){
		params[(idVOX*nparams)+idSubVOX] = myparams[idSubVOX];
	}
}

//in diffmodel.cc
extern "C" __global__ void get_residuals_PVM_multi_kernel(	//INPUT
								const float* 		data, 
								const float* 		params,
								const float* 		bvecs, 
								const float* 		bvals, 
								const float		R,
								const float		invR,
								const int 		nvox, 
								const int		ndirections,
								const int 		nfib, 
								const int		nparams,
								const int		Gamma_for_ball_only,
								const bool 		m_include_f0,
								const bool		gradnonlin,
								const bool* 		includes_f0,								
								//OUTPUT
								float*			residuals)
{
	int idSubVOX = threadIdx.x;
	int idVOX = blockIdx.x;

	////////// DYNAMIC SHARED MEMORY ///////////
	extern __shared__ double shared[];
	float* myparams = (float*) shared;			//nparams
	float* fs = (float*) &myparams[nparams];		//nfib
  	float* x = (float*) &fs[nfib];				//nfib*3
	float* _a = (float*) &x[nfib*3];			//1
	float* _b = (float*) &_a[1];				//1
  	float* sumf = (float*) &_b[1];				//1
	int* my_include_f0 = (int*) &sumf[1];			//1	
	////////// DYNAMIC SHARED MEMORY ///////////

	float val;
	float predicted_signal;
	float mydata;

	if(idSubVOX==0){
		*my_include_f0 = includes_f0[idVOX];

  		//m_s0-myparams[0]  m_d-myparams[1]  m_d_std-myparams[2]  m_f-m_th-m_ph-myparams[3,4,5,6 etc..]  m_f0-myparams[nparams-1]

  		myparams[0] = params[(idVOX*nparams)+0];
		float aux1 = params[(idVOX*nparams)+1];
		float aux2 = params[(idVOX*nparams)+2];
		
  		myparams[1] = aux1*aux1/aux2/aux2;		//m_d*m_d/m_d_std/m_d_std;
  		myparams[2] = aux2*aux2/aux1;			//m_d_std*m_d_std/m_d; // =1/beta
  		
  		if (*my_include_f0)
    			myparams[nparams-1]=f2x_gpu(params[(idVOX*nparams)+nparams-1]);
	}

	if(idSubVOX<nfib){
		int kk = 3+3*idSubVOX;
		float sinth,costh,sinph,cosph;
	
		myparams[kk]   = f2x_gpu(params[(idVOX*nparams)+kk]);
    		myparams[kk+1] = params[(idVOX*nparams)+kk+1];
    		myparams[kk+2] = params[(idVOX*nparams)+kk+2];

		sincos(myparams[kk+1],&sinth,&costh);
		sincos(myparams[kk+2],&sinph,&cosph);		
    		fs[idSubVOX] = x2f_gpu(myparams[kk]);
    		x[idSubVOX*3] = sinth*cosph;
    		x[idSubVOX*3+1] = sinth*sinph;
    		x[idSubVOX*3+2] = costh;
  	}

	__syncthreads(); 

	if(idSubVOX==0){
  		*_a = abs(myparams[1]);
  		*_b = abs(myparams[2]);
  		*sumf=0;
  		for(int k=0;k<nfib;k++){
	    		*sumf += fs[k];
		}
  	}
  	
	int ndir = ndirections/THREADS_BLOCK_FIT;
	if(idSubVOX<(ndirections%THREADS_BLOCK_FIT)) ndir++;
	
	float3 x2;
	int dir_iter=idSubVOX; 

	__syncthreads();

	int pos_bvals, pos_bvecs;
	if(gradnonlin){ 
		pos_bvals=idVOX*ndirections;
		pos_bvecs=idVOX*3*ndirections;
	}else{ 
		pos_bvals=0;
		pos_bvecs=0;
	}

  	for(int dir=0;dir<ndir;dir++){
		mydata = data[(idVOX*ndirections)+dir_iter];
  		predicted_signal=0;	//pred = 0;
    		val = 0.0;
    		for(int k=0;k<nfib;k++){
			x2.x=x[k*3];
			x2.y=x[k*3+1];
			x2.z=x[k*3+2];	 
      			val += fs[k]*anisoterm_PVM_multi(dir_iter,_a,_b,x2,&bvecs[pos_bvecs],&bvals[pos_bvals],R,invR,ndirections,Gamma_for_ball_only);
    		}	
    		if (*my_include_f0){
      			float temp_f0=x2f_gpu(myparams[nparams-1]);
      			predicted_signal = abs(myparams[0])*(temp_f0+(1-*sumf-temp_f0)*isoterm_PVM_multi(dir_iter,_a,_b,&bvals[pos_bvals])+val);
    		}else{
      			predicted_signal = abs(myparams[0])*((1-*sumf)*isoterm_PVM_multi(dir_iter,_a,_b,&bvals[pos_bvals])+val); 
  		}   

		//residuals=m_data-predicted_signal;
		residuals[idVOX*ndirections+dir_iter]= mydata - predicted_signal;

		dir_iter+=THREADS_BLOCK_FIT;
	}
}
