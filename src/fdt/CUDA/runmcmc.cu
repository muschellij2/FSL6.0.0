/*  runmcmc.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include "xfibresoptions.h"
#include <hiprand.h>
#include "runmcmc_kernels.cu"
#include "sync_check.h"

#include <host_vector.h>
#include <device_vector.h> 

#include <time.h>
#include <sys/time.h>
#include "init_gpu.h"

using namespace Xfibres;

////////////////////////////////////////////////////// 
//   MCMC ON GPU
////////////////////////////////////////////////////// 

void init_Fibres_Multifibres(	//INPUT
				thrust::device_vector<float>& 			datam_gpu,
				thrust::device_vector<float>& 			params_gpu,
				thrust::device_vector<float>& 			tau_gpu,
				thrust::device_vector<float>& 			bvals_gpu,
				thrust::device_vector<double>& 			alpha_gpu,
				thrust::device_vector<double>& 			beta_gpu,
				const int 					ndirections,
				string 						output_file,
				double 						seed,
				//OUTPUT
				thrust::device_vector<FibreGPU>& 		fibres_gpu,
				thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
				thrust::device_vector<double>&			signals_gpu,
				thrust::device_vector<double>&			isosignals_gpu,
				thrust::device_vector<hiprandState>&		randStates_gpu)
{
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app );
   	myfile << "----- MCMC ALGORITHM PART INITIALITATION ON GPU ----- " << "\n";  	

   	struct timeval t1,t2;
   	double time;
   	gettimeofday(&t1,NULL);

	int nvox = multifibres_gpu.size();

	xfibresOptions& opts = xfibresOptions::getInstance();
	int nfib= opts.nfibres.value();
	int nparams_fit = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()>=2) nparams_fit++;
	if(opts.f0.value()) nparams_fit++;

	thrust::device_vector<double> angtmp_gpu;
	angtmp_gpu.resize(nvox*ndirections*nfib);
	

	bool gradnonlin = opts.grad_file.set();

	int blocks = nvox/VOXELS_BLOCK_MCMC;
	if(nvox%VOXELS_BLOCK_MCMC) blocks++;
	int nthreads_block = THREADS_VOXEL_MCMC*VOXELS_BLOCK_MCMC;
  	dim3 Dim_Grid_MCMC(blocks, 1);
  	dim3 Dim_Block_MCMC(nthreads_block ,1);	///dimensions for MCMC

	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *params_ptr = thrust::raw_pointer_cast(params_gpu.data());	
	float *tau_ptr = thrust::raw_pointer_cast(tau_gpu.data());	
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());
	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	hiprandState *randStates_ptr = thrust::raw_pointer_cast(randStates_gpu.data());

	int amount_shared = VOXELS_BLOCK_MCMC*((THREADS_VOXEL_MCMC)*sizeof(double) + (3*nfib + 9)*sizeof(float) + sizeof(int));

	myfile << "Shared Memory Used in init_Fibres_Multifibres: " << amount_shared << "\n";

	init_Fibres_Multifibres_kernel<<< Dim_Grid_MCMC, Dim_Block_MCMC, amount_shared>>>(datam_ptr, params_ptr, tau_ptr, bvals_ptr, alpha_ptr, beta_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams_fit, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.rician.value(), opts.ardf0.value(), opts.all_ard.value(), opts.no_ard.value(), gradnonlin, angtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr);
	sync_check("init_Fibres_Multifibres_kernel");

	// Initialise Randoms
	int total_threads= nvox;
	int blocks_Rand = total_threads/THREADS_BLOCK_RAND;
	if(total_threads%THREADS_BLOCK_RAND) blocks_Rand++;
	dim3 Dim_Grid_Rand(blocks_Rand,1);
	dim3 Dim_Block_Rand(THREADS_BLOCK_RAND,1); 
	setup_randoms_kernel <<<Dim_Grid_Rand,Dim_Block_Rand>>>(randStates_ptr,seed,nvox);
	sync_check("Setup_Randoms_kernel");

	gettimeofday(&t2,NULL);
    	time=timeval_diff(&t2,&t1);
   	myfile << "TIME: " << time << " seconds\n"; 
	myfile << "-----------------------------------------------------" << "\n\n" ; 
	myfile.close();
}

void runmcmc_burnin(	//INPUT
			thrust::device_vector<float>& 			datam_gpu,
			thrust::device_vector<float>& 			bvals_gpu,
			thrust::device_vector<double>& 			alpha_gpu,
			thrust::device_vector<double>& 			beta_gpu,
			const int 					ndirections,
			string 						output_file, 
			//INPUT-OUTPUT
			thrust::device_vector<FibreGPU>& 		fibres_gpu,
			thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
			thrust::device_vector<double>&			signals_gpu,
			thrust::device_vector<double>&			isosignals_gpu,
			thrust::device_vector<hiprandState>&		randStates_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app ); 
   	myfile << "--------- MCMC ALGORITHM PART BURNIN ON GPU --------- " << "\n";  	

   	struct timeval t_tot1,t_tot2;
   	double time;
   	time=0;

   	gettimeofday(&t_tot1,NULL);
	
	int nvox = multifibres_gpu.size();
   	int nfib= opts.nfibres.value();
	int nparams;

	bool gradnonlin=opts.grad_file.set();

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;	
	if(opts.modelnum.value()>=2) nparams++;
	if(opts.modelnum.value()==3) nparams++;	
	if(opts.rician.value()) nparams++;

	thrust::device_vector<float> recors_null_gpu;
	recors_null_gpu.resize(1);

	thrust::device_vector<double> angtmp_gpu;
	thrust::device_vector<double> oldangtmp_gpu;
	thrust::device_vector<double> oldsignals_gpu;
	thrust::device_vector<double> oldisosignals_gpu;
	
	angtmp_gpu.resize(nvox*ndirections*nfib);
	oldangtmp_gpu.resize(nvox*ndirections);
	oldsignals_gpu.resize(nvox*ndirections*nfib);
	oldisosignals_gpu.resize(nvox*ndirections);

	myfile << "Processing " << nvox << " voxels \n";

  	int blocks = nvox/VOXELS_BLOCK_MCMC;
	if(nvox%VOXELS_BLOCK_MCMC) blocks++;
	int nthreads_block = THREADS_VOXEL_MCMC*VOXELS_BLOCK_MCMC;
  	dim3 Dim_Grid(blocks, 1);
  	dim3 Dim_Block(nthreads_block,1);	//dimensions for MCMC   

   	myfile << "NUM BLOCKS: " << blocks << "\n"; 
   	myfile << "THREADS PER BLOCK : " << nthreads_block << "\n"; 	


	//get pointers
	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());
	hiprandState *randStates_ptr = thrust::raw_pointer_cast(randStates_gpu.data());

	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	double *oldangtmp_ptr = thrust::raw_pointer_cast(oldangtmp_gpu.data());
	double *oldsignals_ptr = thrust::raw_pointer_cast(oldsignals_gpu.data());
	double *oldisosignals_ptr = thrust::raw_pointer_cast(oldisosignals_gpu.data());

	float *records_null = thrust::raw_pointer_cast(recors_null_gpu.data());

	int amount_shared = VOXELS_BLOCK_MCMC*((THREADS_VOXEL_MCMC)*sizeof(double) + (10*nfib + 27)*sizeof(float) + (7*nfib + 20)*sizeof(int)+ sizeof(hiprandState));

	myfile << "Shared Memory Used in runmcmc_burnin: " << amount_shared << "\n";

   	if(nvox!=0){
		runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr, randStates_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), opts.nburn.value(), 0, 0, 0, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr,records_null,records_null,records_null,records_null,records_null,records_null,records_null, records_null,records_null); 
   		sync_check("runmcmc_burnin_kernel");
   	}

	gettimeofday(&t_tot2,NULL);
    	time=timeval_diff(&t_tot2,&t_tot1);
   	myfile << "TIME: " << time << " seconds\n"; 
	myfile << "-----------------------------------------------------" << "\n\n" ; 
	myfile.close();
}


void runmcmc_record(	//INPUT
			thrust::device_vector<float>& 			datam_gpu,
			thrust::device_vector<float>& 			bvals_gpu,
			thrust::device_vector<double>& 			alpha_gpu,
			thrust::device_vector<double>& 			beta_gpu,
			thrust::device_vector<FibreGPU>& 		fibres_gpu,
			thrust::device_vector<MultifibreGPU>& 		multifibres_gpu,
			thrust::device_vector<double>&			signals_gpu,
			thrust::device_vector<double>&			isosignals_gpu,
			const int 					ndirections,
			thrust::device_vector<hiprandState>&		randStates_gpu,
			string 						output_file, 
			//OUTPUT
			thrust::device_vector<float>&			rf0_gpu,
			thrust::device_vector<float>&			rtau_gpu,
			thrust::device_vector<float>&			rs0_gpu,
			thrust::device_vector<float>&			rd_gpu,
			thrust::device_vector<float>&			rdstd_gpu,
			thrust::device_vector<float>&			rR_gpu,
			thrust::device_vector<float>&			rth_gpu,
			thrust::device_vector<float>&			rph_gpu,
			thrust::device_vector<float>&			rf_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app );
   	myfile << "--------- MCMC ALGORITHM PART RECORD ON GPU --------- " << "\n"; 	

   	struct timeval t_tot1,t_tot2;
   	double time;
   	time=0;

   	gettimeofday(&t_tot1,NULL);

	int totalrecords = (opts.njumps.value()/opts.sampleevery.value()); 
	
	int nvox = multifibres_gpu.size();
   	int nfib= opts.nfibres.value();
	int nparams;

	bool gradnonlin=opts.grad_file.set();

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;	
	if(opts.modelnum.value()>=2) nparams++;
	if(opts.modelnum.value()==3) nparams++;	
	if(opts.rician.value()) nparams++;

	thrust::device_vector<double> angtmp_gpu;
	thrust::device_vector<double> oldangtmp_gpu;
	thrust::device_vector<double> oldsignals_gpu;
	thrust::device_vector<double> oldisosignals_gpu;
	
	angtmp_gpu.resize(nvox*ndirections*nfib);
	oldangtmp_gpu.resize(nvox*ndirections);
	oldsignals_gpu.resize(nvox*ndirections*nfib);
	oldisosignals_gpu.resize(nvox*ndirections);
   
	myfile << "Processing " << nvox << " voxels \n";
   
  	int blocks = nvox/VOXELS_BLOCK_MCMC;
	int nthreads_block = THREADS_VOXEL_MCMC*VOXELS_BLOCK_MCMC;
	if(nvox%VOXELS_BLOCK_MCMC) blocks++;
  	dim3 Dim_Grid(blocks, 1);
  	dim3 Dim_Block(nthreads_block,1);	//dimensions for MCMC   

   	myfile << "NUM BLOCKS: " << blocks << "\n"; 
   	myfile << "THREADS PER BLOCK : " << nthreads_block << "\n"; 	

	//get pointers
	float *datam_ptr = thrust::raw_pointer_cast(datam_gpu.data());
	float *bvals_ptr = thrust::raw_pointer_cast(bvals_gpu.data());
	double *alpha_ptr = thrust::raw_pointer_cast(alpha_gpu.data());
	double *beta_ptr = thrust::raw_pointer_cast(beta_gpu.data());
	FibreGPU *fibres_ptr =  thrust::raw_pointer_cast(fibres_gpu.data());
	MultifibreGPU *multifibres_ptr = thrust::raw_pointer_cast(multifibres_gpu.data());
	double *signals_ptr = thrust::raw_pointer_cast(signals_gpu.data());
	double *isosignals_ptr = thrust::raw_pointer_cast(isosignals_gpu.data());
	hiprandState *randStates_ptr = thrust::raw_pointer_cast(randStates_gpu.data());

	double *angtmp_ptr = thrust::raw_pointer_cast(angtmp_gpu.data());
	double *oldangtmp_ptr = thrust::raw_pointer_cast(oldangtmp_gpu.data());
	double *oldsignals_ptr = thrust::raw_pointer_cast(oldsignals_gpu.data());
	double *oldisosignals_ptr = thrust::raw_pointer_cast(oldisosignals_gpu.data());
	
	float *rf0_ptr = thrust::raw_pointer_cast(rf0_gpu.data());
	float *rtau_ptr = thrust::raw_pointer_cast(rtau_gpu.data());
	float *rs0_ptr = thrust::raw_pointer_cast(rs0_gpu.data());
	float *rd_ptr = thrust::raw_pointer_cast(rd_gpu.data());
	float *rdstd_ptr = thrust::raw_pointer_cast(rdstd_gpu.data());
	float *rR_ptr = thrust::raw_pointer_cast(rR_gpu.data());	
	float *rth_ptr = thrust::raw_pointer_cast(rth_gpu.data());
	float *rph_ptr = thrust::raw_pointer_cast(rph_gpu.data());
	float *rf_ptr = thrust::raw_pointer_cast(rf_gpu.data());

	int amount_shared = VOXELS_BLOCK_MCMC*((THREADS_VOXEL_MCMC)*sizeof(double) + (10*nfib + 27)*sizeof(float) + (7*nfib + 20)*sizeof(int)+ sizeof(hiprandState));

	myfile << "Shared Memory Used in runmcmc_record: " << amount_shared << "\n";

   	if(nvox!=0){
		runmcmc_kernel<<< Dim_Grid, Dim_Block, amount_shared >>>(datam_ptr, bvals_ptr, alpha_ptr, beta_ptr, randStates_ptr, opts.R_prior_mean.value(), opts.R_prior_std.value(),opts.R_prior_fudge.value(), ndirections, nfib, nparams, opts.modelnum.value(), opts.fudge.value(), opts.f0.value(), opts.ardf0.value(), !opts.no_ard.value(), opts.rician.value(), gradnonlin, opts.updateproposalevery.value(), opts.njumps.value(), opts.nburn.value(), opts.sampleevery.value(), totalrecords, oldsignals_ptr, oldisosignals_ptr, angtmp_ptr, oldangtmp_ptr, fibres_ptr, multifibres_ptr, signals_ptr, isosignals_ptr, rf0_ptr, rtau_ptr, rs0_ptr, rd_ptr, rdstd_ptr, rR_ptr, rth_ptr, rph_ptr, rf_ptr);
   		sync_check("runmcmc_record_kernel");
   	}

   	gettimeofday(&t_tot2,NULL);
    	time=timeval_diff(&t_tot2,&t_tot1);
   	myfile << "TIME: " << time << " seconds\n"; 
	myfile << "-----------------------------------------------------" << "\n" ;
	myfile.close(); 
}
