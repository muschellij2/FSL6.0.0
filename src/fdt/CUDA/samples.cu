#include "hip/hip_runtime.h"
/*  samples.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include "newmat.h"
#include "newimage/newimageall.h"
#include "xfibresoptions.h"
#include "samples.h"

using namespace Xfibres;

////////////////////////////////////////////
//       MCMC SAMPLE STORAGE
////////////////////////////////////////////

Samples::Samples(int nvoxels,int nsamples):
opts(xfibresOptions::getInstance()){

	/////////////// GPU version /////////////////////
    	m_sum_d=new float[nvoxels];
    	m_sum_S0=new float[nvoxels];
    	for(int i=0;i<nvoxels;i++){
    		m_sum_d[i]=0;
     		m_sum_S0[i]=0;
    	}
    	m_vec=new ColumnVector[nvoxels];
    	m_dyad=new vector<SymmetricMatrix>[nvoxels];
    	m_sum_f=new vector<float> [nvoxels];
    	m_sum_lam=new vector<float> [nvoxels];	
    	////////////////////////////////////////////////
    
    	m_dsamples.ReSize(nsamples,nvoxels);
    	m_dsamples=0;
    	m_S0samples.ReSize(nsamples,nvoxels);
    	m_S0samples=0;

    	m_mean_dsamples.ReSize(nvoxels);
    	m_mean_dsamples=0;
    	m_mean_S0samples.ReSize(nvoxels);
    	m_mean_S0samples=0;
    	Matrix tmpvecs(3,nvoxels);
    	tmpvecs=0;
    	//m_sum_d=0;  changed GPU version
    	//m_sum_S0=0;  changed GPU version

    	if(opts.modelnum.value()>=2){
      		m_d_stdsamples.ReSize(nsamples,nvoxels);
      		m_d_stdsamples=0;
      		m_mean_d_stdsamples.ReSize(nvoxels);
      		m_mean_d_stdsamples=0;
      		//m_sum_d_std=0;  changed GPU version

      		/////////////// GPU version /////////////////////
      		m_sum_d_std=new float[nvoxels];
      		for(int i=0;i<nvoxels;i++){
      			m_sum_d_std[i]=0;
      		}
      		////////////////////////////////////////////////
		if(opts.modelnum.value()==3){
			m_Rsamples.ReSize(nsamples,nvoxels);
			m_Rsamples=0;
			m_mean_Rsamples.ReSize(nvoxels);
			m_mean_Rsamples=0;
			m_sum_R=new float[nvoxels];
			for(int i=0;i<nvoxels;i++){
      				m_sum_R[i]=0;
      			}
		}
    	}

    	if (opts.f0.value()){
      		m_f0samples.ReSize(nsamples,nvoxels);
      		m_f0samples=0;
      		m_mean_f0samples.ReSize(nvoxels);
      		m_mean_f0samples=0;
      		//m_sum_f0=0;  changed GPU version

     	 	/////////////// GPU version /////////////////////
      		m_sum_f0=new float[nvoxels];
      		for(int i=0;i<nvoxels;i++)
      			m_sum_f0[i]=0;
      		////////////////////////////////////////////////
    	}

    	if (opts.rician.value()){
      		m_mean_tausamples.ReSize(nvoxels);
      		m_mean_tausamples=0;
      		//m_sum_tau=0;  changed GPU version

      		/////////////// GPU version /////////////////////
      		m_sum_tau=new float[nvoxels];
      		for(int i=0;i<nvoxels;i++)
      			m_sum_tau[i]=0;
      		////////////////////////////////////////////////
    	}

    	SymmetricMatrix tmpdyad(3);
    	tmpdyad=0;
    	m_nsamps=nsamples;
    	//m_vec.ReSize(3);  changed GPU version

    	/////////////// GPU version /////////////////////
    	for(int i=0;i<nvoxels;i++){ 
        	m_vec[i].ReSize(3);
		for(int f=0;f<opts.nfibres.value();f++){
			m_dyad[i].push_back(tmpdyad);
                	m_sum_f[i].push_back(0);
                	m_sum_lam[i].push_back(0);
        	}
    	}	 
    	////////////////////////////////////////////////

    	for(int f=0;f<opts.nfibres.value();f++){
      		m_thsamples.push_back(m_S0samples);
      		m_phsamples.push_back(m_S0samples);
      		m_fsamples.push_back(m_S0samples);
      		m_lamsamples.push_back(m_S0samples);

      		m_dyadic_vectors.push_back(tmpvecs);
      		m_mean_fsamples.push_back(m_mean_S0samples);
      		m_mean_lamsamples.push_back(m_mean_S0samples);

      		//m_sum_lam.push_back(0);  changed GPU version
      		//m_sum_f.push_back(0);  changed GPU version
      		//m_dyad.push_back(tmpdyad);  changed GPU version
    	}
}

	//new version for GPU
void Samples::record(float rd,float rf0,float rtau,float rdstd,float rR,float rs0,float *rth,float *rph, float *rf, int vox, int samp){
    	m_dsamples(samp,vox)=rd;
    	m_sum_d[vox-1]+=rd;

    	if(opts.modelnum.value()>=2){
		m_d_stdsamples(samp,vox)=rdstd;
      		m_sum_d_std[vox-1]+=rdstd;
    	}
	if(opts.modelnum.value()==3){
		m_Rsamples(samp,vox)=rR;
		m_sum_R[vox-1]+=rR;
    	}	
    	if (opts.f0.value()){
     		m_f0samples(samp,vox)=rf0;
      		m_sum_f0[vox-1]+=rf0;
    	}
    	if (opts.rician.value()){
      		m_sum_tau[vox-1]+=rtau;
    	}

    	m_S0samples(samp,vox)=rs0;
    	m_sum_S0[vox-1]+=rs0;

    	for(int f=0;f<opts.nfibres.value();f++){
      		float th=rth[f];
      		float ph=rph[f];
      		m_thsamples[f](samp,vox)=th;
      		m_phsamples[f](samp,vox)=ph;
      		m_fsamples[f](samp,vox)=rf[f];
     	 	//for means
      		m_vec[vox-1] << sin(th)*cos(ph) << sin(th)*sin(ph)<<cos(th) ;

      		m_dyad[vox-1][f] << m_dyad[vox-1][f]+m_vec[vox-1]*m_vec[vox-1].t();
      		m_sum_f[vox-1][f]+=rf[f];
      		m_sum_lam[vox-1][f]+=0;
    	}
}  

//new version for GPU
 void Samples::finish_voxel(int vox){
    	m_mean_dsamples(vox)=m_sum_d[vox-1]/m_nsamps;

    	if(opts.modelnum.value()>=2)
      		m_mean_d_stdsamples(vox)=m_sum_d_std[vox-1]/m_nsamps;
	if(opts.modelnum.value()==3)
      		m_mean_Rsamples(vox)=m_sum_R[vox-1]/m_nsamps;
    	if(opts.f0.value())
      		m_mean_f0samples(vox)=m_sum_f0[vox-1]/m_nsamps;
    	if(opts.rician.value())
      		m_mean_tausamples(vox)=m_sum_tau[vox-1]/m_nsamps;

    	m_mean_S0samples(vox)=m_sum_S0[vox-1]/m_nsamps;

    	m_sum_d[vox-1]=0;
    	m_sum_S0[vox-1]=0;
   
    	if(opts.rician.value())
    		m_sum_tau[vox-1]=0;

    	if(opts.modelnum.value()>=2)
      		m_sum_d_std[vox-1]=0;
	if(opts.modelnum.value()==3)
		m_sum_R[vox-1]=0;
    	if (opts.f0.value())
      		m_sum_f0[vox-1]=0;

    	DiagonalMatrix dyad_D; //eigenvalues
    	Matrix dyad_V; //eigenvectors
    	int nfibs=0;
    	for(int f=0;f<opts.nfibres.value();f++){
      		EigenValues(m_dyad[vox-1][f],dyad_D,dyad_V);
      		int maxeig;
      		if(dyad_D(1)>dyad_D(2)){
			if(dyad_D(1)>dyad_D(3)) maxeig=1;
			else maxeig=3;
      		}
      		else{
			if(dyad_D(2)>dyad_D(3)) maxeig=2;
			else maxeig=3;
      		}
      		m_dyadic_vectors[f](1,vox)=dyad_V(1,maxeig);
      		m_dyadic_vectors[f](2,vox)=dyad_V(2,maxeig);
      		m_dyadic_vectors[f](3,vox)=dyad_V(3,maxeig);
      
      		if((m_sum_f[vox-1][f]/m_nsamps)>0.05){
			nfibs++;
      		}
      		m_mean_fsamples[f](vox)=m_sum_f[vox-1][f]/m_nsamps;
      		m_mean_lamsamples[f](vox)=m_sum_lam[vox-1][f]/m_nsamps;
      
      		m_dyad[vox-1][f]=0;
      		m_sum_f[vox-1][f]=0;
      		m_sum_lam[vox-1][f]=0;
    	}
    	//m_beenhere(int(m_matrix2volkey(vox,1)),int(m_matrix2volkey(vox,2)),int(m_matrix2volkey(vox,3)))=nfibs;
}

void save_part(RowVector data, string name, int idpart){
	Log& logger = LogSingleton::getInstance();
    	xfibresOptions& opts = xfibresOptions::getInstance();

	int nvox = data.Ncols();

	string file_name;

	file_name = logger.appendDir(name+"_"+num2str(idpart));
	ofstream out;
	out.open(file_name.data(), ios::out | ios::binary);
	out.write((char*)&data(1),nvox*sizeof(Real));
	out.close();
}

void save_part(Matrix data, string name, int idpart){
	Log& logger = LogSingleton::getInstance();
    	xfibresOptions& opts = xfibresOptions::getInstance();

	int nvox = data.Ncols();
	int nsamples = data.Nrows();

	string file_name;

	file_name = logger.appendDir(name+"_"+num2str(idpart));
	ofstream out;
	out.open(file_name.data(), ios::out | ios::binary);
	out.write((char*)&data(1,1),nvox*nsamples*sizeof(Real));
	out.close();
}
  
void Samples::save(int idpart){

	vector<Matrix> thsamples_out=m_thsamples;
	vector<Matrix> phsamples_out=m_phsamples;
	vector<Matrix> fsamples_out=m_fsamples;
	vector<Matrix> lamsamples_out=m_lamsamples;
    
    	vector<Matrix> dyadic_vectors_out=m_dyadic_vectors;
    	vector<Matrix> mean_fsamples_out;
    	for(unsigned int f=0;f<m_mean_fsamples.size();f++)
      		mean_fsamples_out.push_back(m_mean_fsamples[f]);

    	Log& logger = LogSingleton::getInstance();
    	if(opts.modelnum.value()==1){
		save_part(m_mean_dsamples,"mean_dsamples",idpart);
    	}
    	else if(opts.modelnum.value()>=2){
		save_part(m_mean_dsamples,"mean_dsamples",idpart);
		save_part(m_mean_d_stdsamples,"mean_d_stdsamples",idpart);
		//save_part(m_dsamples,"m_d_stdsamples",idpart);
		//save_part(m_d_stdsamples,"d_stdsamples",idpart);
		if(opts.modelnum.value()==3){
			save_part(m_mean_Rsamples,"mean_Rsamples",idpart);
		}
    	}
    	if (opts.f0.value()){
		save_part(m_mean_f0samples,"mean_f0samples",idpart);
		//save_part(m_f0samples,"f0samples",idpart);
    	}
    	if (opts.rician.value()){
		save_part(m_mean_tausamples,"mean_tausamples",idpart);	
    	}

	save_part(m_mean_S0samples,"mean_S0samples",idpart);
	
    	//Sort the output based on mean_fsamples
    	// 
    	vector<Matrix> sumf;
    	for(int f=0;f<opts.nfibres.value();f++){
      		Matrix tmp=sum(m_fsamples[f],1);
      		sumf.push_back(tmp);
    	}  
    	for(int vox=1;vox<=m_dsamples.Ncols();vox++){
      		vector<pair<float,int> > sfs;
      		pair<float,int> ftmp;
      
      		for(int f=0;f<opts.nfibres.value();f++){
			ftmp.first=sumf[f](1,vox);
			ftmp.second=f;
			sfs.push_back(ftmp);
      		}
      		sort(sfs.begin(),sfs.end());
      
      		for(int samp=1;samp<=m_dsamples.Nrows();samp++){
			for(int f=0;f<opts.nfibres.value();f++){;
	  			thsamples_out[f](samp,vox)=m_thsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
	  			phsamples_out[f](samp,vox)=m_phsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
	  			fsamples_out[f](samp,vox)=m_fsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
	  			lamsamples_out[f](samp,vox)=m_lamsamples[sfs[(sfs.size()-1)-f].second](samp,vox);
			}
      		}
      
      		for(int f=0;f<opts.nfibres.value();f++){
			mean_fsamples_out[f](1,vox)=m_mean_fsamples[sfs[(sfs.size()-1)-f].second](vox);
			dyadic_vectors_out[f](1,vox)=m_dyadic_vectors[sfs[(sfs.size()-1)-f].second](1,vox);
			dyadic_vectors_out[f](2,vox)=m_dyadic_vectors[sfs[(sfs.size()-1)-f].second](2,vox);
			dyadic_vectors_out[f](3,vox)=m_dyadic_vectors[sfs[(sfs.size()-1)-f].second](3,vox);
      		}
    	}
    	// save the sorted fibres
    	for(int f=0;f<opts.nfibres.value();f++){
      		//      element_mod_n(thsamples_out[f],M_PI);
      		//      element_mod_n(phsamples_out[f],2*M_PI);

		save_part(thsamples_out[f],"th"+num2str(f+1)+"samples",idpart);

		save_part(phsamples_out[f],"ph"+num2str(f+1)+"samples",idpart);

		save_part(fsamples_out[f],"f"+num2str(f+1)+"samples",idpart);

		//save_part(mean_fsamples_out[f],"mean_f"+num2str(f+1)+"samples",idpart);
		//save_part(dyadic_vectors_out[f],"dyads"+num2str(f+1),idpart);
      
      			
    	}
}
