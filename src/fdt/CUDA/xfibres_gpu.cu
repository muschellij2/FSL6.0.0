#include "hip/hip_runtime.h"
/*  xfibres_gpu.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include "newmat.h"
#include "newimage/newimageall.h"
#include "xfibresoptions.h"

#include "xfibres_gpu.cuh"
#include "diffmodels.cuh"
#include "runmcmc.h"
#include "samples.h"
#include "options.h"

#include <host_vector.h>
#include <device_vector.h> 

#include <time.h>
#include <sys/time.h>
#include "init_gpu.h"
#include <fstream>

using namespace Xfibres;

void xfibres_gpu(	//INPUT
			const Matrix			datam,
			const Matrix			bvecs,
			const Matrix			bvals,
			const Matrix	 		gradm,
			int				idpart,
			int				idSubpart,
			float				seed,
			string				subjdir)
{
	//write num of part in a string for log file
	char part_str[8];
	char subpart_str[8];
	char aux[8];
	sprintf(part_str,"%d",idpart);
	while(strlen(part_str)<4){
		strcpy(aux,"0");
		strcat(aux,part_str);
		strcpy(part_str,aux);
	}
	sprintf(subpart_str,"%d",idSubpart);
	while(strlen(subpart_str)<4){
		strcpy(aux,"0");
		strcat(aux,subpart_str);
		strcpy(subpart_str,aux);
	}
	string gpu_log(subjdir);		//logfile
	gpu_log += ".bedpostX/logs/logs_gpu/part_";
	gpu_log += part_str;
	gpu_log += "-subpart_";
	gpu_log += subpart_str;
	std::ofstream myfile;
	myfile.open (gpu_log.data(), ios::out | ios::app );
	myfile << "----------------------------------------------------- " << "\n"; 
   	myfile << "---------------- PART " << idpart  << " SUBPART "<< idSubpart << " ------------------- " << "\n"; 
   	myfile << "----------------------------------------------------- " << "\n\n"; 
	myfile.close();

	xfibresOptions& opts = xfibresOptions::getInstance();

	int nvox = datam.Ncols();
	int ndirections = datam.Nrows();
	int nfib= opts.nfibres.value(); 
	bool gradnonlin=opts.grad_file.set();

	if(nvox>0){
		thrust::host_vector<float> datam_host, bvecs_host, bvals_host, params_host;
		thrust::host_vector<double> alpha_host, beta_host;
		thrust::host_vector<float> tau_host;
		vector<ColumnVector> datam_vec;
		vector<Matrix> bvecs_vec, bvals_vec;

		///// FIT /////
		prepare_data_gpu_FIT(datam,bvecs,bvals,gradm,datam_vec, bvecs_vec, bvals_vec, datam_host, bvecs_host,  bvals_host, alpha_host, beta_host, params_host, tau_host);	

		thrust::device_vector<float> datam_gpu=datam_host;
		thrust::device_vector<float> bvecs_gpu=bvecs_host;
		thrust::device_vector<float> bvals_gpu=bvals_host;	
		thrust::device_vector<float> params_gpu=params_host;
		thrust::host_vector<int> vox_repeat;	//contains the id's of voxels repeated
		vox_repeat.resize(nvox);
		int nrepeat=0;

		fit(datam_vec,bvecs_vec,bvals_vec,datam_host,bvecs_host,bvals_host,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,gpu_log,params_gpu,vox_repeat,nrepeat);

		if(opts.rician.value()){
			float R = opts.R_prior_mean.value();
			int Gamma_ball_only=0;  //That flag for diffmodels means default model2
      			if (opts.modelnum.value()==3) Gamma_ball_only=2;  //That flag for diffmodels means default model3 (with constant R)	
			calculate_tau(datam_gpu,params_gpu,bvecs_gpu,bvals_gpu,vox_repeat,nrepeat,ndirections,nfib,opts.modelnum.value(),opts.f0.value(),opts.nonlin.value(),gradnonlin,R,Gamma_ball_only,gpu_log,tau_host);
		}

		bvecs_gpu.clear();		//free bvecs_gpu
		bvecs_gpu.shrink_to_fit();
	
		//////   RUN MCMC  //////
		// At MCMC step it can happens that the number of voxels is not multipe of VOXELS_BLOCK_MCMC 
		// make it multiple
		// At Levenberg step, I launch as many blocks as voxels, so there is not such a problem 
		int nVOX_multiple = int(nvox/VOXELS_BLOCK_MCMC)*VOXELS_BLOCK_MCMC;
		if(nvox%VOXELS_BLOCK_MCMC) nVOX_multiple=nVOX_multiple+VOXELS_BLOCK_MCMC;

		thrust::host_vector<double> signals_host;
		thrust::host_vector<double> isosignals_host;
		thrust::host_vector<FibreGPU> fibres_host;
		thrust::host_vector<MultifibreGPU> multifibres_host;
	
		prepare_data_gpu_MCMC(nVOX_multiple, ndirections, nfib, signals_host, isosignals_host, fibres_host, multifibres_host);
		
		thrust::device_vector<double> signals_gpu=signals_host;
		thrust::device_vector<double> isosignals_gpu=isosignals_host;
		thrust::device_vector<FibreGPU> fibres_gpu=fibres_host;
		thrust::device_vector<MultifibreGPU> multifibres_gpu=multifibres_host;
		thrust::device_vector<float> tau_gpu = tau_host;
		thrust::device_vector<double> alpha_gpu=alpha_host;
		thrust::device_vector<double> beta_gpu=beta_host;

		thrust::device_vector<hiprandState> randStates_gpu;
		resize_structures(nVOX_multiple,ndirections,datam_gpu, params_gpu, tau_gpu, bvals_gpu, alpha_gpu, beta_gpu, randStates_gpu);

		init_Fibres_Multifibres(datam_gpu, params_gpu, tau_gpu, bvals_gpu, alpha_gpu, beta_gpu, ndirections, gpu_log, seed, fibres_gpu, multifibres_gpu, signals_gpu, isosignals_gpu,randStates_gpu);

		runmcmc_burnin(datam_gpu, bvals_gpu, alpha_gpu, beta_gpu, ndirections, gpu_log, fibres_gpu,multifibres_gpu, signals_gpu, isosignals_gpu, randStates_gpu);

		thrust::device_vector<float> rf0_gpu,rtau_gpu,rs0_gpu,rd_gpu,rdstd_gpu,rR_gpu,rth_gpu,rph_gpu,rf_gpu;

		prepare_data_gpu_MCMC_record(nVOX_multiple,rf0_gpu,rtau_gpu,rs0_gpu,rd_gpu,rdstd_gpu,rR_gpu,rth_gpu,rph_gpu,rf_gpu);

		runmcmc_record(datam_gpu, bvals_gpu, alpha_gpu,beta_gpu, fibres_gpu, multifibres_gpu, signals_gpu, isosignals_gpu, ndirections, randStates_gpu, gpu_log, rf0_gpu, rtau_gpu, rs0_gpu, rd_gpu, rdstd_gpu, rR_gpu, rth_gpu, rph_gpu, rf_gpu);

		/////// FINISH ALL VOXELS  ///////
		record_finish_voxels(rf0_gpu,rtau_gpu,rs0_gpu,rd_gpu,rdstd_gpu,rR_gpu,rth_gpu,rph_gpu,rf_gpu,nvox,nVOX_multiple,idSubpart);
	}else{
		/////// FINISH EMPTY SLICE  ///////	
		Samples samples(nvox,ndirections);
		samples.save(idSubpart);
	}
}


// Correct bvals/bvecs accounting for Gradient Nonlinearities
// ColumnVector grad_nonlin has 9 entries, corresponding to the 3 components of each of the x,y and z gradient deviation
void correct_bvals_bvecs(const Matrix& bvals,const Matrix& bvecs, const ColumnVector& grad_nonlin, Matrix& bvals_c, Matrix& bvecs_c){
  	bvals_c=bvals; bvecs_c=bvecs;
  	Matrix L(3,3);  //gradient coil tensor
  	float mag;
  	L(1,1)=grad_nonlin(1);  L(1,2)=grad_nonlin(4);  L(1,3)=grad_nonlin(7);
  	L(2,1)=grad_nonlin(2);  L(2,2)=grad_nonlin(5);  L(2,3)=grad_nonlin(8);
  	L(3,1)=grad_nonlin(3);  L(3,2)=grad_nonlin(6);  L(3,3)=grad_nonlin(9);

  	IdentityMatrix Id(3);
  
  	for (int l=1; l<=bvals.Ncols(); l++){
    		if (bvals(1,l)>0){ //do not correct b0s
     		 	bvecs_c.Column(l)=(Id+L)*bvecs.Column(l);
      			mag=sqrt(bvecs_c(1,l)*bvecs_c(1,l)+bvecs_c(2,l)*bvecs_c(2,l)+bvecs_c(3,l)*bvecs_c(3,l));
      			if (mag!=0)
				bvecs_c.Column(l)=bvecs_c.Column(l)/mag;
      			bvals_c(1,l)=mag*mag*bvals(1,l);//mag^2 as b propto |G|^2
    		}
  	}
}

//////   FIT  //////
void fit(	//INPUT
		const vector<ColumnVector> 	datam_vec, 
		const vector<Matrix> 		bvecs_vec,
		const vector<Matrix> 		bvals_vec,
		thrust::host_vector<float> 	datam_host,
		thrust::host_vector<float>	bvecs_host, 
		thrust::host_vector<float>	bvals_host,
		thrust::device_vector<float> 	datam_gpu, 
		thrust::device_vector<float>	bvecs_gpu, 
		thrust::device_vector<float>	bvals_gpu,
		int 				ndirections,
		string 				output_file,
		//OUTPUT
		thrust::device_vector<float>&	params_gpu,
		thrust::host_vector<int>&	vox_repeat,	//for get residuals with or withot f0
		int&				nrepeat)
{
	std::ofstream myfile;
	myfile.open (output_file.data(), ios::out | ios::app );
   	myfile << "------------------- FIT ON GPU ---------------------- " << "\n";
	myfile.close();

	struct timeval t1,t2;
   	double time;
   	gettimeofday(&t1,NULL);

	xfibresOptions& opts = xfibresOptions::getInstance();
	int nvox = datam_vec.size();
	int nfib= opts.nfibres.value();
	int nparams_fit = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()>=2) nparams_fit++;
	if(opts.f0.value()) nparams_fit++;
	bool gradnonlin=opts.grad_file.set();

	if(opts.modelnum.value()==1){
		if(opts.nonlin.value()){ 
			fit_PVM_single(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,nfib,opts.f0.value(),gradnonlin,output_file,params_gpu);

			if (opts.f0.value()){
				float md,mf,f0;	
				thrust::host_vector<float> params_host;
				params_host.resize(nvox*nparams_fit);
				thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	
				for(int vox=0;vox<nvox;vox++){			
					md = params_host[vox*nparams_fit+(1)];
					mf = params_host[vox*nparams_fit+(2)];
					f0 = params_host[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<float> 	datam_repeat_host;
					thrust::host_vector<float> 	bvecs_repeat_host;	
					thrust::host_vector<float> 	bvals_repeat_host;	
					thrust::host_vector<float> 	params_repeat_host;	
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, ndirections, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host, bvals_repeat_host, params_repeat_host);

					thrust::device_vector<float> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<float> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<float> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<float> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single(datam_repeat_vec,bvecs_repeat_vec,bvals_repeat_vec,datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,ndirections,nfib,false,gradnonlin,output_file,params_repeat_gpu);
					thrust::copy(params_repeat_gpu.begin(), params_repeat_gpu.end(), params_repeat_host.begin());	
					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_host,vox_repeat, nrepeat, nvox, params_gpu);
				}
	  		}
		}else{
			fit_PVM_single_c(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,nfib,opts.f0.value(),gradnonlin,output_file,params_gpu);

			if (opts.f0.value()){
				float md,mf,f0;	
				thrust::host_vector<float> params_host;
				params_host.resize(nvox*nparams_fit);
				thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	
				for(int vox=0;vox<nvox;vox++){		
					md = params_host[vox*nparams_fit+(1)];
					mf = params_host[vox*nparams_fit+(2)];
					f0 = params_host[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<float> 	datam_repeat_host;
					thrust::host_vector<float> 	bvecs_repeat_host;	
					thrust::host_vector<float> 	bvals_repeat_host;	
					thrust::host_vector<float> 	params_repeat_host;	
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, ndirections, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host, bvals_repeat_host, params_repeat_host);

					thrust::device_vector<float> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<float> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<float> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<float> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single_c(datam_repeat_vec,bvecs_repeat_vec,bvals_repeat_vec,datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,ndirections,nfib,false,gradnonlin,output_file,params_repeat_gpu);
					thrust::copy(params_repeat_gpu.begin(), params_repeat_gpu.end(), params_repeat_host.begin());	

					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_host ,vox_repeat, nrepeat, nvox, params_gpu);
				}
	  		}
		}
	}else{
      		//model 2 or 3
		fit_PVM_single(datam_vec,bvecs_vec,bvals_vec,datam_gpu,bvecs_gpu,bvals_gpu,ndirections,nfib,opts.f0.value(),gradnonlin,output_file,params_gpu);
	
		float R = opts.R_prior_mean.value();
		int Gamma_ball_only=0;  //That flag for diffmodels means default model2
      		if (opts.modelnum.value()==3) Gamma_ball_only=2;  //That flag for diffmodels means default model3 (with constant R)

		fit_PVM_multi(datam_gpu,bvecs_gpu,bvals_gpu,nvox,ndirections,nfib,opts.f0.value(),gradnonlin,R,Gamma_ball_only,output_file,params_gpu);	

		if (opts.f0.value()){
				float md,mf,f0;	
				thrust::host_vector<float> params_host;
				params_host.resize(nvox*nparams_fit);
				thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	
				for(int vox=0;vox<nvox;vox++){			
					md = params_host[vox*nparams_fit+(1)];
					mf = params_host[vox*nparams_fit+(3)];
					f0 = params_host[vox*nparams_fit+(nparams_fit-1)];
					if ((opts.nfibres.value()>0 && mf<0.05) || md>0.007 || f0>0.4){		//if true we need to repeat this voxel
						vox_repeat[nrepeat]=vox;
						nrepeat++;
					}
				}
				if(nrepeat>0){
					//prepare structures for the voxels that need to be reprocessed
					vector<ColumnVector> 	datam_repeat_vec; 
					vector<Matrix> 		bvecs_repeat_vec;
					vector<Matrix> 		bvals_repeat_vec;
					thrust::host_vector<float> 	datam_repeat_host;
					thrust::host_vector<float> 	bvecs_repeat_host;	
					thrust::host_vector<float> 	bvals_repeat_host;	
					thrust::host_vector<float> 	params_repeat_host;		
								
					prepare_data_gpu_FIT_repeat(datam_host, bvecs_host, bvals_host, vox_repeat, nrepeat, ndirections, datam_repeat_vec, bvecs_repeat_vec, bvals_repeat_vec, datam_repeat_host, bvecs_repeat_host,  bvals_repeat_host, params_repeat_host);

					thrust::device_vector<float> datam_repeat_gpu=datam_repeat_host;
					thrust::device_vector<float> bvecs_repeat_gpu=bvecs_repeat_host;
					thrust::device_vector<float> bvals_repeat_gpu=bvals_repeat_host;	
					thrust::device_vector<float> params_repeat_gpu=params_repeat_host;
				
		 			fit_PVM_single_c(datam_repeat_vec,bvecs_repeat_vec,bvals_repeat_vec,datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,ndirections,nfib,false,gradnonlin,output_file,params_repeat_gpu);

					fit_PVM_multi(datam_repeat_gpu,bvecs_repeat_gpu,bvals_repeat_gpu,nrepeat,ndirections,nfib,false,gradnonlin,R,Gamma_ball_only,output_file,params_repeat_gpu);	
					thrust::copy(params_repeat_gpu.begin(), params_repeat_gpu.end(), params_repeat_host.begin());	
		
					//mix all the parameteres: repeated and not repeated
					mix_params(params_repeat_host ,vox_repeat, nrepeat,  nvox, params_gpu);
				}
	  		}	
	}

	gettimeofday(&t2,NULL);
    	time=timeval_diff(&t2,&t1);
	myfile.open (output_file.data(), ios::out | ios::app );
   	myfile << "TIME: " << time << " seconds\n"; 
	myfile << "-----------------------------------------------------" << "\n\n" ; 
	myfile.close();
}

void remove_NonPositive_entries(ColumnVector& Voxdata){  
//Zero, Negative Entries can be obtained from spline interpolation 
  	int pos; 
  	float MinS=Voxdata.Minimum1(pos); 
  	float MaxS=Voxdata.Maximum();
	if (MinS<=0 && MaxS>0){  
	//when there are some non-positive entries, but not all are zero
    		vector<int> minpositions;
	    	while (MinS<=0){
	      		minpositions.push_back(pos);
	      		Voxdata(pos)=MaxS;    //temporarilly make the non-positive values Max
	      		MinS=Voxdata.Minimum1(pos);
	    	}
	    	MinS=Voxdata.Minimum(); //Now find the Minimum of positive entries
	    	for (unsigned int i=0; i<minpositions.size(); i++)
	      		Voxdata(minpositions[i])=MinS; //Replace non-positive entries with that minimum
  	}
}

//prepare the structures for copy all neccesary data to FIT on GPU
void prepare_data_gpu_FIT(	//INPUT
				const Matrix				datam,
				const Matrix				bvecs,
				const Matrix				bvals,
				const Matrix	 			gradm, 
				//OUTPUT
				vector<ColumnVector>&			datam_vec,
				vector<Matrix>&				bvecs_vec,
				vector<Matrix>&				bvals_vec,
				thrust::host_vector<float>&   		datam_host,	//data prepared for copy to GPU
				thrust::host_vector<float>&		bvecs_host,				
				thrust::host_vector<float>&		bvals_host,
				thrust::host_vector<double>&		alpha_host,
				thrust::host_vector<double>&		beta_host,
				thrust::host_vector<float>&		params_host,
				thrust::host_vector<float>&		tau_host)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	int nvox = datam.Ncols(); 
	int ndirections = datam.Nrows(); 

	datam_vec.resize(nvox);
	datam_host.resize(nvox*ndirections); 
	for(int vox=0;vox<nvox;vox++){
		ColumnVector voxdata;
		voxdata=datam.Column(vox+1);
     		if(opts.rician.value()) remove_NonPositive_entries(voxdata); //So that log(data) does not give infinity in the likelihood
		datam_vec[vox]=voxdata;
		for(int j=0;j<ndirections;j++){
			datam_host[vox*ndirections+j]=voxdata(j+1);
		}
	}

	if (opts.grad_file.set()){
		bvecs_vec.resize(nvox);
		bvals_vec.resize(nvox);
		bvecs_host.resize(nvox*bvecs.Nrows()*bvecs.Ncols());
		bvals_host.resize(nvox*bvals.Ncols());
		alpha_host.resize(nvox*bvecs.Ncols());
		beta_host.resize(nvox*bvecs.Ncols());
	}else{
		bvecs_vec.resize(1);
		bvals_vec.resize(1);
		bvecs_host.resize(1*bvecs.Nrows()*bvecs.Ncols());
		bvals_host.resize(1*bvals.Ncols());
		alpha_host.resize(1*bvecs.Ncols());
		beta_host.resize(1*bvecs.Ncols());
	}

	ColumnVector alpha,beta;

	if (opts.grad_file.set()){
		for(int vox=0;vox<nvox;vox++){
			correct_bvals_bvecs(bvals,bvecs, gradm.Column(vox+1),bvals_vec[vox],bvecs_vec[vox]); //correct for gradient nonlinearities
 			MISCMATHS::cart2sph(bvecs_vec[vox],alpha,beta);
			
			for(int dir=0;dir<ndirections;dir++){
				bvecs_host[vox*ndirections*3+dir] = bvecs_vec[vox](1,dir+1);
				bvecs_host[vox*ndirections*3+ndirections+dir] = bvecs_vec[vox](2,dir+1);
				bvecs_host[vox*ndirections*3+ndirections*2+dir] = bvecs_vec[vox](3,dir+1);
				bvals_host[vox*ndirections+dir] = bvals_vec[vox](1,dir+1);

				alpha_host[vox*ndirections+dir] = alpha(dir+1);
        			beta_host[vox*ndirections+dir] = beta(dir+1);
			}
		}
		
	}else{
 		MISCMATHS::cart2sph(bvecs,alpha,beta);
		bvecs_vec[0]=bvecs;
		bvals_vec[0]=bvals;
		for(int dir=0;dir<ndirections;dir++){
			bvecs_host[dir] = bvecs(1,dir+1);
			bvecs_host[ndirections+dir] = bvecs(2,dir+1);
			bvecs_host[ndirections*2+dir] = bvecs(3,dir+1);
        		bvals_host[dir] = bvals(1,dir+1);
			
			alpha_host[dir] = alpha(dir+1);
        		beta_host[dir] = beta(dir+1);
		}
	}
	
	int nfib= opts.nfibres.value();
	int nparams;

	if(opts.f0.value()) nparams=3+nfib*3;
	else nparams=2+nfib*3;	
	if(opts.modelnum.value()>=2) nparams++;

	params_host.resize(nvox*nparams);
	tau_host.resize(nvox);
}

//prepare the structures for copy all neccesary data to FIT on GPU when is repeated because f0. Only some voxels
void prepare_data_gpu_FIT_repeat(	//INPUT
					thrust::host_vector<float>   		datam_host,	
					thrust::host_vector<float>		bvecs_host,				
					thrust::host_vector<float>		bvals_host,
					thrust::host_vector<int>		vox_repeat,
					int					nrepeat,
					int					ndirections,
					//OUTPUT
					vector<ColumnVector>&			datam_repeat_vec,
					vector<Matrix>&				bvecs_repeat_vec,
					vector<Matrix>&				bvals_repeat_vec,
					thrust::host_vector<float>&   		datam_repeat_host,	//data prepared for copy to GPU
					thrust::host_vector<float>&		bvecs_repeat_host,				
					thrust::host_vector<float>&		bvals_repeat_host,
					thrust::host_vector<float>&		params_repeat_host)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	ColumnVector datam(ndirections);
	Matrix	bvecs(3,ndirections);
	Matrix	bvals(1,ndirections);

	datam_repeat_vec.resize(nrepeat);
	datam_repeat_host.resize(nrepeat*ndirections); 
	
	if (opts.grad_file.set()){
		bvecs_repeat_vec.resize(nrepeat);
		bvals_repeat_vec.resize(nrepeat);
		bvecs_repeat_host.resize(nrepeat*3*ndirections);
		bvals_repeat_host.resize(nrepeat*ndirections);
	}else{
		bvecs_repeat_vec.resize(1);
		bvals_repeat_vec.resize(1);
		bvecs_repeat_host.resize(1*3*ndirections);
		bvals_repeat_host.resize(1*ndirections);
	}

	
	for(int vox=0;vox<nrepeat;vox++){
		for(int dir=0;dir<ndirections;dir++){
			datam(dir+1)= datam_host[vox_repeat[vox]*ndirections+dir]; 
			datam_repeat_host[vox*ndirections+dir]=datam_host[vox_repeat[vox]*ndirections+dir];
		}
		datam_repeat_vec[vox]=datam;
	}

	if (opts.grad_file.set()){
		for(int vox=0;vox<nrepeat;vox++){
			for(int dir=0;dir<ndirections;dir++){
				bvecs_repeat_host[vox*ndirections*3+dir] = bvecs_host[vox_repeat[vox]*ndirections*3+dir];
				bvecs_repeat_host[vox*ndirections*3+ndirections+dir] = bvecs_host[vox_repeat[vox]*ndirections*3+ndirections+dir];
				bvecs_repeat_host[vox*ndirections*3+ndirections*2+dir] = bvecs_host[vox_repeat[vox]*ndirections*3+ndirections*2+dir];
				bvals_repeat_host[vox*ndirections+dir] = bvals_host[vox_repeat[vox]*ndirections+dir];
			
				bvecs(1,dir+1)= bvecs_host[vox_repeat[vox]*ndirections*3+dir];
				bvecs(2,dir+1)= bvecs_host[vox_repeat[vox]*ndirections*3+ndirections+dir];
				bvecs(3,dir+1)= bvecs_host[vox_repeat[vox]*ndirections*3+ndirections*2+dir];
				bvals(1,dir+1)= bvals_host[vox_repeat[vox]*ndirections+dir];
			}
			bvecs_repeat_vec[vox]=bvecs;
			bvals_repeat_vec[vox]=bvals;
		}
	}else{
		for(int dir=0;dir<ndirections;dir++){
			bvecs_repeat_host[dir] = bvecs_host[dir];
			bvecs_repeat_host[ndirections+dir] = bvecs_host[ndirections+dir];
			bvecs_repeat_host[ndirections*2+dir] = bvecs_host[ndirections*2+dir];
			bvals_repeat_host[dir] = bvals_host[dir];
			
			bvecs(1,dir+1)= bvecs_host[dir];
			bvecs(2,dir+1)= bvecs_host[ndirections+dir];
			bvecs(3,dir+1)= bvecs_host[ndirections*2+dir];
			bvals(1,dir+1)= bvals_host[dir];
		}
		bvecs_repeat_vec[0]=bvecs;
		bvals_repeat_vec[0]=bvals;
	}
	
	int nfib= opts.nfibres.value();
	int nparams;

	nparams=2+nfib*3;	
	if(opts.modelnum.value()>=2) nparams++;

	params_repeat_host.resize(nrepeat*nparams);
}


void mix_params(	//INPUT
			thrust::host_vector<float>   		params_repeat_host,
			thrust::host_vector<int>		vox_repeat,
			int					nrepeat,
			int					nvox,
			//INPUT-OUTPUT
			thrust::device_vector<float>&   	params_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	int nfib= opts.nfibres.value();
	int nparams = 2+3*opts.nfibres.value();
	if(opts.modelnum.value()>=2) nparams++;

	thrust::host_vector<float> params_host;
	params_host.resize(nvox*(nparams+1));
	thrust::copy(params_gpu.begin(), params_gpu.end(), params_host.begin());	

	for(int vox=0;vox<nrepeat;vox++){
		for(int par=0;par<nparams;par++){
			params_host[vox_repeat[vox]*(nparams+1)+par] = params_repeat_host[vox*nparams+par]; //(nparams+1) to count f0
		}
		params_host[vox_repeat[vox]*(nparams+1)+nparams] = 0.001;	//pvmf0=0.001
	}
	thrust::copy(params_host.begin(), params_host.end(), params_gpu.begin());	
}

void prepare_data_gpu_MCMC(	//INPUT
				int 					nvox,
				int					ndirections,
				int 					nfib,
				//OUTPUT
				thrust::host_vector<double>&		signals_host,
				thrust::host_vector<double>&		isosignals_host,
				thrust::host_vector<FibreGPU>& 		fibres_host,
				thrust::host_vector<MultifibreGPU>& 	multifibres_host)
{ 	
	signals_host.resize(nvox*nfib*ndirections);
	isosignals_host.resize(nvox*ndirections);	
	fibres_host.resize(nvox*nfib);	
	multifibres_host.resize(nvox);
}

void prepare_data_gpu_MCMC_record(	//INPUT
					int 						nvox,
					//OUTPUT
					thrust::device_vector<float>&			rf0_gpu,
					thrust::device_vector<float>&			rtau_gpu,
					thrust::device_vector<float>&			rs0_gpu,
					thrust::device_vector<float>&			rd_gpu,
					thrust::device_vector<float>&			rdstd_gpu,
					thrust::device_vector<float>&			rR_gpu,
					thrust::device_vector<float>&			rth_gpu,
					thrust::device_vector<float>&			rph_gpu,
					thrust::device_vector<float>&			rf_gpu)
{ 	
	xfibresOptions& opts = xfibresOptions::getInstance();

	int nfib = opts.nfibres.value();	
	int nsamples = (opts.njumps.value()/opts.sampleevery.value());   
	
	if(opts.f0.value()) rf0_gpu.resize(nvox*nsamples); 
	if(opts.rician.value()) rtau_gpu.resize(nvox*nsamples);  
	rs0_gpu.resize(nvox*nsamples);  
	rd_gpu.resize(nvox*nsamples);
	if(opts.modelnum.value()>=2) rdstd_gpu.resize(nvox*nsamples);  
	if(opts.modelnum.value()==3) rR_gpu.resize(nvox*nsamples); 
	rth_gpu.resize(nvox*nsamples*nfib);  
	rph_gpu.resize(nvox*nsamples*nfib);  
	rf_gpu.resize(nvox*nsamples*nfib);  
}

void resize_structures(		//INPUT
				int					nVOX_multiple,
				int 					ndirections,
				//OUTPUT
				thrust::device_vector<float>&   	datam_gpu,
				thrust::device_vector<float>&		params_gpu,
				thrust::device_vector<float>&		tau_gpu,
				thrust::device_vector<float>&		bvals_gpu,				
				thrust::device_vector<double>&		alpha_gpu,
				thrust::device_vector<double>&		beta_gpu,
				thrust::device_vector<hiprandState>&	randStates_gpu)
{
	xfibresOptions& opts = xfibresOptions::getInstance();
	int nfib= opts.nfibres.value();
	int nparams;
	nparams=2+nfib*3;	
	if(opts.modelnum.value()>=2) nparams++;


	datam_gpu.resize(nVOX_multiple*ndirections);
	params_gpu.resize(nVOX_multiple*nparams);
	tau_gpu.resize(nVOX_multiple);

	if (opts.grad_file.set()){
		bvals_gpu.resize(nVOX_multiple*ndirections);
		alpha_gpu.resize(nVOX_multiple*ndirections);
		beta_gpu.resize(nVOX_multiple*ndirections);
	}else{
		bvals_gpu.resize(1*ndirections);
		alpha_gpu.resize(1*ndirections);
		beta_gpu.resize(1*ndirections);
	}

	randStates_gpu.resize(nVOX_multiple);
}

void record_finish_voxels(	//INPUT
				thrust::device_vector<float>&			rf0_gpu,
				thrust::device_vector<float>&			rtau_gpu,
				thrust::device_vector<float>&			rs0_gpu,
				thrust::device_vector<float>&			rd_gpu,
				thrust::device_vector<float>&			rdstd_gpu,
				thrust::device_vector<float>&			rR_gpu,
				thrust::device_vector<float>&			rth_gpu,
				thrust::device_vector<float>&			rph_gpu,
				thrust::device_vector<float>&			rf_gpu,
				int 						nvox,
				int						nVOX_multiple,
				int						idpart)
{
	xfibresOptions& opts = xfibresOptions::getInstance();

	int nfib = opts.nfibres.value();	
	int nsamples = (opts.njumps.value()/opts.sampleevery.value());   

	thrust::host_vector<float> rf0_host,rtau_host,rs0_host,rd_host,rdstd_host,rR_host,rth_host,rph_host,rf_host;

	rf0_host.resize(nVOX_multiple*nsamples);
	rtau_host.resize(nVOX_multiple*nsamples);
	rs0_host.resize(nVOX_multiple*nsamples);
	rd_host.resize(nVOX_multiple*nsamples);
	if(opts.modelnum.value()>=2) rdstd_host.resize(nVOX_multiple*nsamples);
	if(opts.modelnum.value()==3) rR_host.resize(nVOX_multiple*nsamples);
	rth_host.resize(nVOX_multiple*nfib*nsamples);
	rph_host.resize(nVOX_multiple*nfib*nsamples);
	rf_host.resize(nVOX_multiple*nfib*nsamples);

	if(opts.f0.value()) thrust::copy(rf0_gpu.begin(), rf0_gpu.end(), rf0_host.begin());
	if(opts.rician.value()) thrust::copy(rtau_gpu.begin(), rtau_gpu.end(), rtau_host.begin());
	thrust::copy(rs0_gpu.begin(), rs0_gpu.end(), rs0_host.begin());
	thrust::copy(rd_gpu.begin(), rd_gpu.end(), rd_host.begin());
	if(opts.modelnum.value()>=2) thrust::copy(rdstd_gpu.begin(), rdstd_gpu.end(), rdstd_host.begin());
	if(opts.modelnum.value()==3) thrust::copy(rR_gpu.begin(), rR_gpu.end(), rR_host.begin());
	thrust::copy(rth_gpu.begin(), rth_gpu.end(), rth_host.begin());
	thrust::copy(rph_gpu.begin(), rph_gpu.end(), rph_host.begin());
	thrust::copy(rf_gpu.begin(), rf_gpu.end(), rf_host.begin());	

	Samples samples(nvox,nsamples);

	float ard,arf0,artau,ardstd,arR,ars0;	
	float *arth = new float[nfib];
    	float *arph = new float[nfib]; 
    	float *arf = new float[nfib];

	for(int vox=0;vox<nvox;vox++){
		for(int rec=0;rec<nsamples;rec++){	
			ard=rd_host[(vox*nsamples)+rec];
			if(opts.f0.value()){	
				arf0=rf0_host[(vox*nsamples)+rec];
			}

			if(opts.rician.value()){	
				artau=rtau_host[(vox*nsamples)+rec];
			}

			if(opts.modelnum.value()>=2){	
				ardstd=rdstd_host[(vox*nsamples)+rec];
			}
			if(opts.modelnum.value()==3){	
				arR=rR_host[(vox*nsamples)+rec];
			}
		
			ars0=rs0_host[(vox*nsamples)+rec];

			for(int j=0;j<nfib;j++){
				arth[j]=rth_host[(vox*nfib*nsamples)+(j*nsamples)+rec];
				arph[j]=rph_host[(vox*nfib*nsamples)+(j*nsamples)+rec];
				arf[j]=rf_host[(vox*nfib*nsamples)+(j*nsamples)+rec];

			}
			samples.record(ard,arf0,artau,ardstd,arR,ars0,arth,arph,arf,vox+1,rec+1);
		}	
		samples.finish_voxel(vox+1);
   	}
	samples.save(idpart);
}

