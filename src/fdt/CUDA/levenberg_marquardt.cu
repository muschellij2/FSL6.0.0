#include "hip/hip_runtime.h"
/*  levenberg_marquardt.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#ifndef __LEVENBERG
#define __LEVENBERG

#include "solver_mult_inverse.cu"
#include "diffmodels.cuh"
#include "options.h"

//CPU version in nonlin.h
__device__ const double EPS_gpu = 2.0e-16;       	//Losely based on NRinC 20.1

//CPU version in nonlin.cpp
__device__ inline bool zero_cf_diff_conv(double* cfo,double* cfn,double* cftol){
  	return(2.0*fabs(*cfo-*cfn) <= *cftol*(fabs(*cfo)+fabs(*cfn)+EPS_gpu));
}

__device__ void levenberg_marquardt_PVM_single_gpu(	//INPUT
							const float*		mydata, 
							const float*		bvecs, 
							const float*		bvals, 
							const int		ndirections,
							const int		nfib,
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idSubVOX,
							float* 			step,		//shared memory
							float*			grad,           //shared memory     	          
						   	float* 			hess,		//shared memory
							float* 			inverse,	//shared memory
							double* 		pcf,		//shared memory
							double* 		ncf,		//shared memory
							double* 		lambda,		//shared memory
							double* 		cftol,		//shared memory
							double* 		ltol,		//shared memory
							double* 		olambda,	//shared memory
							int* 			success,    	//shared memory
							int* 			end,    	//shared memory
							float*			J,		//shared memory
							float*			reduction,	//shared memory
							float* 			fs,		//shared memory
						  	float*			x,		//shared memory
							float* 			_d,		//shared memory
						  	float* 			sumf,		//shared memory
							float*			C,		//shared memory
							float*			el,		//shared memory
							int*			indx,		//shared memory
							//INPUT-OUTPUT
							float*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idSubVOX==0){
		*end=false;
   		*lambda=0.1;
   		*cftol=1.0e-8;
   		*ltol=1.0e20;                  
   		*success = true;               
   		*olambda = 0.0;              
   		*ncf=0;  
	}

   	cf_PVM_single(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,pcf);  
	__syncthreads();

   	while (!(*success&&niter++>=maxiter)){ 	//if success we don't increase niter (first condition is true)
						//function cost has been decreased, we have advanced.
   		if(*success){
    			grad_PVM_single(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,x,_d,sumf,grad); 
			__syncthreads(); 
    			hess_PVM_single(myparams,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,x,_d,sumf,hess);  
    		}

		if(idSubVOX==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=*lambda-*olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,C,el,indx,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}
		
		__syncthreads();
   		cf_PVM_single(step,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,ncf); 

		if(idSubVOX==0){
   			if (*success = (*ncf < *pcf)){ 
				*olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			*lambda=*lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					*end=true;
				}
				*pcf=*ncf;
    			}else{
				*olambda=*lambda;
				*lambda=*lambda*10.0;
				if(*lambda> *ltol){ 
					*end=true;
				}
			}
    		}	
		__syncthreads();
		if(*end) return;		
   	}
}

__device__ void levenberg_marquardt_PVM_single_c_gpu(	//INPUT
							const float*		mydata, 
							const float*		bvecs, 
							const float*		bvals,
							const int		ndirections, 
							const int		nfib,
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idSubVOX,
							float* 			step,		//shared memory
							float*			grad,           //shared memory     	          
						   	float* 			hess,		//shared memory
							float* 			inverse,	//shared memory
							double* 		pcf,		//shared memory
							double* 		ncf,		//shared memory
							double* 		lambda,		//shared memory
							double* 		cftol,		//shared memory
							double* 		ltol,		//shared memory
							double* 		olambda,	//shared memory
							int* 			success,    	//shared memory
							int* 			end,    	//shared memory
							float*			J,		//shared memory
							float*			reduction,	//shared memory
							float* 			fs,		//shared memory
							float*			f_deriv,	//shared memory
						  	float*			x,		//shared memory
							float* 			_d,		//shared memory
						  	float* 			sumf,		//shared memory
							float*			C,		//shared memory
							float*			el,		//shared memory
							int*			indx,		//shared memory
							//INPUT-OUTPUT
							float*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idSubVOX==0){
		*end=false;
   		*lambda=0.1;
   		*cftol=1.0e-8;
   		*ltol=1.0e20;                  
   		*success = true;               
   		*olambda = 0.0;              
   		*ncf=0;  
	}
			
	cf_PVM_single_c(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,pcf);  
	__syncthreads();
	
   	while (!(*success&&niter++ >= maxiter)){ 	//if success we don't increase niter (first condition is true)
							//function cost has been decreased, we have advanced.
   		if(*success){
			grad_PVM_single_c(myparams,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,f_deriv,x,_d,sumf,grad);  
			__syncthreads();
    			hess_PVM_single_c(myparams,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,J,reduction,fs,f_deriv,x,_d,sumf,hess);  
    		}

		if(idSubVOX==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=*lambda-*olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,C,el,indx,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}

		__syncthreads();
   		cf_PVM_single_c(step,mydata,bvecs,bvals,ndirections,nfib,nparams,m_include_f0,idSubVOX,reduction,fs,x,_d,sumf,ncf); 

		if(idSubVOX==0){
   			if (*success = (*ncf < *pcf)) {
				*olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			*lambda=*lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					*end=true;
				}
				*pcf=*ncf;
    			}else{
				*olambda=*lambda;
				*lambda=*lambda*10.0;
				if(*lambda> *ltol){ 
					*end=true;
				}
    			}
		}
		__syncthreads();
		if(*end) return;		
   	}
}


__device__ void levenberg_marquardt_PVM_multi_gpu(	//INPUT
							const float*		mydata, 
							const float*		bvecs, 
							const float*		bvals, 
							const float		R,
							const float		invR,
							const int		ndirections,
							const int		nfib,
							const int 		nparams,
							const bool 		m_include_f0,
							const int		idSubVOX,
							const int		Gamma_for_ball_only,
							float* 			step,		//shared memory
							float*			grad,           //shared memory     	          
						   	float* 			hess,		//shared memory
							float* 			inverse,	//shared memory
							double* 		pcf,		//shared memory
							double* 		ncf,		//shared memory
							double* 		lambda,		//shared memory
							double* 		cftol,		//shared memory
							double* 		ltol,		//shared memory
							double* 		olambda,	//shared memory
							int* 			success,    	//shared memory
							int* 			end,    	//shared memory
							float*			J,		//shared memory
							float*			reduction,	//shared memory
							float* 			fs,		//shared memory
						  	float*			x,		//shared memory
							float* 			_a,		//shared memory
							float* 			_b,		//shared memory
						  	float* 			sumf,		//shared memory
							float*			C,		//shared memory
							float*			el,		//shared memory
							int*			indx,		//shared memory
							//INPUT-OUTPUT
							float*			myparams)	//shared memory
{
	int niter=0; 
	int maxiter=200;

   	if(idSubVOX==0){
		*end=false;
   		*lambda=0.1;
   		*cftol=1.0e-8;
   		*ltol=1.0e20;                  
   		*success = true;               
   		*olambda = 0.0;              
   		*ncf=0;  
	}

	cf_PVM_multi(myparams,mydata,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,idSubVOX,Gamma_for_ball_only,reduction,fs,x,_a,_b,sumf,pcf);  
	__syncthreads();
	
   	while (!(*success&&niter++ >= maxiter)){ 	//if success we don't increase niter (first condition is true)
							//function cost has been decreased, we have advanced.
   		if(*success){
			grad_PVM_multi(myparams,mydata,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,
			idSubVOX,Gamma_for_ball_only,J,reduction,fs,x,_a,_b,sumf,grad);  

			__syncthreads(); 
    			hess_PVM_multi(myparams,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,idSubVOX,Gamma_for_ball_only,J,reduction,fs,x,_a,_b,sumf,hess);  
    		}

		if(idSubVOX==0){
    			for (int i=0; i<nparams; i++) {                         
				hess[(i*nparams)+i]+=*lambda-*olambda;	//Levenberg LM_L
    			}

    			solver(hess,grad,nparams,C,el,indx,inverse);

    			for (int i=0;i<nparams;i++){
				step[i]=-inverse[i];		
    			}

   			for(int i=0;i<nparams;i++){
				step[i]=myparams[i]+step[i];
   			}
		}

		__syncthreads();
   		cf_PVM_multi(step,mydata,bvecs,bvals,R,invR,ndirections,nfib,nparams,m_include_f0,idSubVOX,Gamma_for_ball_only,reduction,fs,x,_a,_b,sumf,ncf); 

		if(idSubVOX==0){
   			if (*success = (*ncf < *pcf)) {
				*olambda = 0.0;
        			for(int i=0;i<nparams;i++){
					myparams[i]=step[i];
   				}
        			*lambda=*lambda/10.0;

				if (zero_cf_diff_conv(pcf,ncf,cftol)){
					*end=true;
				}
				*pcf=*ncf;
    			}else{
				*olambda=*lambda;
				*lambda=*lambda*10.0;
				if(*lambda> *ltol){ 
					*end=true;
				}
    			}
		}
		__syncthreads();
		if(*end) return;				
   	}
}
#endif
