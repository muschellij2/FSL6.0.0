#include "hip/hip_runtime.h"
/*  runmcmc_kernels.cu

    Tim Behrens, Saad Jbabdi, Stam Sotiropoulos, Moises Hernandez  - FMRIB Image Analysis Group

    Copyright (C) 2005 University of Oxford  */

/*  Part of FSL - FMRIB's Software Library
    http://www.fmrib.ox.ac.uk/fsl
    fsl@fmrib.ox.ac.uk
    
    Developed at FMRIB (Oxford Centre for Functional Magnetic Resonance
    Imaging of the Brain), Department of Clinical Neurology, Oxford
    University, Oxford, UK
    
    
    LICENCE
    
    FMRIB Software Library, Release 5.0 (c) 2012, The University of
    Oxford (the "Software")
    
    The Software remains the property of the University of Oxford ("the
    University").
    
    The Software is distributed "AS IS" under this Licence solely for
    non-commercial use in the hope that it will be useful, but in order
    that the University as a charitable foundation protects its assets for
    the benefit of its educational and research purposes, the University
    makes clear that no condition is made or to be implied, nor is any
    warranty given or to be implied, as to the accuracy of the Software,
    or that it will be suitable for any particular purpose or for use
    under any specific conditions. Furthermore, the University disclaims
    all responsibility for the use which is made of the Software. It
    further disclaims any liability for the outcomes arising from using
    the Software.
    
    The Licensee agrees to indemnify the University and hold the
    University harmless from and against any and all claims, damages and
    liabilities asserted by third parties (including claims for
    negligence) which arise directly or indirectly from the use of the
    Software or the sale of any products based on the Software.
    
    No part of the Software may be reproduced, modified, transmitted or
    transferred in any form or by any means, electronic or mechanical,
    without the express permission of the University. The permission of
    the University is not required if the said reproduction, modification,
    transmission or transference is done without financial return, the
    conditions of this Licence are imposed upon the receiver of the
    product, and all original and amended source code is included in any
    transmitted product. You may be held legally responsible for any
    copyright infringement that is caused or encouraged by your failure to
    abide by these terms and conditions.
    
    You are not permitted under this Licence to use this Software
    commercially. Use for which any financial return is received shall be
    defined as commercial use, and includes (1) integration of all or part
    of the source code or the Software into a product for sale or license
    by or on behalf of Licensee to third parties or (2) use of the
    Software or any derivative of it for research with the final aim of
    developing software products for sale or license to a third party or
    (3) use of the Software or any derivative of it for research with the
    final aim of developing non-software products for sale or license to a
    third party, or (4) use of the Software to provide any service to an
    external organisation for which payment is received. If you are
    interested in using the Software commercially, please contact Oxford
    University Innovation ("OUI"), the technology transfer company of the
    University, to negotiate a licence. Contact details are:
    Innovation@innovation.ox.ac.uk quoting reference DE/9564. */

#include <iostream>
#include <fstream>
#include <stdio.h>
#include "fibre_gpu.h"
#include <math.h>
#include <string.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <options.h>

#define maxfloat 1e10
#define UPPERDIFF 0.005

extern "C" __global__ void setup_randoms_kernel(hiprandState* randstate, double seed, int nvox){
	int id = blockIdx.x*THREADS_BLOCK_RAND+threadIdx.x;
	if(id<nvox){
		hiprand_init(seed,id,0,&randstate[id]);
	}
}

__device__ inline void propose(float* param, float* old, float prop, hiprandState* localrandState){
	*old=*param;
	*param = *param + hiprand_normal(localrandState)*prop;
}
__device__ inline void reject(float* param, float* prior, float* old, float* m_prior_en, float* m_old_prior_en, float* m_energy, float* m_old_energy, int* rej){
	*param=old[0];
	*prior=old[1];
	*m_prior_en=*m_old_prior_en;
	*rej=*rej+1;
	//restore_energy()
      	*m_energy=*m_old_energy;
}
__device__ inline void rejectF(float* param, float* prior, float* old, float* m_prior_en, float* m_old_prior_en, float* fm_prior_en, float* fm_old_prior_en, float* m_energy, float* m_old_energy, int* rej){
	*param=old[0];
	*prior=old[1];
	*fm_prior_en=*fm_old_prior_en;
	*m_prior_en=*m_old_prior_en;
	*rej=*rej+1;
	//restore_energy()
      	*m_energy=*m_old_energy;
}				
__device__ inline void getfsum(float* fsum, float* m_f, float m_f0, int nfib){
	*fsum=m_f0;
	for(int f=0;f<nfib;f++){  
		*fsum = *fsum + m_f[f];
	}
}
__device__ inline bool compute_test_energy(float *m_energy, float* m_old_energy, float m_prior_en, float m_likelihood_en, hiprandState* localrandState){
	*m_old_energy=*m_energy;
      	*m_energy=m_prior_en+m_likelihood_en;

	double tmp=exp(double(*m_old_energy-*m_energy));
	return (tmp>hiprand_uniform(localrandState));
}
__device__ inline void compute_signal(double *signals,double *oldsignals,float mbvals,float* m_d, float* m_dstd, float* m_R, double angtmp, int model){
	*oldsignals=*signals;
	if(model==1 || (*m_dstd<1e-5 && model==2)){   	
		*signals=exp(double(-*m_d*mbvals*angtmp));
	}else if(model==2){
		//float dbeta= *m_d/(*m_dstd**m_dstd);
	 	//float dalpha= *m_d*dbeta;   
		//*signals=expf((logf(dbeta/(dbeta+mbvals*angtmp))*dalpha);   
		float sig2=*m_dstd**m_dstd;
	 	float dalpha=*m_d**m_d/sig2;      
		*signals=exp(log(double(*m_d/(*m_d + mbvals*angtmp*sig2)))*dalpha); // more stable
	}else if(model==3){
		float invR=1.0/(2.0**m_R+1.0);
	   	*signals=exp(-mbvals*3**m_d*invR*((1-*m_R)*angtmp+*m_R));
       	}
}
__device__ inline void compute_iso_signal(double *isosignals,double *oldisosignals, float mbvals,float* m_d, float* m_dstd, int model){
	*oldisosignals=*isosignals;
	if(model==1 || *m_dstd<1e-5){
	 	*isosignals=exp(double(-m_d[0]*mbvals));	
	}else if(model>=2){
		//float dbeta= *m_d/(*m_dstd**m_dstd);
	  	//float dalpha= *m_d*dbeta;
		//*isosignals=expf(logf(dbeta/(dbeta+mbvals))*dalpha);
		float sig2=*m_dstd**m_dstd;
		float dalpha=*m_d**m_d/sig2;	
		*isosignals=exp(log(double(*m_d/(*m_d+mbvals*sig2)))*dalpha); // more numerically stable
	}
}
__device__ inline void restore_signals(double* signals, double* oldsignals, int idVOX, int idSubVOX, int mydirs, int nfib, int ndirections){
	for(int f=0;f<nfib;f++){
		for(int i=0; i<mydirs; i++){
			int pos = idVOX*ndirections*nfib + f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
			signals[pos] = oldsignals[pos];
		}	
	}
}
__device__ inline void restore_isosignals(double* isosignals, double* oldisosignals, int idVOX, int idSubVOX, int mydirs, int ndirections){
	for(int i=0; i<mydirs; i++){
		int pos = idVOX*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
		isosignals[pos]=oldisosignals[pos];
	}
}
__device__ inline void restore_angtmp_signals(double* signals, double* oldsignals,double* angtmp, double* oldangtmp, int idVOX, int idSubVOX, int mydirs, int nfib, int fibre, int ndirections){
	for(int i=0; i<mydirs; i++){
		int pos = idVOX*ndirections*nfib + fibre*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
		int pos2 = idVOX*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
		angtmp[pos]=oldangtmp[pos2];
		signals[pos] = oldsignals[pos];	
	}
}
__device__  inline void compute_prior(float *m_prior_en, float *m_prior_en_old,float* m_d_prior,float* m_S0_prior,float *m_prior_enf, float* m_f0_prior, float* m_tau_prior, float* m_dstd_prior, float* m_R_prior, int nfib){			
        *m_prior_en_old=*m_prior_en;
	*m_prior_en=*m_d_prior+*m_S0_prior+*m_dstd_prior+*m_R_prior+*m_tau_prior+*m_f0_prior;
	for(int f=0;f<nfib;f++){
		*m_prior_en=*m_prior_en+m_prior_enf[f];
	}	
}

__device__ inline float logIo(const float& x){
    	float y,b;
    	b= fabs(x);
    	if (b<3.75){
      		float a=x/3.75;
      		a*=a;
      		//Bessel function evaluation
					y=1.0+a*(3.5156229+a*(3.0899424+a*(1.2067492+a*(0.2659732+a*(0.0360768+a*0.0045813)))));
      		y=log(double(y));
    	}else{
      		float a=3.75/b; 
      		//Bessel function evaluation
      		//y=(expf(b)/sqrt(b))*(0.39894228+a*(0.01328592+a*(0.00225319+a*(-0.00157565+a*(0.00916281+a*(-0.02057706+a*(0.02635537+a*(-0.01647633+a*0.00392377))))))));
      		//Logarithm of Bessel function

		y=b+log(double((0.39894228+a*(0.01328592+a*(0.00225319+a*(-0.00157565+a*(0.00916281+a*(-0.02057706+a*(0.02635537+a*(-0.01647633+a*0.00392377))))))))/sqrt(b)));
    	}

    	return y;
}

__device__ inline void compute_likelihood(int idSubVOX,float* m_S0,float *m_likelihood_en,float *m_f,double *signals,double *isosignals,const float *mdata,float* fsum,double *reduction, float* m_f0, const bool rician, float* m_tau,int mydirs, int ndirections, int nfib){
	
	double pred;
	int pos;

	reduction[idSubVOX]=0;
	for(int i=0; i<mydirs; i++){
		pred=0;
	      	for(int f=0;f<nfib;f++){
			pos = f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
			pred= pred+m_f[f]*signals[pos];
	     	}
		pos = idSubVOX + i*THREADS_VOXEL_MCMC;
		pred= *m_S0*(pred+(1-*fsum)*isosignals[pos]+*m_f0); //F0
	
		if(!rician){
			double diff = mdata[pos]-pred;
			reduction[idSubVOX] = reduction[idSubVOX]+(diff*diff);
		}else{
			pred= log(mdata[pos])+(-0.5**m_tau*(mdata[pos]*mdata[pos]+pred*pred)+logIo(*m_tau*pred*mdata[pos]));  
			reduction[idSubVOX] = reduction[idSubVOX]+pred;
		}
	}

	__syncthreads();

	unsigned int s2=THREADS_VOXEL_MCMC;
	for(unsigned int s=THREADS_VOXEL_MCMC>>1; s>0; s>>=1) {
		if((s2%2)&&(idSubVOX==(s-1))) reduction[idSubVOX]= reduction[idSubVOX] + reduction[idSubVOX + s +1]; 
        	if (idSubVOX < s){
            		reduction[idSubVOX] = reduction[idSubVOX] + reduction[idSubVOX + s];
       	 	}
		s2=s;
        	__syncthreads();
    	}
	if(idSubVOX==0){
		double sumsquares=0;
		sumsquares+=reduction[0];
		if(!rician){ 
		 	*m_likelihood_en=(ndirections/2.0)*log(sumsquares/2.0);  
		}else{
			*m_likelihood_en= -ndirections*log(*m_tau)-sumsquares;
		}
	}
}
			  
extern "C" __global__ void init_Fibres_Multifibres_kernel(	//INPUT
								const float*			datam,
								const float*			params,
								const float*			tau,
								const float*			bvals,
								const double*			alpha,
								const double*			beta,
								const float			R_priormean,
								const float			R_priorstd,
								const float			R_priorfudge,		
								const int			ndirections,
								const int 			nfib,
								const int 			nparams_fit,
								const int 			model,
								const float 			fudgevalue,
								const bool			m_includef0,
								const bool			rician,
								const bool 			m_ardf0,	// opts.ardf0.value()
								const bool 			ard_value,	// opts.all_ard.value()
								const bool 			no_ard_value,	// opts.no_ard.value()
								const bool			gradnonlin,
								//TO USE
								double*				angtmp,
								//OUTPUT
								FibreGPU*			fibres,
								MultifibreGPU*			multifibres,
								double*				signals,
								double*				isosignals)
{
	int idSubVOX= threadIdx.x%THREADS_VOXEL_MCMC;
	int idVOX= (blockIdx.x*VOXELS_BLOCK_MCMC)+int(threadIdx.x/THREADS_VOXEL_MCMC);
	int idVOX_block =  threadIdx.x/THREADS_VOXEL_MCMC;
	bool leader = (idSubVOX==0);
	
	// there may be several voxels per block: VOXELS_BLOCK_MCMC 
	////////// DYNAMIC SHARED MEMORY ///////////				// each voxel:
	extern __shared__ double shared[];
	double* reduction = (double*)shared;					//THREADS_VOXEL_MCMC
	float* m_S0 = (float*) &reduction[THREADS_VOXEL_MCMC*VOXELS_BLOCK_MCMC];//1
	float* m_d = (float*) &m_S0[VOXELS_BLOCK_MCMC];				//1
	float* m_dstd =(float*) &m_d[VOXELS_BLOCK_MCMC];			//1
	float* m_R =(float*) &m_dstd[VOXELS_BLOCK_MCMC];			//1	
	float* m_f0 = (float*) &m_R[VOXELS_BLOCK_MCMC];				//1
	float* m_tau = (float*) &m_f0[VOXELS_BLOCK_MCMC];			//1
	float* m_th = (float*) &m_tau[VOXELS_BLOCK_MCMC];			//nfib
	float* m_ph = (float*) &m_th[VOXELS_BLOCK_MCMC*nfib];			//nfib
	float* m_f = (float*) &m_ph[VOXELS_BLOCK_MCMC*nfib];			//nfib
	float* fsum = (float*) &m_f[VOXELS_BLOCK_MCMC*nfib];			//1
	float* m_likelihood_en = (float*) &fsum[VOXELS_BLOCK_MCMC];		//1
	float* m_prior_en = (float*) &m_likelihood_en[VOXELS_BLOCK_MCMC];	//1
	int* posBV = (int*) &m_prior_en[VOXELS_BLOCK_MCMC];			//1
	////////// DYNAMIC SHARED MEMORY ///////////

	///// UPDATE shared memory pointers depending on the Id of a voxel within a block /////
										// each voxel:
	reduction = &reduction[idVOX_block*THREADS_VOXEL_MCMC];			//THREADS_VOXEL_MCMC 
	m_S0 = &m_S0[idVOX_block];						//1
	m_d = &m_d[idVOX_block];						//1
	m_dstd = &m_dstd[idVOX_block];						//1
	m_R =&m_R[idVOX_block];							//1
	m_f0 = &m_f0[idVOX_block];						//1
	m_tau = &m_tau[idVOX_block];						//1
	m_th = &m_th[idVOX_block*nfib];						//nfib
	m_ph = &m_ph[idVOX_block*nfib];						//nfib
	m_f = &m_f[idVOX_block*nfib];						//nfib
	fsum = &fsum[idVOX_block];						//1
	m_likelihood_en = &m_likelihood_en[idVOX_block];			//1
	m_prior_en = &m_prior_en[idVOX_block];					//1
	posBV = &posBV[idVOX_block];						//1
	/////////////////////////////////////////////////////////////////////////////////////////////
	
	// m_s0-params[0]	m_d-params[1]	m_f-m_th-m_ph-params[add+2,3,4,5, etc..]	m_f0-params[nparams-1]
	if(leader){
		if(gradnonlin) *posBV = (idVOX*ndirections);
		else *posBV = 0;

		*m_S0 = params[idVOX*nparams_fit];
		multifibres[idVOX].m_S0 = *m_S0;
		multifibres[idVOX].m_S0_prior = 0;
		multifibres[idVOX].m_S0_acc = 0;
		multifibres[idVOX].m_S0_rej = 0;
	
		*m_d=params[idVOX*nparams_fit+1];
		if(*m_d<0 || *m_d> UPPERDIFF) *m_d=2e-3;	//this is in xfibres...after fit
		multifibres[idVOX].m_d = *m_d;
		multifibres[idVOX].m_d_prior = 0;
		multifibres[idVOX].m_d_acc = 0;
		multifibres[idVOX].m_d_rej = 0;

		if(model>=2){ 
			*m_dstd=params[idVOX*nparams_fit+2];
			float upper_d_std=0.01;
			if (model==3) upper_d_std=0.004;
      if(*m_dstd<0 || *m_dstd>upper_d_std) *m_dstd=*m_d/10;   //this is in xfibres...after fit
			if (model==3){ 
				*m_R=R_priormean;	
			}else{ 
				*m_R=0;
			}
		}
		else *m_dstd = 0;
		multifibres[idVOX].m_dstd = *m_dstd;
		multifibres[idVOX].m_dstd_prior = 0;
		multifibres[idVOX].m_dstd_acc = 0;
		multifibres[idVOX].m_dstd_rej = 0;

		multifibres[idVOX].m_R = *m_R;
		multifibres[idVOX].m_R_prior = 0;
		multifibres[idVOX].m_R_acc = 0;
		multifibres[idVOX].m_R_rej = 0;

		if (m_includef0) *m_f0=params[idVOX*nparams_fit+nparams_fit-1];
		else *m_f0=0;
		multifibres[idVOX].m_f0 = *m_f0;
		multifibres[idVOX].m_f0_prior = 0;
		multifibres[idVOX].m_f0_acc = 0;
		multifibres[idVOX].m_f0_rej = 0;

		*m_tau = tau[idVOX];
		multifibres[idVOX].m_tau = *m_tau;
		multifibres[idVOX].m_tau_prior = 0;
		multifibres[idVOX].m_tau_acc = 0;
		multifibres[idVOX].m_tau_rej = 0;
	}
	__syncthreads();

	int mydirs = ndirections/THREADS_VOXEL_MCMC;
	int mod = ndirections%THREADS_VOXEL_MCMC;
	if(mod&&(idSubVOX<mod)) mydirs++;

	//------ Fibre constructor ------
	if(idSubVOX<nfib){
		int add=0;
		if(model>=2) add=1;		// if model 2 we have d_std and then 1 more parameter in position 2
		int pos = (idVOX*nfib)+idSubVOX;

		m_th[idSubVOX]=params[idVOX*nparams_fit+2+3*idSubVOX+1+add];
		fibres[pos].m_th = m_th[idSubVOX];
		fibres[pos].m_th_prop = 0.2;
		float m_th_prior = 0;
		fibres[pos].m_th_acc = 0;
		fibres[pos].m_th_rej = 0;
		
		//compute_th_prior();
	      	if(m_th[idSubVOX]==0){
			m_th_prior=0;
		}else{
			m_th_prior=-log(double(fabs(sin(double(m_th[idSubVOX]))/2)));
	      	}
		fibres[pos].m_th_prior = m_th_prior;
		
		float m_ph_prior=0;	//compute_ph_prior();
		m_ph[idSubVOX]=params[idVOX*nparams_fit+2+3*idSubVOX+2+add];
		fibres[pos].m_ph = m_ph[idSubVOX];
		fibres[pos].m_ph_prop = 0.2;
		fibres[pos].m_ph_prior = 0;	//compute_ph_prior();
		fibres[pos].m_ph_acc = 0;
		fibres[pos].m_ph_rej = 0;

		m_f[idSubVOX] = params[idVOX*nparams_fit+2+3*idSubVOX+add]; 
		fibres[pos].m_f=m_f[idSubVOX];
		fibres[pos].m_f_prop = 0.2;
		float m_f_prior = 0;
		fibres[pos].m_f_acc = 0;
		fibres[pos].m_f_rej = 0;
			
		if(idSubVOX==0){
			fibres[pos].m_lam_jump = ard_value;
		}else{
			fibres[pos].m_lam_jump = !no_ard_value;
		}

		//compute_f_prior();
      	if (m_f[idSubVOX]<=0 | m_f[idSubVOX]>=1 ){
      		}else{
	  		if(fibres[pos].m_lam_jump){              
	    			m_f_prior=log(double(m_f[idSubVOX]));
	  		}else{
	    			m_f_prior=0;
			}
			m_f_prior= fudgevalue* m_f_prior;
      		}
		fibres[pos].m_f_prior = m_f_prior;

		//fibres[vox].m_lam = m_lam; ??
		//fibres[vox].m_lam_prop = 1;
		//fibres[vox].m_lam_prior = 0;
		//compute_lam_prior();

		//compute_prior();
		fibres[pos].m_prior_en= m_th_prior + m_ph_prior + m_f_prior;
	}

	__syncthreads();

	//compute_signal_pre
	for(int f=0;f<nfib;f++){	
		for(int i=0; i<mydirs; i++){
			double myalpha = alpha[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC];
			double cos_alpha_minus_theta=cos(double(myalpha-m_th[f]));
			double cos_alpha_plus_theta=cos(double(myalpha+m_th[f]));
			int pos = idVOX*ndirections*nfib + f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
			double aux = (cos(double(m_ph[f]-beta[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC]))*(cos_alpha_minus_theta-cos_alpha_plus_theta)/2)+(cos_alpha_minus_theta+cos_alpha_plus_theta)/2;
		  aux =  aux*aux;
		 	angtmp[pos]= aux;
		}
	}
	//------ Fibre constructor ------
	//compute_signal()
	double old;
	for(int f=0;f<nfib;f++){
		for(int i=0; i<mydirs; i++){
			int pos = idVOX*ndirections*nfib + f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
			compute_signal(&signals[pos],&old,bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,m_R,angtmp[pos],model);
		}
	}

	//------ initialise_energies ------
	if(leader){
		getfsum(fsum,m_f,*m_f0,nfib);
		
	      	//compute_d_prior(); 
		if(*m_d>=0 && *m_d<=UPPERDIFF){
			if (model==3){
 	          		//float alpha=3.0; float beta=4000;  //Gamma_prior around 0.5-1E-3
 	          		multifibres[idVOX].m_d_prior =(1.0f-3.0f)*log(*m_d)+4000.0f**m_d;
 	        	}
       		}

	      	if(model>=2){
			//compute_d_std_prior();
			float upper_d_std=0.01;
			if (model==3) upper_d_std=0.004;
			if(*m_dstd>0 && *m_dstd<=upper_d_std){
				multifibres[idVOX].m_dstd_prior=log(*m_dstd);
			}
			if (model==3){
	  			//compute_R_prior();
				float upper_R=2.0f*R_priormean;
				float lower_R=R_priormean-2.0f*R_priorstd;
				if (R_priormean>0.5f)
					upper_R=1.0f;
				if (lower_R<0.0f)
					lower_R=1E-8f;
				if (R_priorfudge>0.0f && *m_d>UPPERDIFF/2.0f){
					//then use an ARD prior to avoid competition with the isotropic compartments
 	        			if (*m_R>=1E-8f && *m_R<=upper_R){
 	          				multifibres[idVOX].m_R_prior=R_priorfudge*log(*m_R);
 	        			}
 	      			}else{	
      					if(*m_R>lower_R && *m_R<=upper_R){
						float Rstd2=R_priorstd*R_priorstd; 
						multifibres[idVOX].m_R_prior=(*m_R-R_priormean)*(*m_R-R_priormean)/Rstd2;  //Gaussian prior
      					}
				}
			}
		}
	      	//compute_tau_prior(); m_tau_prior=0; so it doesn't do nothing, it is already 0
	      	if (m_includef0){
			//compute_f0_prior();
			if (*m_f0<=0 || *m_f0>=1){
	      		}else{
				if(!m_ardf0){}     	//Without ARD
				else              	//With ARD
		  			multifibres[idVOX].m_f0_prior= log(double(*m_f0));
	      		}
		}
	      	//compute_S0_prior(); m_S0_prior=0; so i don't do nothing, it is already 0
		//*m_prior_en = 0;
	      	//compute_prior();
	      	*m_prior_en=multifibres[idVOX].m_d_prior+multifibres[idVOX].m_S0_prior;
	      	if(model>=2)
			*m_prior_en= *m_prior_en+multifibres[idVOX].m_dstd_prior;
		if(model==3)
			*m_prior_en= *m_prior_en+multifibres[idVOX].m_R_prior;
	      	//if(m_rician) m_prior_en=m_prior_en+m_tau_prior; is 0
	      	if (m_includef0)
			*m_prior_en=*m_prior_en+multifibres[idVOX].m_f0_prior;
	      	for(int fib=0;fib<nfib;fib++){
			*m_prior_en=*m_prior_en+ fibres[idVOX*nfib+fib].m_prior_en;
	      	} 
		multifibres[idVOX].m_prior_en = *m_prior_en;
	}
	//------ initialise_energies ------
	//compute_iso_signal()
	for(int i=0; i<mydirs; i++){
		int pos = idVOX*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
		compute_iso_signal(&isosignals[pos],&old,bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,model);
	}		
 
	__syncthreads();

	//------ initialise_energies ------
	//compute_likelihood()
	compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[idVOX*nfib*ndirections],&isosignals[idVOX*ndirections],&datam[idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);

	__syncthreads();

	if(leader){
		multifibres[idVOX].m_likelihood_en = *m_likelihood_en;
	      	//------ initialise_energies ------
		//compute_energy();	
		multifibres[idVOX].m_energy = (*m_prior_en)+(*m_likelihood_en);

	    	//initialise_props();
	      	multifibres[idVOX].m_S0_prop=multifibres[idVOX].m_S0/10.0; 
	      	multifibres[idVOX].m_d_prop=*m_d/10.0;
	      	multifibres[idVOX].m_dstd_prop=*m_dstd/10.0;
	      	multifibres[idVOX].m_tau_prop=*m_tau/2.0;
	      	multifibres[idVOX].m_f0_prop=0.2;
		multifibres[idVOX].m_R_prop=*m_R/10.0;
	}
}

extern "C" __global__ void runmcmc_kernel(	//INPUT 
						const float*			datam,
						const float*			bvals,
						const double*			alpha,
						const double*			beta,
						hiprandState*			randstate,
						const float			R_priormean,
						const float			R_priorstd,	
						const float			R_priorfudge,			
						const int			ndirections,
						const int			nfib,
						const int			nparams,
						const int 			model,
						const float 			fudgevalue,
						const bool 			m_include_f0,
						const bool 			m_ardf0,
						const bool 			can_use_ard, 
						const bool 			rician,
						const bool			gradnonlin,
						const int 			updateproposalevery, 	//update every this number of iterations	
						const int 			iterations,
						const int 			iters_burnin,		//iters in burin, we need it to continue the updates at the correct time. 
						const int 			record_every, 		//record every this number
						const int 			totalrecords,		//total number of records to do
						//TO USE
						double*				oldsignals,
						double*				oldisosignals,
						double*				angtmp,
						double*				oldangtmp,
						//INPUT-OUTPUT
						FibreGPU*			fibres,
						MultifibreGPU*			multifibres,
						double*				signals,
						double*				isosignals,
						//OUTPUT
						float*				rf0,			//record of parameters
						float*				rtau,
						float*				rs0,
						float*				rd,
						float*				rdstd,
						float*				rR,
						float*				rth,
						float*				rph, 
						float*				rf)
{	
	int idSubVOX= threadIdx.x%THREADS_VOXEL_MCMC;
	int idVOX_block =  threadIdx.x/THREADS_VOXEL_MCMC;
	bool leader = (idSubVOX==0);

	// there may be several voxels per block: VOXELS_BLOCK_MCMC 
	////////// DYNAMIC SHARED MEMORY ///////////				// each voxel:
	extern __shared__ double shared[];
	hiprandState* localrandState = (hiprandState*)shared;			//1 hiprandState 
	double* reduction = (double*)&localrandState[VOXELS_BLOCK_MCMC];	// THREADS_VOXEL_MCMC 

	float* m_S0 = (float*) &reduction[VOXELS_BLOCK_MCMC*THREADS_VOXEL_MCMC];//1
	float* m_d = (float*) &m_S0[VOXELS_BLOCK_MCMC];				//1
	float* m_dstd =(float*) &m_d[VOXELS_BLOCK_MCMC];			//1
	float* m_R =(float*) &m_dstd[VOXELS_BLOCK_MCMC];			//1	
	float* m_f0 = (float*) &m_R[VOXELS_BLOCK_MCMC];				//1
	float* m_tau = (float*) &m_f0[VOXELS_BLOCK_MCMC];			//1
	float* m_th = (float*) &m_tau[VOXELS_BLOCK_MCMC];			//nfib
	float* m_ph = (float*) &m_th[VOXELS_BLOCK_MCMC*nfib];			//nfib
	float* m_f = (float*) &m_ph[VOXELS_BLOCK_MCMC*nfib];			//nfib

	float* m_S0_prior = (float*) &m_f[VOXELS_BLOCK_MCMC*nfib];		//1
	float* m_d_prior = (float*) &m_S0_prior[VOXELS_BLOCK_MCMC];		//1
	float* m_dstd_prior = (float*) &m_d_prior[VOXELS_BLOCK_MCMC];		//1
	float* m_R_prior = (float*) &m_dstd_prior[VOXELS_BLOCK_MCMC];		//1	
	float* m_f0_prior = (float*) &m_R_prior[VOXELS_BLOCK_MCMC];		//1
	float* m_tau_prior = (float*) &m_f0_prior[VOXELS_BLOCK_MCMC];		//1
	float* m_th_prior = (float*) &m_tau_prior[VOXELS_BLOCK_MCMC];		//nfib
	float* m_ph_prior = (float*) &m_th_prior[VOXELS_BLOCK_MCMC*nfib];	//nfib
	float* m_f_prior = (float*) &m_ph_prior[VOXELS_BLOCK_MCMC*nfib];	//nfib

	float* m_S0_prop = (float*) &m_f_prior[VOXELS_BLOCK_MCMC*nfib];		//1
	float* m_d_prop = (float*) &m_S0_prop[VOXELS_BLOCK_MCMC];		//1
	float* m_dstd_prop = (float*) &m_d_prop[VOXELS_BLOCK_MCMC];		//1
	float* m_R_prop = (float*) &m_dstd_prop[VOXELS_BLOCK_MCMC];		//1
	float* m_f0_prop = (float*) &m_R_prop[VOXELS_BLOCK_MCMC];		//1
	float* m_tau_prop = (float*) &m_f0_prop[VOXELS_BLOCK_MCMC];		//1
	float* m_th_prop = (float*) &m_tau_prop[VOXELS_BLOCK_MCMC];		//nfib
	float* m_ph_prop = (float*) &m_th_prop[VOXELS_BLOCK_MCMC*nfib];		//nfib
	float* m_f_prop = (float*) &m_ph_prop[VOXELS_BLOCK_MCMC*nfib];		//nfib

	float* fsum = (float*) &m_f_prop[VOXELS_BLOCK_MCMC*nfib];		//1
	float* m_likelihood_en = (float*) &fsum[VOXELS_BLOCK_MCMC];		//1
	float* m_prior_en = (float*) &m_likelihood_en[VOXELS_BLOCK_MCMC];	//1
	float* m_old_prior_en = (float*) &m_prior_en[VOXELS_BLOCK_MCMC];	//1
	float* fm_prior_en = (float*) &m_old_prior_en[VOXELS_BLOCK_MCMC];	//nfib
	float* fm_old_prior_en = (float*) &fm_prior_en[VOXELS_BLOCK_MCMC*nfib];	//1
	float* m_energy = (float*) &fm_old_prior_en[VOXELS_BLOCK_MCMC];		//1
	float* m_old_energy = (float*) &m_energy[VOXELS_BLOCK_MCMC];		//1
	float* old = (float*) &m_old_energy[VOXELS_BLOCK_MCMC];			//2

	int* m_S0_acc = (int*) &old[VOXELS_BLOCK_MCMC*2];			//1
	int* m_d_acc = (int*) &m_S0_acc[VOXELS_BLOCK_MCMC];			//1
	int* m_dstd_acc = (int*) &m_d_acc[VOXELS_BLOCK_MCMC];			//1
	int* m_R_acc = (int*) &m_dstd_acc[VOXELS_BLOCK_MCMC];			//1
	int* m_f0_acc = (int*) &m_R_acc[VOXELS_BLOCK_MCMC];			//1
	int* m_tau_acc = (int*) &m_f0_acc[VOXELS_BLOCK_MCMC];			//1
	int* m_th_acc = (int*) &m_tau_acc[VOXELS_BLOCK_MCMC];			//nfib
	int* m_ph_acc = (int*) &m_th_acc[VOXELS_BLOCK_MCMC*nfib];		//nfib
	int* m_f_acc = (int*) &m_ph_acc[VOXELS_BLOCK_MCMC*nfib];		//nfib

	int* m_S0_rej = (int*) &m_f_acc[VOXELS_BLOCK_MCMC*nfib];		//1
	int* m_d_rej = (int*) &m_S0_rej[VOXELS_BLOCK_MCMC];			//1
	int* m_dstd_rej = (int*) &m_d_rej[VOXELS_BLOCK_MCMC];			//1
	int* m_R_rej = (int*) &m_dstd_rej[VOXELS_BLOCK_MCMC];			//1
	int* m_f0_rej = (int*) &m_R_rej[VOXELS_BLOCK_MCMC];			//1
	int* m_tau_rej = (int*) &m_f0_rej[VOXELS_BLOCK_MCMC];			//1
	int* m_th_rej = (int*) &m_tau_rej[VOXELS_BLOCK_MCMC];			//nfib
	int* m_ph_rej = (int*) &m_th_rej[VOXELS_BLOCK_MCMC*nfib];		//nfib
	int* m_f_rej = (int*) &m_ph_rej[VOXELS_BLOCK_MCMC*nfib];		//nfib
	
	int* rejflag = (int*) &m_f_rej[VOXELS_BLOCK_MCMC*nfib];			//3
	int* m_lam_jump = (int*) &rejflag[VOXELS_BLOCK_MCMC*3];			//nfib
	int* idVOX = (int*) &m_lam_jump[VOXELS_BLOCK_MCMC*nfib];		//1
	int* count_update = (int*) &idVOX[VOXELS_BLOCK_MCMC];			//1	
	int* recordcount = (int*) &count_update[VOXELS_BLOCK_MCMC];		//1
	int* sample = (int*) &recordcount[VOXELS_BLOCK_MCMC];			//1
	int* posBV = (int*) &sample[VOXELS_BLOCK_MCMC];				//1
	////////// DYNAMIC SHARED MEMORY ///////////
	
	///// UPDATE shared memory pointers depending on the Id of a voxel within a block /////
										//each voxel:
	localrandState = (hiprandState*)&localrandState[idVOX_block];			// 1 hiprandState:
	reduction = &reduction[idVOX_block*THREADS_VOXEL_MCMC];			// THREADS_VOXEL_MCMC 

	m_S0 = &m_S0[idVOX_block];						//1
	m_d = &m_d[idVOX_block];						//1
	m_dstd = &m_dstd[idVOX_block];						//1
	m_R =&m_R[idVOX_block];							//1
	m_f0 = &m_f0[idVOX_block];						//1
	m_tau = &m_tau[idVOX_block];						//1
	m_th = &m_th[idVOX_block*nfib];						//nfib
	m_ph = &m_ph[idVOX_block*nfib];						//nfib
	m_f = &m_f[idVOX_block*nfib];						//nfib

	m_S0_prior = &m_S0_prior[idVOX_block];					//1
	m_d_prior = &m_d_prior[idVOX_block];					//1
	m_dstd_prior = &m_dstd_prior[idVOX_block];				//1
	m_R_prior = &m_R_prior[idVOX_block];					//1
	m_f0_prior = &m_f0_prior[idVOX_block];					//1
	m_tau_prior = &m_tau_prior[idVOX_block];				//1
	m_th_prior = &m_th_prior[idVOX_block*nfib];				//nfib
	m_ph_prior = &m_ph_prior[idVOX_block*nfib];				//nfib
	m_f_prior = &m_f_prior[idVOX_block*nfib];				//nfib

	m_S0_prop = &m_S0_prop[idVOX_block];					//1
	m_d_prop = &m_d_prop[idVOX_block];					//1
	m_dstd_prop = &m_dstd_prop[idVOX_block];				//1
	m_R_prop = &m_R_prop[idVOX_block];					//1
	m_f0_prop = &m_f0_prop[idVOX_block];					//1
	m_tau_prop = &m_tau_prop[idVOX_block];					//1
	m_th_prop = &m_th_prop[idVOX_block*nfib];				//nfib
	m_ph_prop = &m_ph_prop[idVOX_block*nfib];				//nfib
	m_f_prop = &m_f_prop[idVOX_block*nfib];					//nfib

	fsum = &fsum[idVOX_block];						//1
	m_likelihood_en = &m_likelihood_en[idVOX_block];			//1
	m_prior_en = &m_prior_en[idVOX_block];					//1
	m_old_prior_en = &m_old_prior_en[idVOX_block];				//1
	fm_prior_en = &fm_prior_en[idVOX_block*nfib];				//nfib
	fm_old_prior_en = &fm_old_prior_en[idVOX_block];			//1
	m_energy = &m_energy[idVOX_block];					//1
	m_old_energy = &m_old_energy[idVOX_block];				//1
	old = &old[idVOX_block*2];						//2

	m_S0_acc = &m_S0_acc[idVOX_block];					//1
	m_d_acc = &m_d_acc[idVOX_block];					//1
	m_dstd_acc = &m_dstd_acc[idVOX_block];					//1
	m_R_acc = &m_R_acc[idVOX_block];					//1
	m_f0_acc = &m_f0_acc[idVOX_block];					//1
	m_tau_acc = &m_tau_acc[idVOX_block];					//1
	m_th_acc = &m_th_acc[idVOX_block*nfib];					//nfib
	m_ph_acc = &m_ph_acc[idVOX_block*nfib];					//nfib
	m_f_acc = &m_f_acc[idVOX_block*nfib];					//nfib

	m_S0_rej = &m_S0_rej[idVOX_block];					//1
	m_d_rej = &m_d_rej[idVOX_block];					//1
	m_dstd_rej = &m_dstd_rej[idVOX_block];					//1
	m_R_rej = &m_R_rej[idVOX_block];					//1
	m_f0_rej = &m_f0_rej[idVOX_block];					//1
	m_tau_rej = &m_tau_rej[idVOX_block];					//1
	m_th_rej = &m_th_rej[idVOX_block*nfib];					//nfib
	m_ph_rej = &m_ph_rej[idVOX_block*nfib];					//nfib
	m_f_rej = &m_f_rej[idVOX_block*nfib];					//nfib
	
	rejflag = &rejflag[idVOX_block*3];					//3
	m_lam_jump =  &m_lam_jump[idVOX_block*nfib];				//nfib
	idVOX = &idVOX[idVOX_block];						//1
	count_update = &count_update[idVOX_block];				//1
	recordcount = &recordcount[idVOX_block];				//1
	sample = &sample[idVOX_block];						//1
	posBV = &posBV[idVOX_block];						//1
	/////////////////////////////////////////////////////////////////////////////////////////////

	if (leader){
		*idVOX= (blockIdx.x*VOXELS_BLOCK_MCMC)+int(threadIdx.x/THREADS_VOXEL_MCMC);
		*count_update = iters_burnin;	//count for updates
		*recordcount = 0;	
		*sample=1;		//the next number of sample.....the index start in 0

		if(gradnonlin)*posBV = (*idVOX*ndirections);
		else *posBV = 0;

		*m_prior_en=multifibres[*idVOX].m_prior_en;

		if(model>=2){
			*m_dstd_acc=multifibres[*idVOX].m_dstd_acc;
			*m_dstd_rej=multifibres[*idVOX].m_dstd_rej;
			*m_dstd_prior=multifibres[*idVOX].m_dstd_prior;
			*m_dstd_prop=multifibres[*idVOX].m_dstd_prop;
			*m_dstd=multifibres[*idVOX].m_dstd;
		}else{
			*m_dstd_acc=0;
			*m_dstd_rej=0;
			*m_dstd_prior=0;
			*m_dstd_prop=0;
			*m_dstd=0;
		}
		if(model==3){
			*m_R_acc=multifibres[*idVOX].m_R_acc;
			*m_R_rej=multifibres[*idVOX].m_R_rej;
			*m_R_prior=multifibres[*idVOX].m_R_prior;
			*m_R_prop=multifibres[*idVOX].m_R_prop;
			*m_R=multifibres[*idVOX].m_R;
		}else{
			*m_R_acc=0;
			*m_R_rej=0;
			*m_R_prior=0;
			*m_R_prop=0;
			*m_R=0;
		}
	
		*m_d=multifibres[*idVOX].m_d;
		*m_energy=multifibres[*idVOX].m_energy;
		*m_d_prop=multifibres[*idVOX].m_d_prop;
		*m_d_prior=multifibres[*idVOX].m_d_prior;
		*m_S0_prior=multifibres[*idVOX].m_S0_prior;
		*m_S0=multifibres[*idVOX].m_S0;
		*m_likelihood_en=multifibres[*idVOX].m_likelihood_en;
		*m_d_acc=multifibres[*idVOX].m_d_acc;
		*m_d_rej=multifibres[*idVOX].m_d_rej;
		*m_S0_acc=multifibres[*idVOX].m_S0_acc;
		*m_S0_rej=multifibres[*idVOX].m_S0_rej;
		*m_S0_prop=multifibres[*idVOX].m_S0_prop;

		if(m_include_f0){
			*m_f0_acc=multifibres[*idVOX].m_f0_acc;
			*m_f0_rej=multifibres[*idVOX].m_f0_rej;
			*m_f0_prop=multifibres[*idVOX].m_f0_prop;
			*m_f0_prior=multifibres[*idVOX].m_f0_prior;
			*m_f0=multifibres[*idVOX].m_f0;
		}else{ 
			*m_f0_acc=0;
			*m_f0_rej=0;
			*m_f0_prop=0;
			*m_f0_prior=0;
			*m_f0=0;
		}
				
		if(rician){
			*m_tau_acc=multifibres[*idVOX].m_tau_acc;
			*m_tau_rej=multifibres[*idVOX].m_tau_rej;
			*m_tau_prop=multifibres[*idVOX].m_tau_prop;
			*m_tau_prior=multifibres[*idVOX].m_tau_prior;
			*m_tau=multifibres[*idVOX].m_tau;	
		}else{ 
			*m_tau_acc=0;
			*m_tau_rej=0;
			*m_tau_prop=0;
			*m_tau_prior=0;
			*m_tau=0;
		}

		*localrandState = randstate[*idVOX];
	}

	__syncthreads();

	int mydirs = ndirections/THREADS_VOXEL_MCMC;
	int mod = ndirections%THREADS_VOXEL_MCMC;
	if(mod&&(idSubVOX<mod)) mydirs++;

	if(idSubVOX<nfib){
		int pos = (*idVOX*nfib)+idSubVOX;
		m_th[idSubVOX]=fibres[pos].m_th;
		m_ph[idSubVOX]=fibres[pos].m_ph;
		m_f[idSubVOX]=fibres[pos].m_f;
	
		m_th_acc[idSubVOX]=fibres[pos].m_th_acc;
		m_th_rej[idSubVOX]=fibres[pos].m_th_rej;
		m_ph_acc[idSubVOX]=fibres[pos].m_ph_acc;
		m_ph_rej[idSubVOX]=fibres[pos].m_ph_rej;
		m_f_acc[idSubVOX]=fibres[pos].m_f_acc;
		m_f_rej[idSubVOX]=fibres[pos].m_f_rej;

		fm_prior_en[idSubVOX]=fibres[pos].m_prior_en;
		m_th_prior[idSubVOX]=fibres[pos].m_th_prior;
		m_ph_prior[idSubVOX]=fibres[pos].m_ph_prior;
		m_f_prior[idSubVOX]=fibres[pos].m_f_prior;

		m_th_prop[idSubVOX]=fibres[pos].m_th_prop;
		m_ph_prop[idSubVOX]=fibres[pos].m_ph_prop;
		m_f_prop[idSubVOX]=fibres[pos].m_f_prop;

		m_lam_jump[idSubVOX]=fibres[pos].m_lam_jump;		
	}
	__syncthreads();
		
	//compute_signal_pre
	for(int f=0;f<nfib;f++){
		for(int i=0; i<mydirs; i++){	
			double myalpha = alpha[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC];
			double cos_alpha_minus_theta=cos(double(myalpha-m_th[f]));
			double cos_alpha_plus_theta=cos(double(myalpha+m_th[f]));
			int pos = *idVOX*ndirections*nfib + f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
			double aux = (cos(double(m_ph[f]-beta[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC]))*(cos_alpha_minus_theta-cos_alpha_plus_theta)/2)+(cos_alpha_minus_theta+cos_alpha_plus_theta)/2;
		     	aux =  aux*aux;
		 	angtmp[pos]= aux;
		}
	}

	if (leader) getfsum(fsum,m_f,*m_f0,nfib);

	// START ITERATIONS: code jump()
	for (int niter=0; niter<iterations; niter++){
		if (leader){
			*count_update=*count_update+1;
			*recordcount=*recordcount+1;
		}
			
////////////////////////////////////////////////////////////////// F0
		if(m_include_f0){
			if (leader){
				propose(m_f0,old,*m_f0_prop,localrandState);
				//compute_f0_prior()     
				old[1]=*m_f0_prior;
	      if(*m_f0<=0 || *m_f0 >=1){ 
					rejflag[0]=true;
				}else{ 	
					rejflag[0]=false;
					if(!m_ardf0){
						*m_f0_prior=0;
	      				}else{
						*m_f0_prior=log(double(*m_f0));
					}
				}
				getfsum(fsum,m_f,*m_f0,nfib);
				//compute_prior()
				compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);
				//reject_f_sum()
				rejflag[1]=(*fsum>1);
			}
			__syncthreads();
			//compute_likelihood()
			compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);
			__syncthreads();

			if (leader){
				rejflag[2]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);
				if(!rejflag[0]){
					if(!rejflag[1]){
						if(rejflag[2]){
		  					*m_f0_acc=*m_f0_acc+1;   
						}else{
							reject(m_f0,m_f0_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_f0_rej);
							getfsum(fsum,m_f,*m_f0,nfib);
						}
					}else{
						reject(m_f0,m_f0_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_f0_rej);
						getfsum(fsum,m_f,*m_f0,nfib);
					}
				}else{ 
					reject(m_f0,m_f0_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_f0_rej);
					getfsum(fsum,m_f,*m_f0,nfib);
				}
			}
			 __syncthreads(); // old
		}
////////////////////////////////////////////////////////////////// TAU
		if(rician){
			if (leader){
				propose(m_tau,old,*m_tau_prop,localrandState);
				//compute_tau_prior()     
				old[1]=*m_tau_prior;
	      			if(*m_tau<=0.0f){ 
					rejflag[0]=true;
				}else{ 	
					rejflag[0]=false;
					*m_tau_prior=0.0f;
				}
				//compute_prior()
				compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);
			}
			__syncthreads();
			//compute_likelihood()
			compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);
			__syncthreads();

			if (leader){
				rejflag[1]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);
				if(!rejflag[0]){
					if(rejflag[1]){
		  				*m_tau_acc=*m_tau_acc+1;   
					}else{ 
						reject(m_tau,m_tau_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_tau_rej);
					}
				}else{ 
					reject(m_tau,m_tau_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_tau_rej);
				}
			}
			 __syncthreads(); // old
		}
////////////////////////////////////////////////////////////////// D
		if (leader){
			propose(m_d,old,*m_d_prop,localrandState);
			//compute_d_prior()      
			old[1]=*m_d_prior;
			if(*m_d<0 || *m_d>UPPERDIFF){
				rejflag[0]=true;
			}else{
				if (model==3){
					//float alpha=3.0; float beta=4000;  //Gamma_prior around 0.5-1E-3
					*m_d_prior=(1.0f-3.0f)*log(*m_d)+4000.0f**m_d;
				}else{
					*m_d_prior=0;
				}
				rejflag[0]=false;
			}
		}
		__syncthreads();	
		//compute_signal()
		for(int f=0;f<nfib;f++){
			for(int i=0; i<mydirs; i++){
				int pos = *idVOX*ndirections*nfib + f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
				compute_signal(&signals[pos],&oldsignals[pos],bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,m_R,angtmp[pos],model);
			}
		}
		//compute_iso_signal()
		for(int i=0; i<mydirs; i++){
			int pos = *idVOX*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
			compute_iso_signal(&isosignals[pos],&oldisosignals[pos],bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,model);
		}				

		if (leader){
			//compute_prior()
			compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);
		}
		__syncthreads();	
		//compute_likelihood()
		compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);		
		__syncthreads();
				
		if (leader){
			rejflag[1]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);
		}	
		__syncthreads();

       		if(!rejflag[0]){
			if(rejflag[1]){
	  			if (leader) *m_d_acc=*m_d_acc+1;   
			}else{
				if (leader){
					reject(m_d,m_d_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_d_rej);
				}
				restore_signals(signals,oldsignals,*idVOX,idSubVOX,mydirs,nfib,ndirections);
				restore_isosignals(isosignals,oldisosignals,*idVOX,idSubVOX,mydirs,ndirections);
			}
        	}else{ 
			if (leader){
				reject(m_d,m_d_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_d_rej);
			}
      			restore_signals(signals,oldsignals,*idVOX,idSubVOX,mydirs,nfib,ndirections);
			restore_isosignals(isosignals,oldisosignals,*idVOX,idSubVOX,mydirs,ndirections);
        	}
		 __syncthreads(); // old
////////////////////////////////////////////////////////////////// D_STD
		if(model>=2){
			if (leader){	
				propose(m_dstd,old,*m_dstd_prop,localrandState);
				//compute_d_std_prior()     
				old[1]=*m_dstd_prior;
				float upper_d_std=0.01;
				if (model==3) upper_d_std=0.004;
				if(*m_dstd<=0 || *m_dstd>upper_d_std){
					rejflag[0]=true;
				}else{
					*m_dstd_prior=log(*m_dstd);
					rejflag[0]=false;	
				}
			}
			__syncthreads();
			//compute_signal()
			if(model==2){
				for(int f=0;f<nfib;f++){
					for(int i=0; i<mydirs; i++){				
						int pos = *idVOX*ndirections*nfib + f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
						compute_signal(&signals[pos],&oldsignals[pos],bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,m_R,angtmp[pos],model);
					}
				}
			}
			//compute_iso_signal()
			for(int i=0; i<mydirs; i++){
				int pos = *idVOX*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
				compute_iso_signal(&isosignals[pos],&oldisosignals[pos],bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,model);
			}
			if (leader){
				//compute_prior()
				compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);
			}
			__syncthreads();
			//compute_likelihood()
			compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);
			__syncthreads();

			if (leader){
				rejflag[1]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);				
			}
			__syncthreads();
				
			if(!rejflag[0]){
				if(rejflag[1]){
		  			if (leader) *m_dstd_acc=*m_dstd_acc+1;   
				}else{ 
					if (leader){
						reject(m_dstd,m_dstd_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_dstd_rej);
					}
					if(model==2){
						restore_signals(signals,oldsignals,*idVOX,idSubVOX,mydirs,nfib,ndirections);
					}
					restore_isosignals(isosignals,oldisosignals,*idVOX,idSubVOX,mydirs,ndirections);
				}
			}else{ 
				if (leader){
					reject(m_dstd,m_dstd_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_dstd_rej);
				}
				if(model==2){
					restore_signals(signals,oldsignals,*idVOX,idSubVOX,mydirs,nfib,ndirections);
				}
				restore_isosignals(isosignals,oldisosignals,*idVOX,idSubVOX,mydirs,ndirections);
			}
			 __syncthreads(); // old
////////////////////////////////////////////////////////////////// R
			if(model==3){
				if (leader){	
					propose(m_R,old,*m_R_prop,localrandState);
					//compute_R_prior()     
					old[1]=*m_R_prior;
					float upper_R=2.0f*R_priormean;
					float lower_R=R_priormean-2.0f*R_priorstd;
      					if (R_priormean>0.5f){
						upper_R=1.0f;
					}
					if (lower_R<0.0f)
						lower_R=1e-8f;
					if (R_priorfudge>0.0f && *m_d>UPPERDIFF/2.0f){
					//then use an ARD prior to avoid competition with the isotropic compartments
 	        				if (*m_R<1E-8f || *m_R>upper_R)
 	          					rejflag[0]=true;
 	        				else{
 	          					*m_R_prior=R_priorfudge*log(*m_R);
 	          					rejflag[0]=false;
 	        				}
 	      				}else{
	      					if(*m_R<=lower_R || *m_R>upper_R){  
							//Truncate prior to avoid too spherical (high m_R) or too anisitropic (small m_R) profiles 
							rejflag[0]=true;
						}else{
							float Rstd2=R_priorstd*R_priorstd; 
							*m_R_prior=(*m_R-R_priormean)*(*m_R-R_priormean)/Rstd2;  //Gaussian prior
							rejflag[0]=false;
	      					}
					}
				}
				__syncthreads();
				//compute_signal()
				for(int f=0;f<nfib;f++){
					for(int i=0; i<mydirs; i++){				
						int pos = *idVOX*ndirections*nfib + f*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
						compute_signal(&signals[pos],&oldsignals[pos],bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,m_R,angtmp[pos],model);
					}
				}
				if (leader){
					//compute_prior()
					compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);
				}
				__syncthreads();
				//compute_likelihood()
				compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);
				__syncthreads();

				if (leader){
					rejflag[1]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);				
				}
				__syncthreads();
				
				if(!rejflag[0]){
					if(rejflag[1]){
		  				if (leader) *m_R_acc=*m_R_acc+1;   
					}else{ 
						if (leader){
							reject(m_R,m_R_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_R_rej);
						}
						restore_signals(signals,oldsignals,*idVOX,idSubVOX,mydirs,nfib,ndirections);
					}
				}else{ 
					if (leader){
						reject(m_R,m_R_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_R_rej);
					}
					restore_signals(signals,oldsignals,*idVOX,idSubVOX,mydirs,nfib,ndirections);
				}
				 __syncthreads(); // old
			}
		}
////////////////////////////////////////////////////////////////// S0
		if (leader){
			propose(m_S0,old,*m_S0_prop,localrandState);
			//compute_S0_prior()
			old[1]=*m_S0_prior;
        		if(*m_S0<0) rejflag[0]=true;
        		else{    
				*m_S0_prior=0;
	  			rejflag[0]=false;
        		}
			//compute_prior()
			compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);
		}
		__syncthreads();
		//compute_likelihood()
		compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);
		__syncthreads();

		if (leader){
			rejflag[1]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);

        		if(!rejflag[0]){
				if(rejflag[1]){
	  				*m_S0_acc=*m_S0_acc+1;   
				}else{
					reject(m_S0,m_S0_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_S0_rej);
				}
        		}else{ 
				reject(m_S0,m_S0_prior,old,m_prior_en,m_old_prior_en,m_energy,m_old_energy,m_S0_rej);
			}
        	}
		 __syncthreads(); // old
////////////////////////////////////////////////////////////////////////////     TH
     		for(int fibre=0;fibre<nfib;fibre++){  
			if (leader){ 
				propose(&m_th[fibre],old,m_th_prop[fibre],localrandState);
				//compute_th_prior()
				old[1]=m_th_prior[fibre];
      	   			if(m_th[fibre]==0){
					m_th_prior[fibre]=0;
		   		}else{
					m_th_prior[fibre]=-log(double(fabs(sin(double(m_th[fibre]))/2)));
	      	   		}
		  		//rejflag[0]=false; /////////////////always false
				//compute_prior()
				*fm_old_prior_en=fm_prior_en[fibre];
	      	   		fm_prior_en[fibre]=m_th_prior[fibre]+m_ph_prior[fibre]+m_f_prior[fibre];	
			}
			__syncthreads();
			//compute_signal()
			//compute_signal_pre	
			for(int i=0; i<mydirs; i++){
				double myalpha = alpha[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC];
				double cos_alpha_minus_theta=cos(double(myalpha-m_th[fibre]));
				double cos_alpha_plus_theta=cos(double(myalpha+m_th[fibre]));
				int pos = *idVOX*ndirections*nfib + fibre*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
				int pos2 = *idVOX*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
				oldangtmp[pos2]=angtmp[pos];
				double aux = (cos(double(m_ph[fibre]-beta[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC]))*(cos_alpha_minus_theta-cos_alpha_plus_theta)/2)+(cos_alpha_minus_theta+cos_alpha_plus_theta)/2;
		     		aux =  aux*aux;
		 		angtmp[pos]= aux;

				compute_signal(&signals[pos],&oldsignals[pos],bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,m_R,angtmp[pos],model);
			}
			if (leader){
				//compute_prior()
				compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);	
			}
			__syncthreads();
			//compute_likelihood()
			compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);
			__syncthreads();

			if (leader){ 
				rejflag[1]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);
			}
			__syncthreads();
			
			if(rejflag[1]){
		  		if (leader) m_th_acc[fibre]++;   
			}else{
				if (leader){
					rejectF(&m_th[fibre],&m_th_prior[fibre],old,m_prior_en,m_old_prior_en,&fm_prior_en[fibre],fm_old_prior_en,m_energy,m_old_energy,&m_th_rej[fibre]);
				}
				//compute_signal_pre undo
				restore_angtmp_signals(signals,oldsignals,angtmp,oldangtmp,*idVOX,idSubVOX,mydirs,nfib,fibre,ndirections);
			}
			__syncthreads();
///////////////////////////////////////     PH
			if (leader){
				propose(&m_ph[fibre],old,m_ph_prop[fibre],localrandState);
				//compute_ph_prior()
				old[1]=m_ph_prior[fibre];
      				m_ph_prior[fibre]=0;
      				//rejflag[0]=false;
				//compute_prior()
				*fm_old_prior_en=fm_prior_en[fibre];
      	   			fm_prior_en[fibre]=m_th_prior[fibre]+m_ph_prior[fibre]+m_f_prior[fibre];
			}
			__syncthreads();
			//compute_signal()
			//compute_signal_pre
			for(int i=0; i<mydirs; i++){
				double myalpha = alpha[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC];
				double cos_alpha_minus_theta=cos(double(myalpha-m_th[fibre]));
			  double cos_alpha_plus_theta=cos(double(myalpha+m_th[fibre]));
				int pos = *idVOX*ndirections*nfib + fibre*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
				int pos2 = *idVOX*ndirections + idSubVOX + i*THREADS_VOXEL_MCMC;
				oldangtmp[pos2]=angtmp[pos];
				double aux = (cos(double(m_ph[fibre]-beta[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC]))*(cos_alpha_minus_theta-cos_alpha_plus_theta)/2)+(cos_alpha_minus_theta+cos_alpha_plus_theta)/2;
		     		aux =  aux*aux;
		 		angtmp[pos]= aux;
				
				compute_signal(&signals[pos],&oldsignals[pos],bvals[*posBV+idSubVOX+i*THREADS_VOXEL_MCMC],m_d,m_dstd,m_R,angtmp[pos],model);
			}

			if (leader){
				//compute_prior()
				compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);
			}
			__syncthreads();
			//compute_likelihood()
			compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);
			__syncthreads();

			if (leader){
				rejflag[1]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);
			}
			__syncthreads();

			//if(!rejflag[0]){
			if(rejflag[1]){
		  		if (leader) m_ph_acc[fibre]++;   
			}else{
				if (leader){
					rejectF(&m_ph[fibre],&m_ph_prior[fibre],old,m_prior_en,m_old_prior_en,&fm_prior_en[fibre],fm_old_prior_en,m_energy,m_old_energy,&m_ph_rej[fibre]);
				}
				//compute_signal_pre undo
				restore_angtmp_signals(signals,oldsignals,angtmp,oldangtmp,*idVOX,idSubVOX,mydirs,nfib,fibre,ndirections);
			}

			__syncthreads();
////////////////////////////////////////////             F
			if (leader){
				propose(&m_f[fibre],old,m_f_prop[fibre],localrandState);

	     			//compute_f_prior()
	        		old[1]=m_f_prior[fibre];
				if (m_f[fibre]<=0 || m_f[fibre]>=1) rejflag[0]=true;
	        		else{
		      			if(!can_use_ard ){
		  				m_f_prior[fibre]=0;
					}else{
		  				if(m_lam_jump[fibre]){
							m_f_prior[fibre]=log(double(m_f[fibre]));
						}else{
		    					m_f_prior[fibre]=0;
		  				}
					}
					m_f_prior[fibre]=fudgevalue*m_f_prior[fibre];
					rejflag[0]=false;
	      			}
				//compute_prior()
				*fm_old_prior_en=fm_prior_en[fibre];
      	   			fm_prior_en[fibre]=m_th_prior[fibre]+m_ph_prior[fibre]+m_f_prior[fibre];
						
				getfsum(fsum,m_f,*m_f0,nfib);
				//reject_f_sum()
				rejflag[1]=(*fsum>1);
				//compute_prior()
				compute_prior(m_prior_en,m_old_prior_en,m_d_prior,m_S0_prior,fm_prior_en,m_f0_prior,m_tau_prior,m_dstd_prior,m_R_prior,nfib);	
			}

			__syncthreads();
			//compute_likelihood()
			compute_likelihood(idSubVOX,m_S0,m_likelihood_en,m_f,&signals[*idVOX*nfib*ndirections],&isosignals[*idVOX*ndirections],&datam[*idVOX*ndirections],fsum,reduction,m_f0,rician,m_tau,mydirs,ndirections,nfib);	
			__syncthreads();

			if (leader){
				rejflag[2]=compute_test_energy(m_energy,m_old_energy,*m_prior_en,*m_likelihood_en,localrandState);

		      		if(!rejflag[0]){
					if(!rejflag[1]){
						if(rejflag[2]){
			  				m_f_acc[fibre]++;   
						}else{
							rejectF(&m_f[fibre],&m_f_prior[fibre],old,m_prior_en,m_old_prior_en,&fm_prior_en[fibre],fm_old_prior_en,m_energy,m_old_energy,&m_f_rej[fibre]);
							getfsum(fsum,m_f,*m_f0,nfib);
						}
					}else{ 
						rejectF(&m_f[fibre],&m_f_prior[fibre],old,m_prior_en,m_old_prior_en,&fm_prior_en[fibre],fm_old_prior_en,m_energy,m_old_energy,&m_f_rej[fibre]);
						getfsum(fsum,m_f,*m_f0,nfib);
					}
				}else{
					rejectF(&m_f[fibre],&m_f_prior[fibre],old,m_prior_en,m_old_prior_en,&fm_prior_en[fibre],fm_old_prior_en,m_energy,m_old_energy,&m_f_rej[fibre]);
					getfsum(fsum,m_f,*m_f0,nfib);
				}
			}
			__syncthreads();	

        	}//end while nfib

		if((record_every)&&((*recordcount%record_every)==0)&&leader){
			rd[(*idVOX*totalrecords)+*sample-1]= *m_d;
			if(m_include_f0) rf0[(*idVOX*totalrecords)+*sample-1]= *m_f0;
			if(rician) rtau[(*idVOX*totalrecords)+*sample-1]= *m_tau;
			if(model>=2) rdstd[(*idVOX*totalrecords)+*sample-1]= *m_dstd;
			if(model==3) rR[(*idVOX*totalrecords)+*sample-1]= *m_R;	
			rs0[(*idVOX*totalrecords)+*sample-1]= *m_S0;
			for(int j=0;j<nfib;j++){
				rth[(*idVOX*totalrecords*nfib)+(j*totalrecords)+*sample-1]=m_th[j];
				rph[(*idVOX*totalrecords*nfib)+(j*totalrecords)+*sample-1]=m_ph[j];
				rf[(*idVOX*totalrecords*nfib)+(j*totalrecords)+*sample-1]=m_f[j];
			}
			*sample=*sample+1;
        	}

        	if(((*count_update%updateproposalevery)==0)&&leader){
			//m_multifibre.update_proposals();
			*m_d_prop*=sqrt(float(*m_d_acc+1)/float(*m_d_rej+1));
			*m_d_prop=min(*m_d_prop,maxfloat);

			if(rician){
				*m_tau_prop*=sqrt(float(*m_tau_acc+1)/float(*m_tau_rej+1));
				*m_tau_prop=min(*m_tau_prop,maxfloat);
				*m_tau_acc=0; 
				*m_tau_rej=0;	
			}

			if(m_include_f0){
				*m_f0_prop*=sqrt(float(*m_f0_acc+1)/float(*m_f0_rej+1));
				*m_f0_prop=min(*m_f0_prop,maxfloat);
				*m_f0_acc=0; 
				*m_f0_rej=0;	
			}	

			if(model>=2){
				*m_dstd_prop*=sqrt(float(*m_dstd_acc+1)/float(*m_dstd_rej+1));
				*m_dstd_prop=min(*m_dstd_prop,maxfloat);
				*m_dstd_acc=0; 
				*m_dstd_rej=0;	
				if(model==3){
					*m_R_prop*=sqrt(float(*m_R_acc+1)/float(*m_R_rej+1));
					*m_R_prop=min(*m_R_prop,maxfloat);
					*m_R_acc=0; 
					*m_R_rej=0;
				}
			}

			*m_S0_prop*=sqrt(float(*m_S0_acc+1)/float(*m_S0_rej+1));
			*m_S0_prop=min(*m_S0_prop,maxfloat);
			*m_d_acc=0; 
			*m_d_rej=0;
			*m_S0_acc=0; 
			*m_S0_rej=0;
			for(int f=0; f<nfib;f++){
				//m_fibres[f].update_proposals();
				m_th_prop[f]*=sqrt(float(m_th_acc[f]+1)/float(m_th_rej[f]+1));
				m_th_prop[f]=min(m_th_prop[f],maxfloat);
		      		m_ph_prop[f]*=sqrt(float(m_ph_acc[f]+1)/float(m_ph_rej[f]+1));
		      		m_ph_prop[f]=min(m_ph_prop[f],maxfloat);
		      		m_f_prop[f]*=sqrt(float(m_f_acc[f]+1)/float(m_f_rej[f]+1));
		      		m_f_prop[f]=min(m_f_prop[f],maxfloat);
			      
		      		m_th_acc[f]=0; 
		      		m_th_rej[f]=0;
		      		m_ph_acc[f]=0; 
		      		m_ph_rej[f]=0;
		      		m_f_acc[f]=0; 
		      		m_f_rej[f]=0;
			}
		}

		__syncthreads();	

        } //end while iterations

	if(leader){
		multifibres[*idVOX].m_S0=*m_S0;
		multifibres[*idVOX].m_S0_prior=*m_S0_prior;
		multifibres[*idVOX].m_S0_prop=*m_S0_prop;
		multifibres[*idVOX].m_S0_acc=*m_S0_acc;
		multifibres[*idVOX].m_S0_rej=*m_S0_rej;

		multifibres[*idVOX].m_d=*m_d;
		multifibres[*idVOX].m_d_prior=*m_d_prior;
		multifibres[*idVOX].m_d_prop=*m_d_prop;
		multifibres[*idVOX].m_d_acc=*m_d_acc;
		multifibres[*idVOX].m_d_rej=*m_d_rej;
	
		multifibres[*idVOX].m_prior_en=*m_prior_en;
		multifibres[*idVOX].m_energy=*m_energy;
		multifibres[*idVOX].m_likelihood_en=*m_likelihood_en;

		if(m_include_f0){
			multifibres[*idVOX].m_f0_prior=*m_f0_prior;
			multifibres[*idVOX].m_f0=*m_f0;
			multifibres[*idVOX].m_f0_acc=*m_f0_acc;
			multifibres[*idVOX].m_f0_rej=*m_f0_rej;
			multifibres[*idVOX].m_f0_prop=*m_f0_prop;
		}
		if(rician){
			multifibres[*idVOX].m_tau_prior=*m_tau_prior;
			multifibres[*idVOX].m_tau=*m_tau;
			multifibres[*idVOX].m_tau_acc=*m_tau_acc;
			multifibres[*idVOX].m_tau_rej=*m_tau_rej;
			multifibres[*idVOX].m_tau_prop=*m_tau_prop;
		}
		if(model>=2){
			multifibres[*idVOX].m_dstd_prior=*m_dstd_prior;
			multifibres[*idVOX].m_dstd=*m_dstd;
			multifibres[*idVOX].m_dstd_acc=*m_dstd_acc;
			multifibres[*idVOX].m_dstd_rej=*m_dstd_rej;
			multifibres[*idVOX].m_dstd_prop=*m_dstd_prop;
			if(model==3){
				multifibres[*idVOX].m_R_prior=*m_R_prior;
				multifibres[*idVOX].m_R=*m_R;
				multifibres[*idVOX].m_R_acc=*m_R_acc;
				multifibres[*idVOX].m_R_rej=*m_R_rej;
				multifibres[*idVOX].m_R_prop=*m_R_prop;
			}
		}
		randstate[*idVOX]=*localrandState; // save state, otherwise random numbers will be repeated (start at the same point)
	}
	
	if(idSubVOX<nfib){
		int pos = (*idVOX*nfib)+idSubVOX;
	
		fibres[pos].m_th=m_th[idSubVOX];
		fibres[pos].m_ph=m_ph[idSubVOX];
		fibres[pos].m_f=m_f[idSubVOX];

		fibres[pos].m_th_acc=m_th_acc[idSubVOX];
		fibres[pos].m_th_rej=m_th_rej[idSubVOX];
		fibres[pos].m_ph_acc=m_ph_acc[idSubVOX];
		fibres[pos].m_ph_rej=m_ph_rej[idSubVOX];
		fibres[pos].m_f_acc=m_f_acc[idSubVOX];
		fibres[pos].m_f_rej=m_f_rej[idSubVOX];

		fibres[pos].m_prior_en=fm_prior_en[idSubVOX];
		fibres[pos].m_th_prior=m_th_prior[idSubVOX];
		fibres[pos].m_ph_prior=m_ph_prior[idSubVOX];
		fibres[pos].m_f_prior=m_f_prior[idSubVOX];

		fibres[pos].m_th_prop=m_th_prop[idSubVOX];
		fibres[pos].m_ph_prop=m_ph_prop[idSubVOX];
		fibres[pos].m_f_prop=m_f_prop[idSubVOX];

		fibres[pos].m_lam_jump=m_lam_jump[idSubVOX];		
	}
}

